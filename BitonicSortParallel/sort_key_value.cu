#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/host.h"
#include "constants.h"
#include "sort.h"
#include "kernels_key_value.h"


/*
Sorts sub-blocks of input data with bitonic sort.
*/
template <order_t sortOrder, uint_t threadsBitonicSort, uint_t elemsThreadBitonicSort>
void BitonicSortParallel::runBitoicSortKernel(data_t *d_keys, data_t *d_values, uint_t arrayLength)
{
    uint_t elemsPerThreadBlock = threadsBitonicSort * elemsThreadBitonicSort;
    // "2 *" becaues of key-value pairs
    uint_t sharedMemSize = 2 * elemsPerThreadBlock * sizeof(*d_keys);

    dim3 dimGrid((arrayLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(threadsBitonicSort, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortKernel
            <sortOrder, threadsBitonicSort, elemsThreadBitonicSort><<<dimGrid, dimBlock, sharedMemSize>>>(
            d_keys, d_values, arrayLength
        );
    }
}

/*
Merges array, if data blocks are larger than shared memory size. It executes only of STEP on PHASE per
kernel launch.
*/
template <order_t sortOrder, uint_t threadsMerge, uint_t elemsThreadMerge>
void BitonicSortParallel::runBitonicMergeGlobalKernel(
    data_t *d_keys, data_t *d_values, uint_t arrayLength, uint_t phase, uint_t step
)
{
    uint_t elemsPerThreadBlock = threadsMerge * elemsThreadMerge;
    dim3 dimGrid((arrayLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(threadsMerge, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (isFirstStepOfPhase)
    {
        bitonicMergeGlobalKernel<sortOrder, true, threadsMerge, elemsThreadMerge><<<dimGrid, dimBlock>>>(
            d_keys, d_values, arrayLength, step
        );
    }
    else
    {
        bitonicMergeGlobalKernel<sortOrder, false, threadsMerge, elemsThreadMerge><<<dimGrid, dimBlock>>>(
            d_keys, d_values, arrayLength, step
        );
    }
}

/*
Merges array when stride is lower than shared memory size. It executes all remaining STEPS of current PHASE.
*/
template <order_t sortOrder, uint_t threadsMerge, uint_t elemsThreadMerge>
void BitonicSortParallel::runBitoicMergeLocalKernel(
    data_t *d_keys, data_t *d_values, uint_t arrayLength, uint_t phase, uint_t step
)
{
    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = threadsMerge * elemsThreadMerge;
    uint_t sharedMemSize = 2 * elemsPerThreadBlock * sizeof(*d_keys);  // "2 *" becaues of key-value pairs
    dim3 dimGrid((arrayLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(threadsMerge, 1, 1);

    bool isFirstStepOfPhase = phase == step;


    if (isFirstStepOfPhase) {
        bitonicMergeLocalKernel
            <sortOrder, true, threadsMerge, elemsThreadMerge><<<dimGrid, dimBlock, sharedMemSize>>>(
            d_keys, d_values, arrayLength, step
        );
    }
    else
    {
        bitonicMergeLocalKernel
            <sortOrder, false, threadsMerge, elemsThreadMerge><<<dimGrid, dimBlock, sharedMemSize>>>(
            d_keys, d_values, arrayLength, step
        );
    }
}

/*
Sorts data with parallel NORMALIZED BITONIC SORT.
*/
template <order_t sortOrder>
void BitonicSortParallel::bitonicSortParallel(
    data_t *d_keys, data_t *d_values, uint_t arrayLength
)
{
    uint_t arrayLenPower2 = nextPowerOf2(arrayLength);
    uint_t elemsPerBlockBitonicSort = THREADS_BITONIC_SORT_KV_BSP * ELEMS_THREAD_BITONIC_SORT_KV_BSP;
    uint_t elemsPerBlockMergeLocal = THREADS_LOCAL_MERGE_KV_BSP * ELEMS_THREAD_LOCAL_MERGE_KV_BSP;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(arrayLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(arrayLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)arrayLenPower2);

    // Sorts blocks of input data with bitonic sort
    runBitoicSortKernel<sortOrder, THREADS_BITONIC_SORT_KV_BSP, ELEMS_THREAD_BITONIC_SORT_KV_BSP>(
        d_keys, d_values, arrayLength
    );

    // Bitonic merge
    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;
        while (step > phasesMergeLocal)
        {
            runBitonicMergeGlobalKernel<sortOrder, THREADS_GLOBAL_MERGE_KV_BSP, ELEMS_THREAD_GLOBAL_MERGE_KV_BSP>(
                d_keys, d_values, arrayLength, phase, step
            );
            step--;
        }

        runBitoicMergeLocalKernel<sortOrder, THREADS_LOCAL_MERGE_KV_BSP, ELEMS_THREAD_LOCAL_MERGE_KV_BSP>(
            d_keys, d_values, arrayLength, phase, step
        );
    }
}

/*
Wrapper for bitonic sort method.
The code runs faster if arguments are passed to method. If members are accessed directly, code runs slower.
*/
void BitonicSortParallel::sortKeyValue()
{
    if (_sortOrder == ORDER_ASC)
    {
        bitonicSortParallel<ORDER_ASC>(_d_keys, _d_values, _arrayLength);
    }
    else
    {
        bitonicSortParallel<ORDER_DESC>(_d_keys, _d_values, _arrayLength);
    }
}
