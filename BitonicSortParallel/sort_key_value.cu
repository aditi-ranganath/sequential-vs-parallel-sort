#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "sort.h"
#include "kernels_key_value.h"


/*
Sorts sub-blocks of input data with bitonic sort.
*/
template <order_t sortOrder>
void BitonicSortParallel::runBitoicSortKernelKeyValue(data_t *d_keys, data_t *d_values, uint_t arrayLength)
{
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT_KV * ELEMS_PER_THREAD_BITONIC_SORT_KV;
    // "2 *" becaues of key-value pairs
    uint_t sharedMemSize = 2 * elemsPerThreadBlock * sizeof(*d_keys);

    dim3 dimGrid((arrayLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT_KV, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortKernel<sortOrder><<<dimGrid, dimBlock, sharedMemSize>>>(d_keys, d_values, arrayLength);
    }
}

/*
Merges array, if data blocks are larger than shared memory size. It executes only of STEP on PHASE per
kernel launch.
*/
template <order_t sortOrder>
void BitonicSortParallel::runBitonicMergeGlobalKernelKeyValue(
    data_t *d_keys, data_t *d_values, uint_t arrayLength, uint_t phase, uint_t step
)
{
    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE_KV * ELEMS_PER_THREAD_GLOBAL_MERGE_KV;
    dim3 dimGrid((arrayLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE_KV, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (isFirstStepOfPhase)
    {
        bitonicMergeGlobalKernel<sortOrder, true><<<dimGrid, dimBlock>>>(d_keys, d_values, arrayLength, step);
    }
    else
    {
        bitonicMergeGlobalKernel<sortOrder, false><<<dimGrid, dimBlock>>>(d_keys, d_values, arrayLength, step);
    }
}

/*
Merges array when stride is lower than shared memory size. It executes all remaining STEPS of current PHASE.
*/
template <order_t sortOrder>
void BitonicSortParallel::runBitoicMergeLocalKernelKeyValue(
    data_t *d_keys, data_t *d_values, uint_t arrayLength, uint_t phase, uint_t step
)
{
    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE_KV * ELEMS_PER_THREAD_LOCAL_MERGE_KV;
    uint_t sharedMemSize = 2 * elemsPerThreadBlock * sizeof(*d_keys);  // "2 *" becaues of key-value pairs
    dim3 dimGrid((arrayLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE_KV, 1, 1);

    bool isFirstStepOfPhase = phase == step;


    if (isFirstStepOfPhase) {
        bitonicMergeLocalKernel<sortOrder, true><<<dimGrid, dimBlock, sharedMemSize>>>(
            d_keys, d_values, arrayLength, step
        );
    }
    else
    {
        bitonicMergeLocalKernel<sortOrder, false><<<dimGrid, dimBlock, sharedMemSize>>>(
            d_keys, d_values, arrayLength, step
        );
    }
}

/*
Sorts data with parallel NORMALIZED BITONIC SORT.
*/
template <order_t sortOrder>
void BitonicSortParallel::bitonicSortParallelKeyValue(
    data_t *d_keys, data_t *d_values, uint_t arrayLength
)
{
    uint_t arrayLenPower2 = nextPowerOf2(arrayLength);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT_KV * ELEMS_PER_THREAD_BITONIC_SORT_KV;
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE_KV * ELEMS_PER_THREAD_LOCAL_MERGE_KV;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(arrayLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(arrayLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)arrayLenPower2);

    // Sorts blocks of input data with bitonic sort
    runBitoicSortKernelKeyValue<sortOrder>(d_keys, d_values, arrayLength);

    // Bitonic merge
    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;
        while (step > phasesMergeLocal)
        {
            runBitonicMergeGlobalKernelKeyValue<sortOrder>(d_keys, d_values, arrayLength, phase, step);
            step--;
        }

        runBitoicMergeLocalKernelKeyValue<sortOrder>(d_keys, d_values, arrayLength, phase, step);
    }
}

/*
Wrapper for bitonic sort method.
The code runs faster if arguments are passed to method. If members are accessed directly, code runs slower.
*/
void BitonicSortParallel::sortKeyValue()
{
    if (_sortOrder == ORDER_ASC)
    {
        bitonicSortParallelKeyValue<ORDER_ASC>(_d_keys, _d_values, _arrayLength);
    }
    else
    {
        bitonicSortParallelKeyValue<ORDER_DESC>(_d_keys, _d_values, _arrayLength);
    }
}
