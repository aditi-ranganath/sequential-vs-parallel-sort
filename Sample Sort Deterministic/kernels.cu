#include "hip/hip_runtime.h"
#include <stdio.h>
#include <climits>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "data_types.h"
#include "constants.h"


__global__ void printElemsKernel(el_t *table, uint_t tableLen) {
    for (uint_t i = 0; i < tableLen; i++) {
        printf("%2d ", table[i].key);
    }
    printf("\n");
}

__global__ void printDataKernel(data_t *table, uint_t tableLen) {
    for (uint_t i = 0; i < tableLen; i++) {
        printf("%2d ", table[i]);
    }
    printf("\n");
}

/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, order_t sortOrder) {
    if (((int_t)(elem1->key - elem2->key) > 0) ^ sortOrder) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
Sorts sub-blocks of input data with NORMALIZED bitonic sort.
*/
__global__ void bitonicSortKernel(el_t *dataTable, data_t *localSamples, uint_t tableLen, order_t sortOrder) {
    extern __shared__ el_t bitonicSortTile[];

    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t offset = blockIdx.x * elemsPerThreadBlock;
    uint_t dataBlockLength = offset + elemsPerThreadBlock <= tableLen ? elemsPerThreadBlock : tableLen - offset;

    // Read data from global to shared memory.
    for (uint_t tx = threadIdx.x; tx < dataBlockLength; tx += THREADS_PER_BITONIC_SORT) {
        bitonicSortTile[tx] = dataTable[offset + tx];
    }
    __syncthreads();

    // Bitonic sort PHASES
    for (uint_t subBlockSize = 1; subBlockSize < dataBlockLength; subBlockSize <<= 1) {
        // Bitonic merge STEPS
        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            for (uint_t tx = threadIdx.x; tx < dataBlockLength >> 1; tx += THREADS_PER_BITONIC_SORT) {
                uint_t indexThread = tx;
                uint_t offset = stride;

                // In normalized bitonic sort, first STEP of every PHASE uses different offset than all other STEPS.
                if (stride == subBlockSize) {
                    indexThread = (tx / stride) * stride + ((stride - 1) - (tx % stride));
                    offset = ((tx & (stride - 1)) << 1) + 1;
                }

                uint_t index = (indexThread << 1) - (indexThread & (stride - 1));
                if (index + offset >= dataBlockLength) {
                    break;
                }

                compareExchange(&bitonicSortTile[index], &bitonicSortTile[index + offset], sortOrder);
            }
            __syncthreads();
        }
    }

    // Store data from shared to global memory
    for (uint_t tx = threadIdx.x; tx < dataBlockLength; tx += THREADS_PER_BITONIC_SORT) {
        dataTable[offset + tx] = bitonicSortTile[tx];

        if (tx * NUM_SAMPLES < dataBlockLength) {
            localSamples[blockIdx.x * NUM_SAMPLES + tx] = bitonicSortTile[tx * NUM_SAMPLES].key;
        }
    }
}
