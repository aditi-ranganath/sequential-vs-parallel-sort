#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <cudpp.h>

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Initializes CUDPP scan.
*/
void cudppInitScan(CUDPPHandle *scanPlan, uint_t tableLen)
{
    // Initializes the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    *scanPlan = 0;
    CUDPPResult result = cudppPlan(theCudpp, scanPlan, config, tableLen, 1, 0);

    if (result != CUDPP_SUCCESS)
    {
        printf("Error creating CUDPPPlan\n");
        getchar();
        exit(-1);
    }
}

/*
Adds padding of MAX/MIN values to input table, deppending if sort order is ascending or descending. This is
needed, if table length is not divisable with number of elements sorted by local radix sort. In order for
parallel sort to work, table length has to be divisable with number of elements processed by one thread block
in local radix sort.
*/
void runAddPaddingKernel(data_t *dataTable, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerLocalSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t tableLenRoundedUp = roundUp(tableLen, elemsPerLocalSort);

    // If table length is already power of 2, than no padding is needed
    if (tableLen == tableLenRoundedUp)
    {
        return;
    }

    uint_t paddingLength = tableLenRoundedUp - tableLen;

    uint_t elemsPerThreadBlock = THREADS_PER_PADDING * ELEMS_PER_THREAD_PADDING;;
    dim3 dimGrid((paddingLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_PADDING, 1, 1);

    // Depending on sort order different value is used for padding.
    if (sortOrder == ORDER_ASC)
    {
        addPaddingKernel<MAX_VAL><<<dimGrid, dimBlock>>>(dataTable, tableLen, paddingLength);
    }
    else
    {
        addPaddingKernel<MIN_VAL><<<dimGrid, dimBlock>>>(dataTable, tableLen, paddingLength);
    }
}

/*
Sorts sub-blocks of input data with NORMALIZED bitonic sort and collects NUM_SAMPLES_PARALLEL samples from every
sorted chunk after the sort is complete.
*/
void runBitonicSortCollectSamplesKernel(data_t *dataTable, data_t *samples, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortCollectSamplesKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, samples, tableLen
        );
    }
    else
    {
        bitonicSortCollectSamplesKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, samples, tableLen
        );
    }
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitonicSortKernel(data_t *dataTable, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, tableLen
        );
    }
    else
    {
        bitonicSortKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, tableLen
        );
    }
}

void runBitonicMergeGlobalKernel(
    data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder
)
{
    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_ASC, true><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_ASC, false><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
    }
    else
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_DESC, true><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_DESC, false><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
    }
}

void runBitoicMergeLocalKernel(data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder)
{
    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeLocalKernel<ORDER_ASC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_ASC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
    }
    else
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeLocalKernel<ORDER_DESC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_DESC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
    }
}

/*
From sorted LOCAL samples collects (NUM_SAMPLES_PARALLEL) GLOBAL samples.
*/
void runCollectGlobalSamplesKernel(data_t *samplesLocal, data_t *samplesGlobal, uint_t samplesLen)
{
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(NUM_SAMPLES_PARALLEL, 1, 1);

    collectGlobalSamplesKernel<<<dimGrid, dimBlock>>>(samplesLocal, samplesGlobal, samplesLen);
}

/*
In all previously sorted (by initial bitonic sort) sub-blocks finds the indexes of all NUM_SAMPLES_PARALLEL
global samples. From these indexes calculates the number of elements in each of the (NUM_SAMPLES_PARALLEL + 1)
local buckets (calculates local bucket sizes) for every sorted sub-block.
*/
void runSampleIndexingKernel(
    data_t *dataTable, data_t *samples, uint_t *bucketSizes, uint_t tableLen, uint_t numAllBuckets,
    order_t sortOrder
)
{
    uint_t elemsPerBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t subBlocksPerThreadBlock = THREADS_PER_SAMPLE_INDEXING / NUM_SAMPLES_PARALLEL;

    // "Number of all sorted sub-blocks" / "number of sorted sub-blocks processed by one thread block"
    dim3 dimGrid((tableLen / elemsPerBitonicSort - 1) / subBlocksPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_SAMPLE_INDEXING, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        sampleIndexingKernel<ORDER_ASC><<<dimGrid, dimBlock>>>(dataTable, samples, bucketSizes, tableLen);
    }
    else
    {
        sampleIndexingKernel<ORDER_DESC><<<dimGrid, dimBlock>>>(dataTable, samples, bucketSizes, tableLen);
    }
}

/*
With respect to local bucket sizes and offsets scatters elements to their global buckets. At the end it coppies
global bucket sizes (sizes of whole global buckets, not just bucket sizes per every sorted sub-block) to host.
*/
void runBucketsRelocationKernel(
    data_t *d_dataTable, data_t *d_dataBuffer, uint_t *h_globalBucketOffsets, uint_t *d_globalBucketOffsets,
    uint_t *localBucketSizes, uint_t *localBucketOffsets, uint_t tableLen
)
{
    // For NUM_SAMPLES_PARALLEL samples (NUM_SAMPLES_PARALLEL + 1) buckets are created
    // "2" -> bucket sizes + bucket offsets
    uint_t sharedMemSize = 2 * (NUM_SAMPLES_PARALLEL + 1) * sizeof(*localBucketSizes);
    uint_t elemsPerInitBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    hipError_t error;

    dim3 dimGrid((tableLen - 1) / elemsPerInitBitonicSort + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BUCKETS_RELOCATION, 1, 1);

    bucketsRelocationKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        d_dataTable, d_dataBuffer, d_globalBucketOffsets, localBucketSizes, localBucketOffsets, tableLen
    );

    error = hipMemcpy(
        h_globalBucketOffsets, d_globalBucketOffsets, (NUM_SAMPLES_PARALLEL + 1) * sizeof(*h_globalBucketOffsets),
        hipMemcpyDeviceToHost
    );
    checkCudaError(error);
}

/*
Performs global bitonic merge, when number of elements is greater than shared memory size.
*/
void bitonicMerge(data_t *dataTable, uint_t tableLen, order_t sortOrder)
{
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;
        while (step > phasesMergeLocal)
        {
            runBitonicMergeGlobalKernel(dataTable, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel(dataTable, tableLen, phase, step, sortOrder);
    }
}

/*
Performs bitonic sort.
*/
void bitonicSort(data_t *dataTable, uint_t tableLen, order_t sortOrder)
{
    runBitonicSortKernel(dataTable, tableLen, sortOrder);
    bitonicMerge(dataTable, tableLen, sortOrder);
}

/*
Sorts array with deterministic sample sort.
*/
void sampleSort(
    data_t *&d_dataTable, data_t *&d_dataBuffer, data_t *d_samplesLocal, data_t *d_samplesGlobal,
    uint_t *h_globalBucketOffsets, uint_t *d_globalBucketOffsets, uint_t *d_localBucketSizes,
    uint_t *d_localBucketOffsets, uint_t tableLen, order_t sortOrder
)
{
    uint_t elemsPerInitBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    // If table length is not multiple of number of elements processed by one thread block in initial
    // bitonic sort, than array is padded to that length.
    uint_t tableLenRoundedUp = roundUp(tableLen, elemsPerInitBitonicSort);
    uint_t localSamplesDistance = (elemsPerInitBitonicSort - 1) / NUM_SAMPLES_PARALLEL + 1;
    uint_t localSamplesLen = (tableLenRoundedUp - 1) / localSamplesDistance + 1;
    // (number of all data blocks (tiles)) * (number buckets generated from NUM_SAMPLES_PARALLEL)
    uint_t localBucketsLen = ((tableLenRoundedUp - 1) / elemsPerInitBitonicSort + 1) * (NUM_SAMPLES_PARALLEL + 1);
    CUDPPHandle scanPlan;

    cudppInitScan(&scanPlan, localBucketsLen);
    runAddPaddingKernel(d_dataTable, tableLen, sortOrder);
    // Sorts sub-blocks of input data with bitonic sort and from every chunk collects NUM_SAMPLES_PARALLEL samples
    runBitonicSortCollectSamplesKernel(d_dataTable, d_samplesLocal, tableLenRoundedUp, sortOrder);

    // Array has already been sorted
    if (tableLen <= elemsPerInitBitonicSort)
    {
        data_t *temp = d_dataTable;
        d_dataTable = d_dataBuffer;
        d_dataBuffer = temp;

        return;
    }

    // Sorts collected local samples
    bitonicSort(d_samplesLocal, localSamplesLen, sortOrder);
    // From sorted LOCAL samples collects NUM_SAMPLES_PARALLEL global samples
    runCollectGlobalSamplesKernel(d_samplesLocal, d_samplesGlobal, localSamplesLen);
    // For all previously sorted sub-blocks calculates bucket sizes for global samples
    runSampleIndexingKernel(
        d_dataTable, d_samplesGlobal, d_localBucketSizes, tableLenRoundedUp, localBucketsLen, sortOrder
    );

    // Performs scan on local bucket sizes to gain local bucket offsets (global offset for all local buckets)
    CUDPPResult result = cudppScan(scanPlan, d_localBucketOffsets, d_localBucketSizes, localBucketsLen);
    if (result != CUDPP_SUCCESS)
    {
        printf("Error in cudppScan()\n");
        getchar();
        exit(-1);
    }

    // Moves elements to their corresponding global buckets and calculates global bucket offsets
    runBucketsRelocationKernel(
        d_dataTable, d_dataBuffer, h_globalBucketOffsets, d_globalBucketOffsets, d_localBucketSizes,
        d_localBucketOffsets, tableLen
    );

    // Sorts every bucket with bitonic sort
    uint_t previousOffset = 0;
    for (uint_t bucket = 0; bucket < NUM_SAMPLES_PARALLEL + 1; bucket++)
    {
        // Padded part of the array doesn't need to be sorted in last bucket
        uint_t currentOffset = bucket < NUM_SAMPLES_PARALLEL ? h_globalBucketOffsets[bucket] : tableLen;
        uint_t bucketLen = currentOffset - previousOffset;

        if (bucketLen > 0)
        {
            bitonicSort(d_dataBuffer + previousOffset, bucketLen, sortOrder);
        }
        previousOffset = currentOffset;
    }
}

/*
Sorts input data with parallel sample sort.
*/
double sortParallel(
    data_t *h_output, data_t *d_dataTable, data_t *d_dataBuffer, data_t *d_samplesLocal, data_t *d_samplesGlobal,
    uint_t *d_localBucketSizes, uint_t *d_localBucketOffsets, uint_t *h_globalBucketOffsets,
    uint_t *d_globalBucketOffsets, uint_t tableLen, order_t sortOrder
)
{
    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    sampleSort(
        d_dataTable, d_dataBuffer, d_samplesLocal, d_samplesGlobal, h_globalBucketOffsets, d_globalBucketOffsets,
        d_localBucketSizes, d_localBucketOffsets, tableLen, sortOrder
    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_output, d_dataBuffer, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
