#include "hip/hip_runtime.h"
#include <stdio.h>
#include <climits>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes DEVICE memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, data_t **d_samples, uint_t tableLen,
                uint_t localSamplesLen) {
    hipError_t error;

    // Data memory allocation
    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);
    error = hipMalloc(d_samples, localSamplesLen * sizeof(**d_samples));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitonicSortCollectSamplesKernel(el_t *dataTable, data_t *samples, uint_t tableLen, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    startStopwatch(&timer);
    bitonicSortCollectSamplesKernel<el_t><<<dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable)>>>(
        dataTable, samples, tableLen, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

template <typename T>
void runBitonicMergeGlobalKernel(T *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeGlobalKernel<T><<<dimGrid, dimBlock>>>(dataTable, tableLen, step, step == phase, sortOrder);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge global kernel");*/
}

template <typename T>
void runBitoicMergeLocalKernel(T *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeLocalKernel<<<dimGrid, dimBlock>>>(
        dataTable, tableLen, step, phase == step, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge local kernel");*/
}

void runCollectGlobalSamplesKernel(data_t *samples, uint_t samplesLen) {
    LARGE_INTEGER timer;

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(NUM_SAMPLES, 1, 1);

    startStopwatch(&timer);
    collectGlobalSamplesKernel<<<dimGrid, dimBlock>>>(samples, samplesLen);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing kernel for collection of global samples");*/
}

void runPrintElemsKernel(el_t *table, uint_t tableLen) {
    printElemsKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void runPrintDataKernel(data_t *table, uint_t tableLen) {
    printDataKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

template <typename T>
void bitonicMerge(T *dataTable, uint_t tableLen, uint_t elemsPerBlockBitonicSort, order_t sortOrder) {
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++) {
        uint_t step = phase;
        while (step > phasesMergeLocal) {
            runBitonicMergeGlobalKernel<T>(dataTable, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel<T>(dataTable, tableLen, phase, step, sortOrder);
    }
}

el_t* sampleSort(el_t *dataTable, el_t *dataBuffer, data_t *samples, uint_t tableLen, uint_t localSamplesLen,
                 order_t sortOrder) {
    runBitonicSortCollectSamplesKernel(dataTable, samples, tableLen, sortOrder);

    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    if (tableLen <= elemsPerBlockBitonicSort) {
        return dataTable;
    }

    bitonicMerge<data_t>(samples, localSamplesLen, NUM_SAMPLES, sortOrder);
    runCollectGlobalSamplesKernel(samples, localSamplesLen);

    // TODO handle case, if all samples are the same
    runPrintDataKernel(samples, NUM_SAMPLES);

    // TODO other steps
    return dataTable;
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, order_t sortOrder) {
    el_t *d_dataInput, *d_dataBuffer, *d_dataResult;
    data_t *d_samples;

    uint_t localSamplesDistance = (THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT) / NUM_SAMPLES;
    uint_t localSamplesLen = (tableLen - 1) / localSamplesDistance + 1;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_dataInput, &d_dataInput, &d_dataBuffer, &d_samples, tableLen, localSamplesLen);

    startStopwatch(&timer);
    d_dataResult = sampleSort(d_dataInput, d_dataBuffer, d_samples, tableLen, localSamplesLen, sortOrder);

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel sample sort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataResult, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
    hipFree(d_dataBuffer);
    hipFree(d_samples);
}
