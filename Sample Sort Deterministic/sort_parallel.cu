#include "hip/hip_runtime.h"
#include <stdio.h>
#include <climits>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include <cudpp.h>

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes DEVICE memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, data_t **d_samples,
                uint_t **d_localBucketSizes, uint_t **d_localBucketOffsets, uint_t tableLen,
                uint_t localSamplesLen, uint_t localBucketsLen) {
    hipError_t error;

    // Data memory allocation
    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);
    error = hipMalloc(d_samples, localSamplesLen * sizeof(**d_samples));
    checkCudaError(error);
    error = hipMalloc(d_localBucketSizes, localBucketsLen * sizeof(**d_localBucketSizes));
    checkCudaError(error);
    error = hipMalloc(d_localBucketOffsets, localBucketsLen * sizeof(**d_localBucketOffsets));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void cudppInitScan(CUDPPHandle *scanPlan, uint_t tableLen) {
    // Initializes the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    *scanPlan = 0;
    CUDPPResult result = cudppPlan(theCudpp, scanPlan, config, tableLen, 1, 0);

    if (result != CUDPP_SUCCESS) {
        printf("Error creating CUDPPPlan\n");
        getchar();
        exit(-1);
    }
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitonicSortCollectSamplesKernel(el_t *dataTable, data_t *samples, uint_t tableLen, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    startStopwatch(&timer);
    bitonicSortCollectSamplesKernel<el_t><<<dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable)>>>(
        dataTable, samples, tableLen, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

template <typename T>
void runBitonicMergeGlobalKernel(T *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeGlobalKernel<T><<<dimGrid, dimBlock>>>(dataTable, tableLen, step, step == phase, sortOrder);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge global kernel");*/
}

template <typename T>
void runBitoicMergeLocalKernel(T *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeLocalKernel<<<dimGrid, dimBlock>>>(
        dataTable, tableLen, step, phase == step, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge local kernel");*/
}

void runCollectGlobalSamplesKernel(data_t *samples, uint_t samplesLen) {
    LARGE_INTEGER timer;

    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(NUM_SAMPLES, 1, 1);

    startStopwatch(&timer);
    collectGlobalSamplesKernel<<<dimGrid, dimBlock>>>(samples, samplesLen);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing kernel for collection of global samples");*/
}

void runSampleIndexingKernel(el_t *dataTable, data_t *samples, data_t *bucketSizes, uint_t tableLen,
                             uint_t numAllBuckets, order_t sortOrder) {
    LARGE_INTEGER timer;

    // TODO comment
    dim3 dimGrid((numAllBuckets - 1) / (THREADS_PER_SAMPLE_INDEXING / NUM_SAMPLES * (NUM_SAMPLES + 1)) + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_SAMPLE_INDEXING, 1, 1);

    startStopwatch(&timer);
    sampleIndexingKernel<<<dimGrid, dimBlock>>>(dataTable, samples, bucketSizes, tableLen, sortOrder);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing kernel sample indexing");*/
}

void runPrintElemsKernel(el_t *table, uint_t tableLen) {
    printElemsKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void runPrintDataKernel(data_t *table, uint_t tableLen) {
    printDataKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

template <typename T>
void bitonicMerge(T *dataTable, uint_t tableLen, uint_t elemsPerBlockBitonicSort, order_t sortOrder) {
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++) {
        uint_t step = phase;
        while (step > phasesMergeLocal) {
            runBitonicMergeGlobalKernel<T>(dataTable, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel<T>(dataTable, tableLen, phase, step, sortOrder);
    }
}

el_t* sampleSort(el_t *dataTable, el_t *dataBuffer, data_t *samples, uint_t *d_localBucketSizes,
                 uint_t *d_localBucketOffsets, uint_t tableLen, uint_t localSamplesLen, uint_t localBucketsLen,
                 order_t sortOrder) {
    CUDPPHandle scanPlan;

    // TODO Should this be done before or after stopwatch?
    cudppInitScan(&scanPlan, localBucketsLen);
    runBitonicSortCollectSamplesKernel(dataTable, samples, tableLen, sortOrder);

    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    if (tableLen <= elemsPerBlockBitonicSort) {
        return dataTable;
    }

    bitonicMerge<data_t>(samples, localSamplesLen, NUM_SAMPLES, sortOrder);
    // TODO handle case, if all samples are the same
    runCollectGlobalSamplesKernel(samples, localSamplesLen);
    runPrintDataKernel(samples, NUM_SAMPLES);
    runSampleIndexingKernel(dataTable, samples, d_localBucketSizes, tableLen, localBucketsLen, sortOrder);

    CUDPPResult result = cudppScan(scanPlan, d_localBucketOffsets, d_localBucketSizes, localBucketsLen);
    if (result != CUDPP_SUCCESS) {
        printf("Error in cudppScan()\n");
        getchar();
        exit(-1);
    }

    runPrintDataKernel(d_localBucketSizes, localBucketsLen);
    runPrintDataKernel(d_localBucketOffsets, localBucketsLen);

    // TODO other steps
    return dataTable;
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, order_t sortOrder) {
    el_t *d_dataInput, *d_dataBuffer, *d_dataResult;
    // First it holds LOCAL and than GLOBAL samples
    data_t *d_samples;
    uint_t *d_globalBucketSizes, *d_localBucketSizes, *d_localBucketOffsets;

    uint_t elemsPerInitBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t localSamplesDistance = (THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT) / NUM_SAMPLES;
    uint_t localSamplesLen = (tableLen - 1) / localSamplesDistance + 1;
    // (number of all data blocks (tiles)) * (number buckets generated from NUM_SAMPLES)
    uint_t localBucketsLen = ((tableLen - 1) / elemsPerInitBitonicSort + 1) * (NUM_SAMPLES + 1);

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(
        h_dataInput, &d_dataInput, &d_dataBuffer, &d_samples, &d_localBucketSizes, &d_localBucketOffsets,
        tableLen, localSamplesLen, localBucketsLen
    );

    startStopwatch(&timer);
    d_dataResult = sampleSort(
        d_dataInput, d_dataBuffer, d_samples, d_localBucketSizes, d_localBucketOffsets, tableLen,
        localSamplesLen, localBucketsLen, sortOrder
    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel sample sort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataResult, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
    hipFree(d_dataBuffer);
    hipFree(d_samples);
}
