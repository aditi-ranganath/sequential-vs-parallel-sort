#include <stdio.h>
#include <climits>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"

/*
Initializes DEVICE memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, uint_t tableLen) {
    hipError_t error;

    // Data memory allocation
    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(el_t *dataTable, uint_t tableLen, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel << <dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable) >> >(
        dataTable, tableLen, sortOrder
        );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sampleSort(el_t *dataTable, el_t *dataBuffer, uint_t tableLen, order_t sortOrder) {
    runBitoicSortKernel(dataTable, tableLen, sortOrder);
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, order_t sortOrder) {
    el_t *d_dataInput, *d_dataBuffer;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_dataInput, &d_dataInput, &d_dataBuffer, tableLen);

    startStopwatch(&timer);
    sampleSort(d_dataInput, d_dataBuffer, tableLen, sortOrder);

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel sample sort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataInput, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
}
