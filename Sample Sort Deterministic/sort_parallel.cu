#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <cudpp.h>

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Initializes CUDPP scan.
*/
void cudppInitScan(CUDPPHandle *scanPlan, uint_t tableLen)
{
    // Initializes the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    *scanPlan = 0;
    CUDPPResult result = cudppPlan(theCudpp, scanPlan, config, tableLen, 1, 0);

    if (result != CUDPP_SUCCESS) {
        printf("Error creating CUDPPPlan\n");
        getchar();
        exit(-1);
    }
}

/*
Sorts sub-blocks of input data with bitonic sort and collects samples after the sort is complete.
*/
void runBitonicSortCollectSamplesKernel(data_t *dataTable, data_t *samples, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortCollectSamplesKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, samples, tableLen
        );
    }
    else
    {
        bitonicSortCollectSamplesKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, samples, tableLen
        );
    }
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitonicSortKernel(data_t *dataTable, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, tableLen
        );
    }
    else
    {
        bitonicSortKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, tableLen
        );
    }
}

void runBitonicMergeGlobalKernel(
    data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder
)
{
    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_ASC, true><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_ASC, false><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
    }
    else
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_DESC, true><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_DESC, false><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
    }
}

void runBitoicMergeLocalKernel(data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder)
{
    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeLocalKernel<ORDER_ASC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_ASC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
    }
    else
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeLocalKernel<ORDER_DESC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_DESC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
    }
}

/*
From all LOCAL samples collects (NUM_SAMPLES) GLOBAL samples.
*/
void runCollectGlobalSamplesKernel(data_t *samples, uint_t samplesLen)
{
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(NUM_SAMPLES, 1, 1);

    collectGlobalSamplesKernel<<<dimGrid, dimBlock>>>(samples, samplesLen);
}

/*
For every sample searches, how many elements in tile are lower than it's value.
*/
void runSampleIndexingKernel(
    data_t *dataTable, data_t *samples, data_t *bucketSizes, uint_t tableLen, uint_t numAllBuckets,
    order_t sortOrder
)
{
    // Number of threads per thread block can be greater than number of samples.
    uint_t elemsPerBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t numBlocks = (tableLen - 1) / elemsPerBitonicSort + 1;
    uint_t threadBlockSize = min(numBlocks * NUM_SAMPLES, THREADS_PER_SAMPLE_INDEXING);

    // Every thread block creates from NUM_SAMPLES samples (NUM_SAMPLES + 1) buckets
    dim3 dimGrid((numAllBuckets - 1) / (threadBlockSize / NUM_SAMPLES * (NUM_SAMPLES + 1)) + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        sampleIndexingKernel<ORDER_ASC><<<dimGrid, dimBlock>>>(dataTable, samples, bucketSizes, tableLen);
    }
    else
    {
        sampleIndexingKernel<ORDER_DESC><<<dimGrid, dimBlock>>>(dataTable, samples, bucketSizes, tableLen);
    }
}

/*
From local bucket sizes and offsets scatters elements to their global buckets. At the end it coppies
global bucket sizes (sizes of whole buckets, not just bucket size per tile (local size)) to host.
*/
void runBucketsRelocationKernel(
    data_t *dataTable, data_t *dataBuffer, uint_t *h_globalBucketOffsets, uint_t *d_globalBucketOffsets,
    uint_t *localBucketSizes, uint_t *localBucketOffsets, uint_t tableLen
)
{
    // For NUM_SAMPLES samples (NUM_SAMPLES + 1) buckets are created
    uint_t sharedMemSize = 2 * (NUM_SAMPLES + 1) * sizeof(*localBucketSizes);
    uint_t elemsPerBitonicSort = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    hipError_t error;

    dim3 dimGrid((tableLen - 1) / elemsPerBitonicSort + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BUCKETS_RELOCATION, 1, 1);

    bucketsRelocationKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        dataTable, dataBuffer, d_globalBucketOffsets, localBucketSizes, localBucketOffsets, tableLen
    );

    error = hipMemcpy(
        h_globalBucketOffsets, d_globalBucketOffsets, (NUM_SAMPLES + 1) * sizeof(*h_globalBucketOffsets),
        hipMemcpyDeviceToHost
    );
    checkCudaError(error);
}

/*
Performs global bitonic merge, when number of elements is greater than shared memory size.
*/
void bitonicMerge(data_t *dataTable, uint_t tableLen, uint_t elemsPerBlockBitonicSort, order_t sortOrder)
{
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;
        while (step > phasesMergeLocal)
        {
            runBitonicMergeGlobalKernel(dataTable, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel(dataTable, tableLen, phase, step, sortOrder);
    }
}

/*
Performs bitonic sort.
*/
void bitonicSort(data_t *dataTable, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;

    runBitonicSortKernel(dataTable, tableLen, sortOrder);
    bitonicMerge(dataTable, tableLen, elemsPerBlockBitonicSort, sortOrder);
}

/*
Sorts array with deterministic sample sort.
*/
void sampleSort(
    data_t *&dataTable, data_t *&dataBuffer, data_t *samples, uint_t *h_globalBucketOffsets,
    uint_t *d_globalBucketOffsets, uint_t *d_localBucketSizes, uint_t *d_localBucketOffsets, uint_t tableLen,
    order_t sortOrder
)
{
    uint_t elemsPerInitBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t localSamplesDistance = (THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT) / NUM_SAMPLES;
    uint_t localSamplesLen = (tableLen - 1) / localSamplesDistance + 1;
    // (number of all data blocks (tiles)) * (number buckets generated from NUM_SAMPLES)
    uint_t localBucketsLen = ((tableLen - 1) / elemsPerInitBitonicSort + 1) * (NUM_SAMPLES + 1);
    CUDPPHandle scanPlan;

    cudppInitScan(&scanPlan, localBucketsLen);
    runBitonicSortCollectSamplesKernel(dataTable, samples, tableLen, sortOrder);

    // Array has already been sorted
    if (tableLen <= elemsPerInitBitonicSort) {
        data_t *temp = dataTable;
        dataTable = dataBuffer;
        dataBuffer = temp;

        return;
    }

    // Local samples are already partially ordered - NUM_SAMPLES per every tile. These partially ordered
    // samples have to be merged.
    bitonicMerge(samples, localSamplesLen, NUM_SAMPLES, sortOrder);

    // TODO handle case, if all samples are the same
    runCollectGlobalSamplesKernel(samples, localSamplesLen);
    runSampleIndexingKernel(dataTable, samples, d_localBucketSizes, tableLen, localBucketsLen, sortOrder);

    CUDPPResult result = cudppScan(scanPlan, d_localBucketOffsets, d_localBucketSizes, localBucketsLen);
    if (result != CUDPP_SUCCESS)
    {
        printf("Error in cudppScan()\n");
        getchar();
        exit(-1);
    }

    runBucketsRelocationKernel(
        dataTable, dataBuffer, h_globalBucketOffsets, d_globalBucketOffsets, d_localBucketSizes,
        d_localBucketOffsets, tableLen
    );

    // Sorts every bucket with bitonic sort
    uint_t previousOffset = 0;
    for (uint_t bucket = 0; bucket < NUM_SAMPLES + 1; bucket++)
    {
        uint_t currentOffset = h_globalBucketOffsets[bucket];
        uint_t bucketLen = currentOffset - previousOffset;

        bitonicSort(dataBuffer + previousOffset, bucketLen, sortOrder);
        previousOffset = currentOffset;
    }
}

/*
Sorts input data with parallel sample sort.
*/
double sortParallel(
    data_t *h_output, data_t *d_dataTable, data_t *d_dataBuffer, data_t *d_samples, uint_t *d_localBucketSizes,
    uint_t *d_localBucketOffsets, uint_t *h_globalBucketOffsets, uint_t *d_globalBucketOffsets, uint_t tableLen,
    order_t sortOrder
)
{
    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    sampleSort(
        d_dataTable, d_dataBuffer, d_samples, h_globalBucketOffsets, d_globalBucketOffsets, d_localBucketSizes,
        d_localBucketOffsets, tableLen, sortOrder
    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_output, d_dataBuffer, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
