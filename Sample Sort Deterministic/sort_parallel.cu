#include <stdio.h>
#include <climits>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"

/*
Initializes DEVICE memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_input, el_t **d_dataInput, uint_t tableLen) {
    hipError_t error;

    // Data memory allocation
    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, bool orderAsc) {
    el_t *d_dataInput;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_dataInput, &d_dataInput, tableLen);

    startStopwatch(&timer);

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel sample sort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataInput, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
}
