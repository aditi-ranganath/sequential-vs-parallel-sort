#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <cudpp.h>

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Initializes CUDPP scan.
*/
void cudppInitScan(CUDPPHandle *scanPlan, uint_t tableLen)
{
    // Initializes the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    *scanPlan = 0;
    CUDPPResult result = cudppPlan(theCudpp, scanPlan, config, tableLen, 1, 0);

    if (result != CUDPP_SUCCESS) {
        printf("Error creating CUDPPPlan\n");
        getchar();
        exit(-1);
    }
}

/*
Sorts sub-blocks of input data with bitonic sort and collects samples after the sort is complete.
*/
void runBitonicSortCollectSamplesKernel(data_t *dataTable, data_t *samples, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortCollectSamplesKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, samples, tableLen
        );
    }
    else
    {
        bitonicSortCollectSamplesKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, samples, tableLen
        );
    }
}

///*
//Sorts sub-blocks of input data with bitonic sort.
//*/
//void runBitonicSortKernel(el_t *dataTable, uint_t tableLen, order_t sortOrder) {
//    hipError_t error;
//    LARGE_INTEGER timer;
//
//    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
//    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
//    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);
//
//    startStopwatch(&timer);
//    bitonicSortKernel<el_t><<<dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable)>>>(
//        dataTable, tableLen, sortOrder
//    );
//    /*error = hipDeviceSynchronize();
//    checkCudaError(error);
//    endStopwatch(timer, "Executing bitonic sort kernel");*/
//}

void runBitonicMergeGlobalKernel(
    data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder
)
{
    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicMergeGlobalKernel<ORDER_ASC><<<dimGrid, dimBlock>>>(dataTable, tableLen, step, step == phase);
    }
    else
    {
        bitonicMergeGlobalKernel<ORDER_DESC><<<dimGrid, dimBlock>>>(dataTable, tableLen, step, step == phase);
    }
}

void runBitoicMergeLocalKernel(data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder)
{
    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicMergeLocalKernel<ORDER_ASC><<<dimGrid, dimBlock>>>(dataTable, tableLen, step, phase == step);
    }
    else
    {
        bitonicMergeLocalKernel<ORDER_DESC><<<dimGrid, dimBlock>>>(dataTable, tableLen, step, phase == step);
    }
}

/*
From all LOCAL samples collects (NUM_SAMPLES) GLOBAL samples.
*/
void runCollectGlobalSamplesKernel(data_t *samples, uint_t samplesLen)
{
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(NUM_SAMPLES, 1, 1);

    collectGlobalSamplesKernel<<<dimGrid, dimBlock>>>(samples, samplesLen);
}

/*
For every sample searches, how many elements in tile are lower than it's value.
*/
void runSampleIndexingKernel(
    data_t *dataTable, data_t *samples, data_t *bucketSizes, uint_t tableLen, uint_t numAllBuckets,
    order_t sortOrder
)
{
    // Number of threads per thread block can be greater than number of samples.
    uint_t elemsPerBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t numBlocks = (tableLen - 1) / elemsPerBitonicSort + 1;
    uint_t threadBlockSize = min(numBlocks * NUM_SAMPLES, THREADS_PER_SAMPLE_INDEXING);

    // Every thread block creates from NUM_SAMPLES samples (NUM_SAMPLES + 1) buckets
    dim3 dimGrid((numAllBuckets - 1) / (threadBlockSize / NUM_SAMPLES * (NUM_SAMPLES + 1)) + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        sampleIndexingKernel<ORDER_ASC><<<dimGrid, dimBlock>>>(dataTable, samples, bucketSizes, tableLen);
    }
    else
    {
        sampleIndexingKernel<ORDER_DESC><<<dimGrid, dimBlock>>>(dataTable, samples, bucketSizes, tableLen);
    }
}

///*
//From local bucket sizes and offsets scatters elements to their global buckets. At the end it coppies
//global bucket sizes (sizes of whole buckets, not just bucket size per tile (local size)) to host.
//*/
//void runBucketsRelocationKernel(el_t *dataTable, el_t *dataBuffer, uint_t *h_globalBucketOffsets,
//                                uint_t *d_globalBucketOffsets, uint_t *localBucketSizes,
//                                uint_t *localBucketOffsets, uint_t tableLen) {
//    // For NUM_SAMPLES samples (NUM_SAMPLES + 1) buckets are created
//    uint_t sharedMemSize = 2 * (NUM_SAMPLES + 1);
//    uint_t elemsPerBitonicSort = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
//    LARGE_INTEGER timer;
//    hipError_t error;
//
//    dim3 dimGrid((tableLen - 1) / elemsPerBitonicSort + 1, 1, 1);
//    dim3 dimBlock(THREADS_PER_BUCKETS_RELOCATION, 1, 1);
//    bucketsRelocationKernel<<<dimGrid, dimBlock, sharedMemSize * sizeof(*localBucketSizes)>>>(
//        dataTable, dataBuffer, d_globalBucketOffsets, localBucketSizes, localBucketOffsets, tableLen
//    );
//
//    /*error = hipDeviceSynchronize();
//    checkCudaError(error);
//    endStopwatch(timer, "Executing kernel for buckets relocation");*/
//
//    error = hipMemcpy(
//        h_globalBucketOffsets, d_globalBucketOffsets, (NUM_SAMPLES + 1) * sizeof(*h_globalBucketOffsets),
//        hipMemcpyDeviceToHost
//    );
//    checkCudaError(error);
//}
//
//void runPrintElemsKernel(el_t *table, uint_t tableLen) {
//    printElemsKernel<<<1, 1>>>(table, tableLen);
//    hipError_t error = hipDeviceSynchronize();
//    checkCudaError(error);
//}
//
//void runPrintDataKernel(data_t *table, uint_t tableLen) {
//    printDataKernel<<<1, 1>>>(table, tableLen);
//    hipError_t error = hipDeviceSynchronize();
//    checkCudaError(error);
//}

/*
Performs global bitonic merge, when number of elements is greater than shared memory size.
*/
void bitonicMerge(data_t *dataTable, uint_t tableLen, uint_t elemsPerBlockBitonicSort, order_t sortOrder)
{
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;
        while (step > phasesMergeLocal)
        {
            runBitonicMergeGlobalKernel(dataTable, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel(dataTable, tableLen, phase, step, sortOrder);
    }
}

///*
//Performs bitonic sort.
//*/
//template <typename T>
//void bitonicSort(T *dataTable, uint_t tableLen, order_t sortOrder) {
//    uint_t tableLenPower2 = nextPowerOf2(tableLen);
//    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
//
//    uint_t phasesAll = log2((double)tableLenPower2);
//
//    runBitonicSortKernel(dataTable, tableLen, sortOrder);
//
//    bitonicMerge<T>(dataTable, tableLen, elemsPerBlockBitonicSort, sortOrder);
//}

// TODO figure out what the bottleneck is
void sampleSort(
    data_t *&dataTable, data_t *&dataBuffer, data_t *samples, uint_t *h_globalBucketOffsets,
    uint_t *d_globalBucketOffsets, uint_t *d_localBucketSizes, uint_t *d_localBucketOffsets, uint_t tableLen,
    order_t sortOrder
)
{
    uint_t elemsPerInitBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t localSamplesDistance = (THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT) / NUM_SAMPLES;
    uint_t localSamplesLen = (tableLen - 1) / localSamplesDistance + 1;
    // (number of all data blocks (tiles)) * (number buckets generated from NUM_SAMPLES)
    uint_t localBucketsLen = ((tableLen - 1) / elemsPerInitBitonicSort + 1) * (NUM_SAMPLES + 1);
    CUDPPHandle scanPlan;

    cudppInitScan(&scanPlan, localBucketsLen);
    runBitonicSortCollectSamplesKernel(dataTable, samples, tableLen, sortOrder);

    // Array has already been sorted
    if (tableLen <= elemsPerInitBitonicSort) {
        data_t *temp = dataTable;
        dataTable = dataBuffer;
        dataBuffer = temp;

        return;
    }

    // Local samples are already partially ordered - NUM_SAMPLES per every tile. These partially ordered
    // samples need to be merged.
    bitonicMerge(samples, localSamplesLen, NUM_SAMPLES, sortOrder);

    // TODO handle case, if all samples are the same
    runCollectGlobalSamplesKernel(samples, localSamplesLen);
    runSampleIndexingKernel(dataTable, samples, d_localBucketSizes, tableLen, localBucketsLen, sortOrder);

    //CUDPPResult result = cudppScan(scanPlan, d_localBucketOffsets, d_localBucketSizes, localBucketsLen);
    //if (result != CUDPP_SUCCESS) {
    //    printf("Error in cudppScan()\n");
    //    getchar();
    //    exit(-1);
    //}

    //runBucketsRelocationKernel(
    //    dataTable, dataBuffer, h_globalBucketOffsets, d_globalBucketOffsets, d_localBucketSizes,
    //    d_localBucketOffsets, tableLen
    //);

    //// Sorts every bucket with bitonic sort
    //uint_t previousOffset = 0;
    //for (uint_t bucket = 0; bucket < NUM_SAMPLES + 1; bucket++) {
    //    uint_t currentOffset = h_globalBucketOffsets[bucket];
    //    uint_t bucketLen = currentOffset - previousOffset;

    //    bitonicSort(dataBuffer, bucketLen, sortOrder);
    //}
}

/*
Sorts input data with parallel sample sort.
*/
double sortParallel(
    data_t *h_output, data_t *d_dataTable, data_t *d_dataBuffer, data_t *d_samples, uint_t *d_localBucketSizes,
    uint_t *d_localBucketOffsets, uint_t *h_globalBucketOffsets, uint_t *d_globalBucketOffsets, uint_t tableLen,
    order_t sortOrder
)
{
    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    sampleSort(
        d_dataTable, d_dataBuffer, d_samples, h_globalBucketOffsets, d_globalBucketOffsets, d_localBucketSizes,
        d_localBucketOffsets, tableLen, sortOrder
    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_output, d_dataBuffer, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
