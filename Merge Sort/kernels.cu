#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "data_types.h"
#include "constants.h"


__device__ void compare(void* val1, void* val2) {
    // TODO
}

__device__ void compare(data_t* elem1, data_t* elem2) {
    // TODO
}

__device__ void printfOnce(char* text) {
    if (threadIdx.x == 0) {
        printf(text);
    }
}

__global__ void bitonicSortKernel(data_t* array, uint_t arrayLen, uint_t sharedMemSize) {
    extern __shared__ data_t tile[];
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    uint_t numStages = ceil(log2((double) sharedMemSize));

    if (index < arrayLen) {
        tile[threadIdx.x] = array[index];
    }
    if (index + blockDim.x < arrayLen) {
        tile[threadIdx.x + blockDim.x] = array[index + blockDim.x];
    }

    for (uint_t stage = 0; stage < numStages; stage++) {
        for (uint_t pass = 0; pass <= stage; pass++) {
            __syncthreads();

            uint_t pairDistance = 1 << (stage - pass);
            uint_t blockWidth = 2 * pairDistance;
            uint_t leftId = (threadIdx.x & (pairDistance - 1)) + (threadIdx.x >> (stage - pass)) * blockWidth;
            uint_t rightId = leftId + pairDistance;

            data_t leftElement, rightElement;
            data_t greater, lesser;
            leftElement = tile[leftId];
            rightElement = tile[rightId];

            uint_t sameDirectionBlockWidth = threadIdx.x >> stage;
            uint_t sameDirection = sameDirectionBlockWidth & 0x1;

            uint_t temp = sameDirection ? rightId : temp;
            rightId = sameDirection ? leftId : rightId;
            leftId = sameDirection ? temp : leftId;

            bool compareResult = (leftElement < rightElement);
            greater = compareResult ? rightElement : leftElement;
            lesser = compareResult ? leftElement : rightElement;

            tile[leftId] = lesser;
            tile[rightId] = greater;
        }
    }

    __syncthreads();

    if (index < arrayLen) {
        array[index] = tile[threadIdx.x];
    }
    if (index + blockDim.x < arrayLen) {
        array[index + blockDim.x] = tile[threadIdx.x + blockDim.x];
    }
}

__device__ uint_t calculateSampleIndex(uint_t tableBlockSize, uint_t tableSubBlockSize, bool firstHalf) {
    // Thread index for first or second half of the sub-table
    uint_t threadIdxX = threadIdx.x + (!firstHalf) * blockDim.x;
    uint_t subBlocksPerBlock = tableBlockSize / tableSubBlockSize;
    // Index of a block from which thread will read the sample
    uint_t indexBlock = threadIdxX / subBlocksPerBlock;
    // Offset to block (we devide and multiply with same value, to lose the offset to 
    // sub-block inside last block)
    uint_t index = indexBlock * subBlocksPerBlock;
    // Offset for sub-block index inside block for ODD block
    index += ((indexBlock % 2 == 0) * threadIdxX) % subBlocksPerBlock;
    // Offset for sub-block index inside block for EVEN block (index has to be reversed)
    index += ((indexBlock % 2 == 1) * (subBlocksPerBlock - (threadIdxX + 1))) % subBlocksPerBlock;

    return index;
}

__device__ uint_t binarySearch(data_t* table, sample_el_t* sampleTile, uint_t tableBlockSize, uint_t tableSubBlockSize, bool firstHalf) {
    uint_t threadIdxX = threadIdx.x + (!firstHalf) * blockDim.x;
    uint_t rank = sampleTile[threadIdxX].rank;
    uint_t sample = sampleTile[threadIdxX].sample;
    uint_t subBlocksPerBlock = tableBlockSize / tableSubBlockSize;
    uint_t subBlocksPerMergedBlock = 2 * subBlocksPerBlock;

    uint_t oppositeBlockOffset = (rank / subBlocksPerMergedBlock) * 2 + !((rank % subBlocksPerMergedBlock) / subBlocksPerBlock);
    uint_t oppositeSubBlockOffset = threadIdxX % subBlocksPerMergedBlock - rank % subBlocksPerBlock - 1;

    // Samples shouldn't be considered
    uint_t indexStart = oppositeBlockOffset * tableBlockSize + oppositeSubBlockOffset * tableSubBlockSize + 1;
    uint_t indexEnd = indexStart + tableSubBlockSize - 2;

    // Has to be explicitly converted to int, because it is unsigned
    if (((int) (indexStart - oppositeBlockOffset * tableBlockSize)) >= 0) {
        while (indexStart < indexEnd) {
            uint_t index = (indexStart + indexEnd) / 2;
            data_t currSample = table[index];

            if (sample < table[index]) {
                indexEnd = index - 1;
            }
            else {
                indexStart = index + 1;
            }
        }

        return indexStart + 1 - oppositeBlockOffset * tableBlockSize;
    }

    return 0;
}

__global__ void generateSublocksKernel(data_t* table, uint_t* rankTable, uint_t tableLen, uint_t tableBlockSize, uint_t tableSubBlockSize) {
    extern __shared__ sample_el_t sampleTile[];
    uint_t sharedMemIdx;
    data_t value;
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x * tableSubBlockSize;
    uint_t subBlocksPerBlock = tableBlockSize / tableSubBlockSize;
    uint_t subBlocksPerMergedBlock = 2 * subBlocksPerBlock;

    // Values are read in coalesced way...
    if (index < tableLen) {
        value = table[index];
    }
    // ...and than reversed when added to shared memory
    sharedMemIdx = calculateSampleIndex(tableBlockSize, tableSubBlockSize, true);
    sampleTile[sharedMemIdx].sample = value;
    sampleTile[threadIdx.x].rank = sharedMemIdx;

    if (threadIdx.x < blockDim.x / 2) {
        for (uint_t stride = subBlocksPerBlock; stride > 0; stride /= 2) {
            __syncthreads();
            uint_t sampleIndex = 2 * threadIdx.x - (threadIdx.x & (stride - 1));

            // TODO use max/min or conditional operator (or something else)
            if (sampleTile[sampleIndex].sample > sampleTile[sampleIndex + stride].sample) {
                sample_el_t temp = sampleTile[sampleIndex];
                sampleTile[sampleIndex] = sampleTile[sampleIndex + stride];
                sampleTile[sampleIndex + stride] = temp;
            }

            if (sampleTile[sampleIndex].sample == sampleTile[sampleIndex + stride].sample && sampleTile[sampleIndex].rank > sampleTile[sampleIndex + stride].rank) {
                sample_el_t temp = sampleTile[sampleIndex];
                sampleTile[sampleIndex] = sampleTile[sampleIndex + stride];
                sampleTile[sampleIndex + stride] = temp;
            }
        }
    }

    // TODO verify if all __syncthreads are needed
    __syncthreads();
    uint_t rank = (sampleTile[threadIdx.x].rank * tableSubBlockSize % tableBlockSize) + 1;
    uint_t oppositeRank = binarySearch(table, sampleTile, tableBlockSize, tableSubBlockSize, true);

    __syncthreads();
    uint_t oddEvenOffset = (sampleTile[threadIdx.x].rank / subBlocksPerBlock) % 2;
    // TODO fix to write in coalesced way
    // TODO comment odd even
    rankTable[threadIdx.x + oddEvenOffset * blockDim.x] = rank;
    rankTable[threadIdx.x + (!oddEvenOffset) * blockDim.x] = oppositeRank;

    /*printf("%2d: %d %d\n", sampleTile[threadIdx.x].sample, rankTable[threadIdx.x], oddEvenOffset);
    __syncthreads();
    printfOnce("\n");
    printf("%2d: %d %d\n", sampleTile[threadIdx.x].sample, rankTable[threadIdx.x + blockDim.x], oddEvenOffset);
    printfOnce("\n\n");*/
}

__global__ void mergeKernel(data_t* dataTable, uint_t* rankTable, uint_t tableLen, uint_t rankTableLen,
                            uint_t tableBlockSize, uint_t tableSubBlockSize) {
    extern __shared__ data_t dataTile[];
    uint_t indexRank = blockIdx.y * tableSubBlockSize + blockIdx.x;
    uint_t dataOffset = blockIdx.y * 2 * tableBlockSize;
    uint_t indexStart1, indexStart2, indexEnd1, indexEnd2;

    // TODO read in coalasced way
    if (blockIdx.x > 0) {
        indexStart1 = rankTable[indexRank - 1];
        indexStart2 = rankTable[(indexRank - 1) + rankTableLen / 2];
    } else {
        indexStart1 = 0;
        indexStart2 = 0;
    }

    if (blockIdx.x < tableSubBlockSize && indexRank < rankTableLen) {
        indexEnd1 = rankTable[indexRank];
        indexEnd2 = rankTable[indexRank + rankTableLen / 2];
    } else {
        indexEnd1 = tableBlockSize;
        indexEnd2 = tableBlockSize;
    }

    /*if (blockIdx.y > 0) {
        printf("%d => odd: (%d, %d), even: (%d, %d)\n", blockIdx.x, indexStart1, indexEnd1, indexStart2, indexEnd2);
    }*/
}
