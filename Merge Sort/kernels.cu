#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "data_types.h"
#include "constants.h"


__global__ void bitonicSortKernel(data_t* array, uint_t arrayLen, uint_t sharedMemSize) {
	extern __shared__ data_t tile[];
	uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
	uint_t numStages = ceil(log2((double) sharedMemSize));

	if (index < arrayLen) {
		tile[threadIdx.x] = array[index];
	}
	if (index + blockDim.x < arrayLen) {
		tile[threadIdx.x + blockDim.x] = array[index + blockDim.x];
	}

	for (uint_t stage = 0; stage < numStages; stage++) {
		for (uint_t pass = 0; pass <= stage; pass++) {
			__syncthreads();

			uint_t pairDistance = 1 << (stage - pass);
			uint_t blockWidth = 2 * pairDistance;
			uint_t leftId = (threadIdx.x & (pairDistance - 1)) + (threadIdx.x >> (stage - pass)) * blockWidth;
			uint_t rightId = leftId + pairDistance;

			data_t leftElement, rightElement;
			data_t greater, lesser;
			leftElement = tile[leftId];
			rightElement = tile[rightId];

			uint_t sameDirectionBlockWidth = threadIdx.x >> stage;
			uint_t sameDirection = sameDirectionBlockWidth & 0x1;

			uint_t temp = sameDirection ? rightId : temp;
			rightId = sameDirection ? leftId : rightId;
			leftId = sameDirection ? temp : leftId;

			bool compareResult = (leftElement < rightElement);
			greater = compareResult ? rightElement : leftElement;
			lesser = compareResult ? leftElement : rightElement;

			tile[leftId] = lesser;
			tile[rightId] = greater;
		}
	}

	__syncthreads();

	if (index < arrayLen) {
		array[index] = tile[threadIdx.x];
	}
	if (index + blockDim.x < arrayLen) {
		array[index + blockDim.x] = tile[threadIdx.x + blockDim.x];
	}
}

__global__ void extractSamples() {
	// TODO
}
