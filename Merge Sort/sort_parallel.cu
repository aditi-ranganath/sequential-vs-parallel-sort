#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Returns the initial size of sorted sub-blocks.

- If table length is lower than max threads per block (every thread loads 2 elements in initial
  bitonic sort kernel), than fewer threads (and steps) are needed for bitonic sort
- If data type used is big (for example double), than only limited ammount of data can be saved
  into shared memory
*/
uint_t getInitSortedBlockSize(uint_t dataElementSizeof, uint_t dataLen) {
    uint_t elementsPerSharedMem = MAX_SHARED_MEM_SIZE / dataElementSizeof;
    uint_t sortedBlockSize = min(min(dataLen, getMaxThreadsPerBlock() * 2), elementsPerSharedMem);
    return sortedBlockSize;
}

/*
Initializes memory needed for parallel implementation of merge sort.
*/
void memoryInit(data_t* inputDataHost, data_t** outputDataHost, data_t** inputDataDevice,
                data_t** outputDataDevice, uint_t** ranksDevice, uint_t dataLen, uint_t ranksLen) {
    hipError_t error;

    // Host memory
    error = hipHostAlloc(outputDataHost, dataLen * sizeof(**outputDataHost), hipHostMallocDefault);
    checkCudaError(error);

    // Device memory
    error = hipMalloc(inputDataDevice, dataLen * sizeof(**inputDataDevice));
    checkCudaError(error);
    error = hipMalloc(outputDataDevice, dataLen * sizeof(**outputDataDevice));
    checkCudaError(error);
    error = hipMalloc(ranksDevice, ranksLen * sizeof(**ranksDevice));
    checkCudaError(error);

    // Memory copy
    error = hipMemcpy(*inputDataDevice, inputDataHost, dataLen * sizeof(**inputDataDevice),
                       hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts data blocks of size sortedBlockSize with bitonic sort.
*/
void runBitonicSortKernel(data_t* data, uint_t dataLen, uint_t sortedBlockSize, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    dim3 dimGrid((dataLen - 1) / sortedBlockSize + 1, 1, 1);
    dim3 dimBlock(sortedBlockSize / 2, 1, 1);  // Every thread loads / sorts 2 elements.

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, sortedBlockSize * sizeof(*data)>>>(
        data, dataLen, sortedBlockSize, orderAsc
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing Bitonic sort Kernel");
}

/*
Generates indexes of sub-blocks that need to be merged.
*/
void runGenerateSublocksKernel(data_t* tableDevice, uint_t* rankTable, uint_t tableLen,
                               uint_t tabBlockSize, uint_t tabSubBlockSize) {
    hipError_t error;
    LARGE_INTEGER timerStart;

    uint_t samplesPerSharedMem = MAX_SHARED_MEM_SIZE / sizeof(sample_el_t);
    uint_t numSamples = tableLen / tabSubBlockSize;
    uint_t threadBlockSize = min(samplesPerSharedMem, numSamples);

    dim3 dimGrid((tableLen - 1) / threadBlockSize + 1, 1, 1);
    dim3 dimBlock(tabSubBlockSize, threadBlockSize / tabSubBlockSize, 1);

    startStopwatch(&timerStart);
    generateSublocksKernel<<<dimGrid, dimBlock, numSamples>>>(
        tableDevice, rankTable, tableLen, tabBlockSize, tabSubBlockSize
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timerStart, "Executing Generate Sublocks kernel");
}

void runMergeKernel(data_t* inputTableDevice, data_t* outputTableDevice, uint_t* rankTable, uint_t tableLen,
                    uint_t rankTableLen, uint_t tabBlockSize, uint_t tabSubBlockSize) {
    hipError_t error;
    LARGE_INTEGER timerStart;

    uint_t subBlocksPerMergedBlock = tabBlockSize / tabSubBlockSize * 2;
    uint_t numMergedBlocks = tableLen / (tabBlockSize * 2);
    uint_t sharedMemSize = tabSubBlockSize * sizeof(*inputTableDevice) * 2;
    dim3 dimGrid(subBlocksPerMergedBlock + 1, numMergedBlocks, 1);
    dim3 dimBlock(tabSubBlockSize, 1, 1);

    startStopwatch(&timerStart);
    mergeKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        inputTableDevice, outputTableDevice, rankTable, tableLen, rankTableLen, tabBlockSize, tabSubBlockSize
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timerStart, "Executing merge kernel");
}

data_t* sortParallel(data_t* inputDataHost, uint_t dataLen, bool orderAsc) {
    data_t* outputDataHost;
    data_t* inputDataDevice;
    data_t* outputDataDevice;
    uint_t* ranksDevice;

    uint_t sortedBlockSize = getInitSortedBlockSize(sizeof(*inputDataDevice), dataLen);
    uint_t mergedBlockSize = sortedBlockSize / 2;
    uint_t ranksLen = (dataLen / mergedBlockSize) * 2;
    hipError_t error;

    memoryInit(inputDataHost, &outputDataHost, &inputDataDevice, &outputDataDevice,
               &ranksDevice, dataLen, ranksLen);
    runBitonicSortKernel(inputDataDevice, dataLen, sortedBlockSize, orderAsc);

    /*runGenerateSublocksKernel(inputDataDevice, ranksDevice, dataLen, sortedBlockSize, mergedBlockSize);
    error = hipDeviceSynchronize();*/

    /*
    // TODO verify, if ALL (also up) device syncs are necessary
    for (; tableBlockSize < dataLen; tableBlockSize *= 2) {
        runGenerateSublocksKernel(inputDataDevice, ranksDevice, dataLen, tableBlockSize, tableSubBlockSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

        runMergeKernel(inputDataDevice, outputDataDevice, ranksDevice, dataLen, ranksLen,
                       tableBlockSize, tableSubBlockSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

        data_t* temp = inputDataDevice;
        inputDataDevice = outputDataDevice;
        outputDataDevice = temp;
    }
    */

    error = hipMemcpy(outputDataHost, inputDataDevice, dataLen * sizeof(*outputDataHost),
                       hipMemcpyDeviceToHost);
    checkCudaError(error);

    return outputDataHost;
}
