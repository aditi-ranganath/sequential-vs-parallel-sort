#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Returns the initial size of sorted sub-blocks.

- If table length is lower than max threads per block (every thread loads 2 elements in initial
  bitonic sort kernel), than fewer threads (and steps) are needed for bitonic sort
- If data type used is big (for example double), than only limited ammount of data can be saved
  into shared memory
*/
uint_t getInitSortedBlockSize(uint_t dataElementSizeof, uint_t dataLen) {
    uint_t elementsPerSharedMem = MAX_SHARED_MEM_SIZE / dataElementSizeof;
    uint_t sortedBlockSize = min(min(dataLen, getMaxThreadsPerBlock() * 2), elementsPerSharedMem);
    return sortedBlockSize;
}

/*
Initializes memory needed for parallel implementation of merge sort.
*/
void memoryInit(data_t* inputDataHost, data_t** outputDataHost, data_t** inputDataDevice,
                data_t** outputDataDevice, uint_t** ranksDevice, uint_t dataLen, uint_t ranksLen) {
    hipError_t error;

    // Host memory
    error = hipHostAlloc(outputDataHost, dataLen * sizeof(**outputDataHost), hipHostMallocDefault);
    checkCudaError(error);

    // Device memory
    error = hipMalloc(inputDataDevice, dataLen * sizeof(**inputDataDevice));
    checkCudaError(error);
    error = hipMalloc(outputDataDevice, dataLen * sizeof(**outputDataDevice));
    checkCudaError(error);
    error = hipMalloc(ranksDevice, ranksLen * sizeof(**ranksDevice));
    checkCudaError(error);

    // Memory copy
    error = hipMemcpy(*inputDataDevice, inputDataHost, dataLen * sizeof(**inputDataDevice),
                       hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts data blocks of size sortedBlockSize with bitonic sort.
*/
void runBitonicSortKernel(data_t* data, uint_t dataLen, uint_t sortedBlockSize, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    dim3 dimGrid((dataLen - 1) / sortedBlockSize + 1, 1, 1);
    dim3 dimBlock(sortedBlockSize / 2, 1, 1);  // Every thread loads / sorts 2 elements.

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, sortedBlockSize * sizeof(*data)>>>(
        data, dataLen, sortedBlockSize, orderAsc
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing Bitonic sort Kernel");
}

/*
Generates ranks of sub-blocks that need to be merged.
*/
void runGenerateRanksKernel(data_t* data, uint_t* ranks, uint_t dataLen, uint_t sortedBlockSize,
                            uint_t subBlockSize) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t ranksPerSharedMem = MAX_SHARED_MEM_SIZE / sizeof(sample_el_t);
    uint_t numAllRanks = dataLen / subBlockSize;
    uint_t threadBlockSize = min(ranksPerSharedMem, numAllRanks);

    dim3 dimGrid((numAllRanks - 1) / threadBlockSize + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    generateRanksKernel<<<dimGrid, dimBlock, threadBlockSize * sizeof(sample_el_t)>>>(
        data, ranks, dataLen, sortedBlockSize, subBlockSize
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing Generate ranks kernel");
}

/*
Executes merge kernel, which merges all consecutive sorted blocks in data.
*/
void runMergeKernel(data_t* inputData, data_t* outputData, uint_t* ranks, uint_t dataLen,
                    uint_t ranksLen, uint_t sortedBlockSize, uint_t tabSubBlockSize) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t subBlocksPerMergedBlock = sortedBlockSize / tabSubBlockSize * 2;
    uint_t numMergedBlocks = dataLen / (sortedBlockSize * 2);
    uint_t sharedMemSize = tabSubBlockSize * sizeof(*inputData) * 2;
    dim3 dimGrid(subBlocksPerMergedBlock + 1, numMergedBlocks, 1);
    dim3 dimBlock(tabSubBlockSize, 1, 1);

    startStopwatch(&timer);
    mergeKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        inputData, outputData, ranks, dataLen, ranksLen, sortedBlockSize, tabSubBlockSize
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing merge kernel");
}

data_t* sortParallel(data_t* inputDataHost, uint_t dataLen, bool orderAsc) {
    data_t* outputDataHost;
    data_t* inputDataDevice;
    data_t* outputDataDevice;
    uint_t* ranksDevice;

    uint_t sortedBlockSize = getInitSortedBlockSize(sizeof(*inputDataDevice), dataLen);
    uint_t subBlockSize = sortedBlockSize / 2;
    uint_t ranksLen = (dataLen / subBlockSize) * 2;
    hipError_t error;

    memoryInit(inputDataHost, &outputDataHost, &inputDataDevice, &outputDataDevice,
               &ranksDevice, dataLen, ranksLen);
    runBitonicSortKernel(inputDataDevice, dataLen, sortedBlockSize, orderAsc);

    // TODO verify, if ALL (also up) device syncs are necessary
    for (; sortedBlockSize < dataLen; sortedBlockSize *= 2) {
        runGenerateRanksKernel(inputDataDevice, ranksDevice, dataLen, sortedBlockSize, subBlockSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

        runMergeKernel(inputDataDevice, outputDataDevice, ranksDevice, dataLen, ranksLen,
                       sortedBlockSize, subBlockSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

        data_t* temp = inputDataDevice;
        inputDataDevice = outputDataDevice;
        outputDataDevice = temp;
    }

    error = hipMemcpy(outputDataHost, inputDataDevice, dataLen * sizeof(*outputDataHost),
                       hipMemcpyDeviceToHost);
    checkCudaError(error);

    return outputDataHost;
}
