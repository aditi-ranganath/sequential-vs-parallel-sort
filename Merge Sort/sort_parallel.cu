#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes memory needed for parallel implementation of merge sort.
*/
void memoryInit(el_t *h_input, el_t **d_input, el_t **d_output, el_t **d_buffer, uint_t **d_ranks,
                uint_t tableLen, uint_t ranksLen) {
    hipError_t error;

    error = hipMalloc(d_input, tableLen * sizeof(**d_input));
    checkCudaError(error);
    error = hipMalloc(d_output, tableLen * sizeof(**d_output));
    checkCudaError(error);
    error = hipMalloc(d_buffer, tableLen * sizeof(**d_buffer));
    checkCudaError(error);
    error = hipMalloc(d_ranks, ranksLen * sizeof(**d_ranks));
    checkCudaError(error);

    error = hipMemcpy(*d_input, h_input, tableLen * sizeof(**d_input), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts data blocks of size sortedBlockSize with merge sort.
*/
void runMergeSortKernel(el_t *input, el_t *output, uint_t tableLen, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t threadBlockSize = SHARED_MEM_SIZE / 2;
    dim3 dimGrid((tableLen - 1) / (threadBlockSize * 2) + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    mergeSortKernel<<<dimGrid, dimBlock>>>(input, output, orderAsc);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing Bitonic sort Kernel")*/;
}

/*
Generates ranks of sub-blocks that need to be merged.
*/
void runGenerateRanksKernel(el_t *table, uint_t *ranks, uint_t tableLen, uint_t sortedBlockSize) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t numAllSamples = tableLen / SUB_BLOCK_SIZE;
    uint_t threadBlockSize = min(numAllSamples, SHARED_MEM_SIZE);
    dim3 dimGrid((numAllSamples - 1) / threadBlockSize + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    generateRanksKernel<<<dimGrid, dimBlock>>>(table, ranks, tableLen, sortedBlockSize);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing Generate ranks kernel");*/
}

/*
Executes merge kernel, which merges all consecutive sorted blocks in data.
*/
void runMergeKernel(el_t *input, el_t *output, uint_t *ranks, uint_t tableLen,
                    uint_t ranksLen, uint_t sortedBlockSize) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t subBlocksPerMergedBlock = sortedBlockSize / SUB_BLOCK_SIZE * 2;
    uint_t numMergedBlocks = tableLen / (sortedBlockSize * 2);
    dim3 dimGrid(subBlocksPerMergedBlock + 1, numMergedBlocks, 1);
    dim3 dimBlock(SUB_BLOCK_SIZE, 1, 1);

    startStopwatch(&timer);
    mergeKernel << <dimGrid, dimBlock>> >(
        input, output, ranks, tableLen, ranksLen, sortedBlockSize, SUB_BLOCK_SIZE
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing merge kernel");*/
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    el_t *d_input, *d_output, *d_buffer;
    uint_t* d_ranks;
    uint_t ranksLen = tableLen / SUB_BLOCK_SIZE * 2;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_input, &d_input, &d_output, &d_buffer, &d_ranks, tableLen, ranksLen);

    startStopwatch(&timer);
    runMergeSortKernel(d_input, d_output, tableLen, orderAsc);

    for (uint_t sortedBlockSize = SHARED_MEM_SIZE; sortedBlockSize < tableLen; sortedBlockSize *= 2) {
        el_t* temp = d_output;
        d_output = d_buffer;
        d_buffer = temp;

        runGenerateRanksKernel(d_buffer, d_ranks, tableLen, sortedBlockSize);
        runMergeKernel(d_buffer, d_output, d_ranks, tableLen, ranksLen, sortedBlockSize);
    }
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing parallel merge sort.");

    error = hipMemcpy(h_output, d_output, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);
}
