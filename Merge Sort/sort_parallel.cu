#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes memory needed for parallel implementation of merge sort.
*/
void memoryInit(data_t* inputDataHost, data_t** outputDataHost, data_t** inputDataDevice,
                data_t** outputDataDevice, uint_t** ranksDevice, uint_t dataLen, uint_t ranksLen) {
    hipError_t error;

    // Host memory
    error = hipHostAlloc(outputDataHost, dataLen * sizeof(**outputDataHost), hipHostMallocDefault);
    checkCudaError(error);

    // Device memory
    error = hipMalloc(inputDataDevice, dataLen * sizeof(**inputDataDevice));
    checkCudaError(error);
    error = hipMalloc(outputDataDevice, dataLen * sizeof(**outputDataDevice));
    checkCudaError(error);
    error = hipMalloc(ranksDevice, ranksLen * sizeof(**ranksDevice));
    checkCudaError(error);

    // Memory copy
    error = hipMemcpy(*inputDataDevice, inputDataHost, dataLen * sizeof(**inputDataDevice),
                       hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts blocks od data table with bitonic sort. Returns the size of one sorted block.
*/
uint_t runBitonicSortKernel(data_t* data, uint_t dataLen, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elementsPerSharedMem = MAX_SHARED_MEM_SIZE / sizeof(*data);
    // If table length is lower than max threads per block, than fewer steps are needed for bitonic sort
    // If data type used is big, than only limited ammount of data can be saved in shared memory
    uint_t threadBlockSize = min(min(dataLen / 2, getMaxThreadsPerBlock()), elementsPerSharedMem / 2);
    uint_t sortedBlockSize = 2 * threadBlockSize; // Every thread compares/sorts 2 elements

    dim3 dimGrid((dataLen - 1) / sortedBlockSize + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, sortedBlockSize * sizeof(*data)>>>(
        data, dataLen, sortedBlockSize, orderAsc
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing Bitonic sort Kernel");

    return sortedBlockSize;
}

void runGenerateSublocksKernel(data_t* tableDevice, uint_t* rankTable, uint_t tableLen,
                               uint_t tabBlockSize, uint_t tabSubBlockSize) {
    hipError_t error;
    LARGE_INTEGER timerStart;

    // * 2 for table of ranks, which has the same size as table of samples
    uint_t sharedMemSize = tableLen / tabSubBlockSize * sizeof(sample_el_t);
    uint_t blockSize = tableLen / tabSubBlockSize;
    dim3 dimGrid((tableLen - 1) / (2 * blockSize * tabSubBlockSize) + 1, 1, 1);
    dim3 dimBlock(blockSize, 1, 1);

    startStopwatch(&timerStart);
    generateSublocksKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        tableDevice, rankTable, tableLen, tabBlockSize, tabSubBlockSize
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timerStart, "Executing Generate Sublocks kernel");
}

void runMergeKernel(data_t* inputTableDevice, data_t* outputTableDevice, uint_t* rankTable, uint_t tableLen,
                    uint_t rankTableLen, uint_t tabBlockSize, uint_t tabSubBlockSize) {
    hipError_t error;
    LARGE_INTEGER timerStart;

    uint_t subBlocksPerMergedBlock = tabBlockSize / tabSubBlockSize * 2;
    uint_t numMergedBlocks = tableLen / (tabBlockSize * 2);
    uint_t sharedMemSize = tabSubBlockSize * sizeof(*inputTableDevice) * 2;
    dim3 dimGrid(subBlocksPerMergedBlock + 1, numMergedBlocks, 1);
    dim3 dimBlock(tabSubBlockSize, 1, 1);

    startStopwatch(&timerStart);
    mergeKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        inputTableDevice, outputTableDevice, rankTable, tableLen, rankTableLen, tabBlockSize, tabSubBlockSize
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timerStart, "Executing merge kernel");
}

data_t* sortParallel(data_t* inputDataHost, uint_t dataLen, bool orderAsc) {
    data_t* outputDataHost;
    data_t* inputDataDevice;
    data_t* outputDataDevice;
    uint_t* ranksDevice;
    uint_t tableBlockSize = 8;
    uint_t tableSubBlockSize = 4;  // TODO could be constant
    uint_t ranksLen = dataLen / tableSubBlockSize * 2;
    hipError_t error;

    memoryInit(inputDataHost, &outputDataHost, &inputDataDevice, &outputDataDevice,
               &ranksDevice, dataLen, ranksLen);
    runBitonicSortKernel(inputDataDevice, dataLen, orderAsc);

    /*
    // TODO verify, if ALL (also up) device syncs are necessary
    for (; tableBlockSize < dataLen; tableBlockSize *= 2) {
        runGenerateSublocksKernel(inputDataDevice, ranksDevice, dataLen, tableBlockSize, tableSubBlockSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

        runMergeKernel(inputDataDevice, outputDataDevice, ranksDevice, dataLen, ranksLen,
                       tableBlockSize, tableSubBlockSize);
        error = hipDeviceSynchronize();
        checkCudaError(error);

        data_t* temp = inputDataDevice;
        inputDataDevice = outputDataDevice;
        outputDataDevice = temp;
    }
    */

    error = hipMemcpy(outputDataHost, inputDataDevice, dataLen * sizeof(*outputDataHost),
                       hipMemcpyDeviceToHost);
    checkCudaError(error);

    return outputDataHost;
}
