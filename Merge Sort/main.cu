#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "sort_parallel.h"
#include "sort_sequential.h"


int main(int argc, char** argv) {
    // Rename array to table everywhere in code
    /*data_t inputData[32] = {
        6, 23, 29, 35, 45, 63, 64, 97, 1, 4, 25, 34, 45, 67, 98, 99, 4, 19, 41, 58,
        68, 80, 81, 96, 4, 13, 18, 33, 55, 66, 88, 90
    };*/
    /*data_t inputData[64] = {
        60, 39, 36, 61, 40, 41, 62, 54, 42, 64, 81, 70, 55, 5, 99, 22, 49, 95, 18, 19,
        73, 84, 90, 16, 50, 22, 1, 60, 6, 74, 58, 18, 43, 64, 18, 86, 33, 81, 92, 42,
        14, 81, 34, 37, 43, 29, 12, 30, 81, 41, 21, 8, 82, 45, 40, 25, 96, 85, 25, 32,
        90, 88, 20, 28
    };*/
    data_t* inputData;
    data_t* outputDataParallel;
    data_t* outputDataSequential;
    data_t* outputDataCorrect;

    uint_t dataLen = 1 << 20;
    bool orderAsc = true;  // TODO use this
    hipError_t error;

    LARGE_INTEGER timerStart;

    // TODO remove bottom comment when tested
    //hipFuncCachePreferNone, hipFuncCachePreferShared, hipFuncCachePreferL1, hipFuncCachePreferEqual
    error = hipDeviceSetCacheConfig(hipFuncCachePreferL1);
    checkCudaError(error);
    hipFree(NULL);  // Initializes CUDA, because CUDA init is lazy
    srand(time(NULL));

    error = hipHostAlloc(&inputData, dataLen * sizeof(*inputData), hipHostMallocDefault);
    checkCudaError(error);

    for (int i = 0; i < 3; i++) {
        fillArrayRand(inputData, dataLen);
        //fillArrayValue(inputData, dataLen, 5);
        //printArray(inputData, dataLen);

        outputDataParallel = sortParallel(inputData, dataLen, orderAsc);
        //printArray(outputDataParallel, dataLen);

        outputDataCorrect = sortCorrect(inputData, dataLen);
        compareArrays(outputDataParallel, outputDataCorrect, dataLen);
    }

    //hipHostFree(inputData);
    hipHostFree(outputDataParallel);
    //free(outputDataSequential);
    free(outputDataCorrect);

    getchar();
    return 0;
}
