#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "sort_parallel.h"


int comparator(const void * elem1, const void * elem2) {
    return (*(data_t*)elem1 - *(data_t*)elem2);
}

int main(int argc, char** argv) {
    // Rename array to table everywhere in code
    data_t input[32] = { 6, 23, 29, 35, 45, 63, 64, 97, 1, 4, 25, 34, 45, 67, 98, 99, 4, 19, 41, 58, 68, 80, 81, 96, 4, 13, 18, 33, 55, 66, 88, 90 };;
    //data_t* input;
    data_t* outputParallel;
    data_t* outputSequential;
    data_t* correctlySorted;

    uint_t tableLen = 1 << 5;
    uint_t blockSize;
    bool orderAsc = TRUE;
    hipError_t error;

    LARGE_INTEGER timerStart;

    hipFree(NULL);  // Initializes CUDA, because CUDA init is lazy
    srand(time(NULL));

    //error = hipHostAlloc(&input, tableLen * sizeof(*input), hipHostMallocDefault);
    //checkCudaError(error);
    error = hipHostAlloc(&outputParallel, tableLen * sizeof(*outputParallel), hipHostMallocDefault);
    checkCudaError(error);
    // fillArrayRand(input, arrayLen);
    //fillArrayValue(input, tableLen, 5);

    sortParallel(input, outputParallel, tableLen, orderAsc);
    printArray(outputParallel, tableLen);

    // TODO free memory

    getchar();
    return 0;
}
