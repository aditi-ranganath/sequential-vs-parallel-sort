#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "sort_parallel.h"


int comparator(const void * elem1, const void * elem2) {
	return (*(data_t*)elem1 - *(data_t*)elem2);
}

int main(int argc, char** argv) {
	// Rename array to table everywhere in code
	data_t* input;
	data_t* outputParallel;
	data_t* outputSequential;
	data_t* correctlySorted;

	uint_t arrayLen = 1 << 5;
	uint_t blockSize;
	bool orderAsc = TRUE;
	hipError_t error;

	LARGE_INTEGER timerStart;

	hipFree(NULL);  // Initializes CUDA, because CUDA init is lazy
	srand(time(NULL));

	error = hipHostAlloc(&input, arrayLen * sizeof(*input), hipHostMallocDefault);
	checkCudaError(error);
	error = hipHostAlloc(&outputParallel, arrayLen * sizeof(*outputParallel), hipHostMallocDefault);
	checkCudaError(error);
	fillArrayRand(input, arrayLen);

	sortParallel(input, outputParallel, arrayLen, orderAsc);
	printArray(outputParallel, arrayLen);

	getchar();
	return 0;
}
