#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "sort_parallel.h"


int comparator(const void * elem1, const void * elem2) {
	return (*(data_t*)elem1 - *(data_t*)elem2);
}

int main(int argc, char** argv) {
	data_t* input;
	data_t* outputParallel;
	data_t* outputSequential;
	data_t* correctlySorted;

	uint_t arrayLen = 1 << 5;
	uint_t blockSize;
	bool orderAsc = TRUE;
	hipError_t error;

	LARGE_INTEGER timerStart;

	hipFree(NULL);  // Initializes CUDA, because CUDA init is lazy
	srand(time(NULL));

	error = hipHostAlloc(&input, arrayLen * sizeof(*input), hipHostMallocDefault);
	checkCudaError(error);
	error = hipHostAlloc(&outputParallel, arrayLen * sizeof(*outputParallel), hipHostMallocDefault);
	checkCudaError(error);
	fillArrayRand(input, arrayLen);

	blockSize = sortParallel(input, outputParallel, arrayLen, orderAsc);

	startStopwatch(&timerStart);
	correctlySorted = copyArray(input, arrayLen);  // TODO Use some C implementation of copy array
	for (int i = 0; i < arrayLen / blockSize; i++) {
		qsort(correctlySorted + i * blockSize, blockSize, sizeof(*correctlySorted), comparator);
	}
	endStopwatch(timerStart, "Sequential sort");

	printf("\n\n");
	compareArrays(outputParallel, correctlySorted, arrayLen);

	printArray(outputParallel, arrayLen);

	getchar();
	return 0;
}
