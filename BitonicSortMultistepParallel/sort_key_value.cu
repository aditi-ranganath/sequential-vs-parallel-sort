#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/host.h"
#include "../Utils/hip/hip_runtime.h"
#include "constants.h"
#include "kernels_key_value.h"
#include "sort.h"


/*
Runs bitonic multistep merge kernel, which uses registers. Multistep means, that every thread reads
multiple elements and sorts them according to bitonic sort exchanges for N steps ahead.
*/
template <order_t sortOrder, uint_t threadsMerge>
void BitonicSortMultistepParallel::runMultiStepKernel(
    data_t *d_keys, data_t *d_values, uint_t arrayLength, uint_t phase, uint_t step, uint_t degree
)
{
    // Breaks table len into its power of 2 length and the remainder.
    uint_t power2arrayLen = previousPowerOf2(arrayLength);
    uint_t residueArrayLen = arrayLength % power2arrayLen;

    uint_t partitionSize = (power2arrayLen - 1) / (1 << degree) + 1;
    // For remainder the size of partition has to be calculated explicitly, becaause it depends on
    // remainder size, step and degree
    if (residueArrayLen > 0)
    {
        // The size of one sub-block which is sorted with same group of comparissons.
        uint_t subBlockSize = 1 << step;
        // Rouns the residue size to the next power of sub-block size
        uint_t power2residueArrayLen = roundUp(residueArrayLen, subBlockSize);
        partitionSize += min(residueArrayLen, (power2residueArrayLen - 1) / (1 << degree) + 1);
    }

    uint_t threadBlockSize = min(partitionSize, threadsMerge);
    dim3 dimGrid((partitionSize - 1) / threadBlockSize + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (degree == 1)
    {
        multiStep1Kernel<sortOrder><<<dimGrid, dimBlock>>>(d_keys, d_values, arrayLength, step);
    }
    else if (degree == 2)
    {
        multiStep2Kernel<sortOrder><<<dimGrid, dimBlock>>>(d_keys, d_values, arrayLength, step);
    }
    else if (degree == 3)
    {
        multiStep3Kernel<sortOrder><<<dimGrid, dimBlock>>>(d_keys, d_values, arrayLength, step);
    }
    else if (degree == 4)
    {
        multiStep4Kernel<sortOrder><<<dimGrid, dimBlock>>>(d_keys, d_values, arrayLength, step);
    }
    else if (degree == 5)
    {
        multiStep5Kernel<sortOrder><<<dimGrid, dimBlock>>>(d_keys, d_values, arrayLength, step);
    }
}

/*
Sorts data with NORMALIZED MULTISTEP BITONIC SORT.
*/
template <order_t sortOrder>
void BitonicSortMultistepParallel::bitonicSortMultistepParallel(data_t *d_keys, data_t *d_values, uint_t arrayLength)
{
    uint_t arrayLengthPower2 = nextPowerOf2(arrayLength);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT_KV * ELEMS_PER_THREAD_BITONIC_SORT_KV;
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE_KV * ELEMS_PER_THREAD_LOCAL_MERGE_KV;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(arrayLengthPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(arrayLengthPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)arrayLengthPower2);

    runBitoicSortKernel<sortOrder, THREADS_PER_BITONIC_SORT_KV, ELEMS_PER_THREAD_BITONIC_SORT_KV>(
        d_keys, d_values, arrayLength
    );

    // Bitonic merge
    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;

        if (step > phasesMergeLocal)
        {
            // Global NORMALIZED bitonic merge for first step of phase, where different pattern of exchanges
            // is used compared to other steps
            runBitonicMergeGlobalKernel<sortOrder, THREADS_PER_GLOBAL_MERGE_KV, ELEMS_PER_THREAD_GLOBAL_MERGE_KV>(
                d_keys, d_values, arrayLength, phase, step
            );
            step--;

            // Multisteps
            for (uint_t degree = min(MAX_MULTI_STEP_KV, step - phasesMergeLocal); degree > 0; degree--)
            {
                for (; step >= phasesMergeLocal + degree; step -= degree)
                {
                    runMultiStepKernel<sortOrder, THREADS_PER_MULTISTEP_MERGE_KV>(
                        d_keys, d_values, arrayLength, phase, step, degree
                    );
                }
            }
        }

        runBitoicMergeLocalKernel<sortOrder, THREADS_PER_LOCAL_MERGE_KV, ELEMS_PER_THREAD_LOCAL_MERGE_KV>(
            d_keys, d_values, arrayLength, phase, step
        );
    }
}


/*
Wrapper for bitonic sort method.
The code runs faster if arguments are passed to method. If members are accessed directly, code runs slower.
*/
void BitonicSortMultistepParallel::sortKeyValue()
{
    if (_sortOrder == ORDER_ASC)
    {
        bitonicSortMultistepParallel<ORDER_ASC>(_d_keys, _d_values, _arrayLength);
    }
    else
    {
        bitonicSortMultistepParallel<ORDER_DESC>(_d_keys, _d_values, _arrayLength);
    }
}
