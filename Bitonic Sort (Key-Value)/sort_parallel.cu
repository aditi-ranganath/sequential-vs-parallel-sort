#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(data_t *keys, data_t *values, uint_t tableLen, order_t sortOrder) {
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = 2 * elemsPerThreadBlock * sizeof(*keys);  // "2 *" becaues of key-value pairs

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(keys, values, tableLen);
    }
    else
    {
        bitonicSortKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(keys, values, tableLen);
    }
}

/*
Merges array, if data blocks are larger than shared memory size. It executes only of STEP on PHASE per
kernel launch.
*/
void runBitonicMergeGlobalKernel(
    data_t *keys, data_t *values, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder
)
{
    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_ASC, true><<<dimGrid, dimBlock>>>(keys, values, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_ASC, false><<<dimGrid, dimBlock>>>(keys, values, tableLen, step);
        }
    }
    else
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_DESC, true><<<dimGrid, dimBlock>>>(keys, values, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_DESC, false><<<dimGrid, dimBlock>>>(keys, values, tableLen, step);
        }
    }
}

/*
Merges array when stride is lower than shared memory size. It executes all remaining STEPS of current PHASE.
*/
void runBitoicMergeLocalKernel(data_t *keys, data_t *values, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder)
{
    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    uint_t sharedMemSize = 2 * elemsPerThreadBlock * sizeof(*keys);  // "2 *" becaues of key-value pairs
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase) {
            bitonicMergeLocalKernel<ORDER_ASC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                keys, values, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_ASC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                keys, values, tableLen, step
            );
        }
    }
    else
    {
        if (isFirstStepOfPhase) {
            bitonicMergeLocalKernel<ORDER_DESC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                keys, values, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_DESC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                keys, values, tableLen, step
            );
        }
    }
}

/*
Sorts data with NORMALIZED BITONIC SORT.
*/
double sortParallel(
    data_t *h_keys, data_t *h_values, data_t *d_keys, data_t *d_values, uint_t tableLen, order_t sortOrder
)
{
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    runBitoicSortKernel(d_keys, d_values, tableLen, sortOrder);

    // Bitonic merge
    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;
        while (step > phasesMergeLocal)
        {
            runBitonicMergeGlobalKernel(d_keys, d_values, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel(d_keys, d_values, tableLen, phase, step, sortOrder);
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_keys, d_keys, tableLen * sizeof(*h_keys), hipMemcpyDeviceToHost);
    checkCudaError(error);
    error = hipMemcpy(h_values, d_values, tableLen * sizeof(*h_values), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
