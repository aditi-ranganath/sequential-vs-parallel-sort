#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <array>
#include <vector>
#include <memory>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/host.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/sort_interface.h"

#include "../BitonicSort/sort_sequential.h"
#include "../BitonicSort/sort_parallel.h"
#include "../BitonicSortMultistep/sort_parallel.h"
#include "../BitonicSortAdaptive/sort_sequential.h"
#include "../BitonicSortAdaptive/sort_parallel.h"

#include "test_sort.h"


int main(int argc, char **argv)
{
    uint_t arrayLenStart = (1 << 20);
    uint_t arrayLenEnd = arrayLenStart;
    uint_t interval = MAX_VAL;
    uint_t testRepetitions = 3;    // How many times are sorts ran
    order_t sortOrder = ORDER_ASC;  // Values: ORDER_ASC, ORDER_DESC

    // Input data distributions
    std::vector<data_dist_t> distributions;
    distributions.push_back(DISTRIBUTION_UNIFORM);

    // Sorting algorithms
    std::vector<SortSequential*> sorts;
    sorts.push_back(new BitonicSortSequential());
    sorts.push_back(new BitonicSortParallel());
    sorts.push_back(new BitonicSortMultistepParallel());
    sorts.push_back(new BitonicSortAdaptiveSequential());
    sorts.push_back(new BitonicSortAdaptiveParallel());
    //sorts.push_back(new QuicksortSequential());
    //sorts.push_back(new QuicksortParallel());

    // This is needed only for testing puproses, because data transfer from device to host shouldn't be stopwatched.
    for (std::vector<SortSequential*>::iterator sort = sorts.begin(); sort != sorts.end(); sort++)
    {
        (*sort)->stopwatchEnable();
    }

    generateStatistics(sorts, distributions, arrayLenStart, arrayLenEnd, sortOrder, testRepetitions, interval);

    printf("Finished\n");
    getchar();
    return 0;
}
