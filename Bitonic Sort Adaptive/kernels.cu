#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "../Utils/data_types_common.h"
#include "constants.h"
#include "data_types.h"


/*---------------------------------------------------------
-------------------------- UTILS --------------------------
-----------------------------------------------------------*/

/*
Compares 2 elements and exchanges them according to sortOrder.
*/
template <order_t sortOrder>
__device__ void compareExchange(data_t *elem1, data_t *elem2)
{
    if ((*elem1 > *elem2) ^ sortOrder)
    {
        data_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
From provided interval and index returns element in table. Index can't be higher than interval span.
*/
__device__ data_t getTableElement(data_t *table, interval_t interval, uint_t index)
{
    bool useInterval1 = index >= interval.length0;
    uint_t offset = useInterval1 ? interval.offset1 : interval.offset0;

    index -= useInterval1 ? interval.length0 : 0;
    index -= useInterval1 && index >= interval.length1 ? interval.length1 : 0;

    return table[offset + index];
}

/*
Finds the index q, which is and index, where the exchanges in the bitonic sequence begin. All
elements after index q have to be exchanged. Bitonic sequence boundaries are provided with interval.

Example: 2, 3, 5, 7 | 8, 7, 3, 1 --> index q = 2 ; (5, 7 and 3, 1 have to be exchanged).
*/
template <order_t sortOrder>
__device__ int_t binarySearch(data_t* table, interval_t interval, uint_t subBlockHalfLen)
{
    // Depending which interval is longer, different start and end indexes are used
    int_t indexStart = interval.length0 <= interval.length1 ? 0 : subBlockHalfLen - interval.length1;
    int_t indexEnd = interval.length0 <= interval.length1 ? interval.length0 : subBlockHalfLen;

    while (indexStart < indexEnd)
    {
        int index = indexStart + (indexEnd - indexStart) / 2;
        data_t el0 = getTableElement(table, interval, index);
        data_t el1 = getTableElement(table, interval, index + subBlockHalfLen);

        if ((el0 > el1) ^ sortOrder)
        {
            indexStart = index + 1;
        }
        else
        {
            indexEnd = index;
        }
    }

    return indexStart;
}

/*
Generates intervals in provided table until size of sub block is grater than end sub block size.
Sub block size is the size of one block in bitonic merge step.
*/
template <order_t sortOrder>
__device__ void generateIntervals(
    data_t *table, uint_t subBlockSize, uint_t subBlockSizeEnd, uint_t stride, uint_t activeThreadsPerBlock,
    uint_t elemsPerThread
)
{
    extern __shared__ interval_t intervalsTile[];
    interval_t *intervals = intervalsTile;
    interval_t *intervalsBuffer = intervalsTile + blockDim.x * elemsPerThread;
    interval_t interval;

    for (; subBlockSize > subBlockSizeEnd; subBlockSize /= 2, stride *= 2, activeThreadsPerBlock *= 2)
    {
        for (uint_t tx = threadIdx.x; tx < activeThreadsPerBlock; tx += blockDim.x)
        {
            interval = intervals[tx];

            uint_t intervalIndex = blockIdx.x * activeThreadsPerBlock + tx;
            bool orderAsc = sortOrder ^ ((intervalIndex / stride) & 1);
            uint_t q;

            if (orderAsc)
            {
                q = binarySearch<ORDER_ASC>(table, interval, subBlockSize / 2);
            }
            else
            {
                q = binarySearch<ORDER_DESC>(table, interval, subBlockSize / 2);
            }

            uint_t index1 = 2 * tx;
            uint_t index2 = index1 + 1;

            // Left sub-block
            intervalsBuffer[index1].offset0 = interval.offset0;
            intervalsBuffer[index1].length0 = q;
            intervalsBuffer[index1].offset1 = interval.offset1 + interval.length1 - subBlockSize / 2 + q;
            intervalsBuffer[index1].length1 = subBlockSize / 2 - q;

            // Right sub-block. Intervals are reversed.
            intervalsBuffer[index2].offset0 = interval.offset0 + q;
            intervalsBuffer[index2].length0 = interval.length0 - q;
            intervalsBuffer[index2].offset1 = interval.offset1;
            intervalsBuffer[index2].length1 = q + interval.length1 - subBlockSize / 2;
        }

        interval_t *temp = intervals;
        intervals = intervalsBuffer;
        intervalsBuffer = temp;

        __syncthreads();
    }
}


/*---------------------------------------------------------
------------------------- KERNELS -------------------------
-----------------------------------------------------------*/

/*
Adds the padding to table from start index (original table length, which is not power of 2) to the end of the
extended array (which is the next power of 2 of the original table length). Needed because of bitonic sort, for
which table length divisable by 2 is needed.
*/
template <data_t value>
__global__ void addPaddingKernel(data_t *dataTable, data_t *dataBuffer, uint_t start, uint_t length)
{
    uint_t elemsPerThreadBlock = THREADS_PER_PADDING * ELEMS_PER_THREAD_PADDING;
    uint_t offset = blockIdx.x * elemsPerThreadBlock;
    uint_t dataBlockLength = offset + elemsPerThreadBlock <= length ? elemsPerThreadBlock : length - offset;
    offset += start;

    for (uint_t tx = threadIdx.x; tx < dataBlockLength; tx += THREADS_PER_PADDING)
    {
        uint_t index = offset + tx;
        dataTable[index] = value;
        dataBuffer[index] = value;
    }
}

template __global__ void addPaddingKernel<MIN_VAL>(
    data_t *dataTable, data_t *dataBuffer, uint_t start, uint_t length
);
template __global__ void addPaddingKernel<MAX_VAL>(
    data_t *dataTable, data_t *dataBuffer, uint_t start, uint_t length
);


/*
Sorts sub-blocks of input data with bitonic sort.
*/
template <order_t sortOrder>
__global__ void bitonicSortKernel(data_t *dataTable, uint_t tableLen)
{
    extern __shared__ data_t sortTile[];
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t offset = blockIdx.x * elemsPerThreadBlock;

    // If shared memory size is lower than table length, than adjecent blocks have to be ordered in opposite
    // direction in order to create bitonic sequences.
    bool blockDirection = sortOrder ^ (blockIdx.x & 1);

    // Loads data into shared memory
    for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock; tx += THREADS_PER_BITONIC_SORT)
    {
        sortTile[tx] = dataTable[offset + tx];
    }

    // Bitonic sort
    for (uint_t subBlockSize = 1; subBlockSize < elemsPerThreadBlock; subBlockSize <<= 1)
    {
        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1)
        {
            __syncthreads();
            for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock >> 1; tx += THREADS_PER_BITONIC_SORT)
            {
                bool direction = blockDirection ^ ((tx & subBlockSize) != 0);
                uint_t start = 2 * tx - (tx & (stride - 1));

                if (direction)
                {
                    compareExchange<ORDER_DESC>(&sortTile[start], &sortTile[start + stride]);
                }
                else
                {
                    compareExchange<ORDER_ASC>(&sortTile[start], &sortTile[start + stride]);
                }
            }
        }
    }

    // Stores sorted elements from shared to global memory
    __syncthreads();
    for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock; tx += THREADS_PER_BITONIC_SORT) {
        dataTable[offset + tx] = sortTile[tx];
    }
}

template __global__ void bitonicSortKernel<ORDER_ASC>(data_t *dataTable, uint_t tableLen);
template __global__ void bitonicSortKernel<ORDER_DESC>(data_t *dataTable, uint_t tableLen);


/*
Generates initial intervals and continues to evolve them until the end step.
*/
template <order_t sortOrder>
__global__ void initIntervalsKernel(
    data_t *table, interval_t *intervals, uint_t tableLen, uint_t stepStart, uint_t stepEnd
)
{
    extern __shared__ interval_t intervalsTile[];
    uint_t subBlockSize = 1 << stepStart;
    uint_t activeThreadsPerBlock = tableLen / subBlockSize / gridDim.x;
    uint_t elemsPerThreadBlock = blockDim.x * ELEMS_PER_INIT_INTERVALS;

    for (uint_t tx = threadIdx.x; tx < activeThreadsPerBlock; tx += blockDim.x)
    {
        uint_t intervalIndex = blockIdx.x * activeThreadsPerBlock + tx;
        uint_t offset0 = intervalIndex * subBlockSize;
        uint_t offset1 = intervalIndex * subBlockSize + subBlockSize / 2;

        // In every odd block intervals have to be rotated
        intervalsTile[tx].offset0 = intervalIndex % 2 ? offset1 : offset0;
        intervalsTile[tx].offset1 = intervalIndex % 2 ? offset0 : offset1;
        intervalsTile[tx].length0 = subBlockSize / 2;
        intervalsTile[tx].length1 = subBlockSize / 2;
    }
    __syncthreads();

    generateIntervals<sortOrder>(
        table, subBlockSize, 1 << stepEnd, 1, activeThreadsPerBlock, ELEMS_PER_INIT_INTERVALS
    );

    // Calculates offset in global intervals array
    interval_t *outputIntervalsGlobal = intervals + blockIdx.x * elemsPerThreadBlock;
    // Depending if the number of repetitions is divisable by 2, generated intervals are located in FIRST half
    // OR in SECOND half of shared memory (shared memory has 2x size of generated intervals for buffer purposes)
    interval_t *outputIntervalsLocal = intervalsTile + ((stepStart - stepEnd) % 2 != 0 ? elemsPerThreadBlock : 0);

    // Stores generated intervals from shared to global memory
    for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock; tx += blockDim.x)
    {
        outputIntervalsGlobal[tx] = outputIntervalsLocal[tx];
    }
}

template __global__ void initIntervalsKernel<ORDER_ASC>(
    data_t *table, interval_t *intervals, uint_t tableLen, uint_t stepStart, uint_t stepEnd
);
template __global__ void initIntervalsKernel<ORDER_DESC>(
    data_t *table, interval_t *intervals, uint_t tableLen, uint_t stepStart, uint_t stepEnd
);


/*
Reads the existing intervals from global memory and evolve them until the end step.
*/
template <order_t sortOrder>
__global__ void generateIntervalsKernel(
    data_t *table, interval_t *inputIntervals, interval_t *outputIntervals, uint_t tableLen, uint_t phase,
    uint_t stepStart, uint_t stepEnd
)
{
    extern __shared__ interval_t intervalsTile[];
    uint_t subBlockSize = 1 << stepStart;
    uint_t activeThreadsPerBlock = tableLen / subBlockSize / gridDim.x;
    interval_t *inputIntervalsGlobal = inputIntervals + blockIdx.x * activeThreadsPerBlock;

    // Active threads read existing intervals from global memory
    for (uint_t tx = threadIdx.x; tx < activeThreadsPerBlock; tx += blockDim.x)
    {
        intervalsTile[tx] = inputIntervalsGlobal[tx];
    }
    __syncthreads();

    generateIntervals<sortOrder>(
        table, subBlockSize, 1 << stepEnd, 1 << (phase - stepStart), activeThreadsPerBlock,
        ELEMS_PER_GEN_INTERVALS
    );

    uint_t elemsPerThreadBlock = blockDim.x * ELEMS_PER_GEN_INTERVALS;
    // Calculates offset in global intervals array
    interval_t *outputIntervalsGlobal = outputIntervals + blockIdx.x * elemsPerThreadBlock;
    // Depending if the number of repetitions is divisable by 2, generated intervals are located in FIRST half
    // OR in SECOND half of shared memory (shared memory has 2x size of all generated intervals for buffer purposes)
    interval_t *outputIntervalsLocal = intervalsTile + ((stepStart - stepEnd) % 2 != 0 ? elemsPerThreadBlock : 0);

    // Stores generated intervals from shared to global memory
    for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock; tx += blockDim.x)
    {
        outputIntervalsGlobal[tx] = outputIntervalsLocal[tx];
    }
}

template __global__ void generateIntervalsKernel<ORDER_ASC>(
    data_t *table, interval_t *input, interval_t *output, uint_t tableLen, uint_t phase, uint_t stepStart,
    uint_t stepEnd
);
template __global__ void generateIntervalsKernel<ORDER_DESC>(
    data_t *table, interval_t *input, interval_t *output, uint_t tableLen, uint_t phase, uint_t stepStart,
    uint_t stepEnd
);


/*
Global bitonic merge for sections, where stride IS GREATER OR EQUAL than max shared memory.
*/
template <order_t sortOrder>
__global__ void bitonicMergeKernel(data_t *input, data_t *output, interval_t *intervals, uint_t phase)
{
    extern __shared__ data_t mergeTile[];
    interval_t interval = intervals[blockIdx.x];

    // Elements inside same sub-block have to be ordered in same direction
    uint_t elemsPerThreadBlock = THREADS_PER_MERGE * ELEMS_PER_MERGE;
    uint_t offset = blockIdx.x * elemsPerThreadBlock;
    bool orderAsc = !sortOrder ^ ((offset >> phase) & 1);

    // Loads data from global to shared memory
    for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock; tx += THREADS_PER_MERGE)
    {
        mergeTile[tx] = getTableElement(input, interval, tx);
    }

    // Bitonic merge
    for (uint_t stride = elemsPerThreadBlock / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();
        for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock / 2; tx += THREADS_PER_MERGE)
        {
            uint_t start = 2 * tx - (tx & (stride - 1));

            if (orderAsc)
            {
                compareExchange<ORDER_ASC>(&mergeTile[start], &mergeTile[start + stride]);
            }
            else
            {
                compareExchange<ORDER_DESC>(&mergeTile[start], &mergeTile[start + stride]);
            }
        }
    }

    // Stores sorted data to buffer array
    __syncthreads();
    for (uint_t tx = threadIdx.x; tx < elemsPerThreadBlock; tx += THREADS_PER_MERGE)
    {
        output[offset + tx] = mergeTile[tx];
    }
}

template __global__ void bitonicMergeKernel<ORDER_ASC>(
    data_t *input, data_t *output, interval_t *intervals, uint_t phase
);
template __global__ void bitonicMergeKernel<ORDER_DESC>(
    data_t *input, data_t *output, interval_t *intervals, uint_t phase
);
