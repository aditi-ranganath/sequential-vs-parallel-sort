#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if (((int_t)(elem1->key - elem2->key) <= 0) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

__device__ el_t getTableElement(el_t *table, interval_t *intervals, uint_t index) {
    uint_t i = 0;
    while (index >= intervals[i].len) {
        index -= intervals[i].len;
        i++;
    }

    return table[intervals[i].offset + index];
}

__device__ int binarySearch(el_t* table, interval_t *intervals, uint_t subBlockHalfLen) {
    int_t indexStart = 0;
    int_t indexEnd = intervals[0].len;

    while (indexStart < indexEnd) {
        int index = indexStart + (indexEnd - indexStart) / 2;
        el_t el0 = getTableElement(table, intervals, index);
        el_t el1 = getTableElement(table, intervals, index + subBlockHalfLen);

        // TODO double-check for stability
        if (el0.key < el1.key) {
            indexStart = index + 1;
        } else {
            indexEnd = index;
        }
    }

    return indexStart;
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
__global__ void bitonicSortKernel(el_t *table, bool orderAsc) {
    extern __shared__ el_t sortTile[];
    // If shared memory size is lower than table length, than every block has to be ordered
    // in opposite direction -> bitonic sequence.
    bool blockDirection = orderAsc ^ (blockIdx.x & 1);

    // Every thread loads 2 elements
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    sortTile[threadIdx.x] = table[index];
    sortTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    // Bitonic sort
    for (uint_t subBlockSize = 1; subBlockSize <= blockDim.x; subBlockSize <<= 1) {
        bool direction = blockDirection ^ ((threadIdx.x & subBlockSize) != 0);

        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            __syncthreads();
            uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            compareExchange(&sortTile[start], &sortTile[start + stride], direction);
        }
    }

    __syncthreads();
    table[index] = sortTile[threadIdx.x];
    table[blockDim.x + index] = sortTile[blockDim.x + threadIdx.x];
}

__global__ void generateIntervalsKernel(el_t *table, interval_t *intervals, uint_t tableLen, uint_t step,
                                        uint_t phasesBitonicMerge) {
    extern __shared__ interval_t intervalsTile[];
    uint_t index = 2 * threadIdx.x;
    uint_t subBlockSize = 1 << step;

    if (threadIdx.x < tableLen / subBlockSize) {
        intervalsTile[index].offset = threadIdx.x * subBlockSize;
        intervalsTile[index].len = subBlockSize / 2;
        intervalsTile[index + 1].offset = threadIdx.x * subBlockSize + subBlockSize / 2;
        intervalsTile[index + 1].len = subBlockSize / 2;
    }

    for (; step > phasesBitonicMerge; step--, subBlockSize /= 2) {
        // TODO try to put in if statement if possible
        __syncthreads();
        interval_t interval0 = intervalsTile[index];
        interval_t interval1 = intervalsTile[index + 1];

        if (interval0.offset > interval1.offset) {
            interval_t temp = interval0;
            interval0 = interval1;
            interval1 = temp;

            intervalsTile[index] = interval0;
            intervalsTile[index + 1] = interval1;
        }

        __syncthreads();
        uint_t activeThreads = tableLen / (1 << step);

        if (threadIdx.x < activeThreads) {
            uint_t q = binarySearch(table, intervalsTile + index, subBlockSize / 2);

            // Left sub-block
            intervalsTile[2 * index].offset = interval0.offset;
            intervalsTile[2 * index].len = q;
            intervalsTile[2 * index + 1].offset = interval1.offset + interval1.len - subBlockSize / 2 + q;
            intervalsTile[2 * index + 1].len = subBlockSize / 2 - q;
            // Right sub-block
            intervalsTile[2 * index + 2].offset = interval1.offset;
            intervalsTile[2 * index + 2].len = q + interval1.len - subBlockSize / 2;
            intervalsTile[2 * index + 3].offset = interval0.offset + q;
            intervalsTile[2 * index + 3].len = interval0.len - q;
        }
    }

    __syncthreads();
    intervals[index] = intervalsTile[index];
    intervals[index + 1] = intervalsTile[index + 1];

    /*if (threadIdx.x == 0) {
        for (int i = 0; i < 8; i++) {
            if (i && (i % 2 == 0)) {
                printf("\n");
            }
            if (i && (i % 2 != 0)){
                printf(", ");
            }

            printf("[%2d, %2d]", intervalsTile[i].offset, intervalsTile[i].len);
        }

        printf("\n\n");
    }*/
}

/*
Global bitonic merge for sections, where stride IS GREATER OR EQUAL than max shared memory.
*/
__global__ void bitonicMergeKernel(el_t *input, el_t *output, interval_t *intervals, uint_t phase, bool orderAsc) {
    extern __shared__ el_t mergeTile[];
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    // Elements inside same sub-block have to be ordered in same direction
    bool direction = orderAsc ^ ((index >> phase) & 1);

    // Every thread loads 2 elements
    mergeTile[threadIdx.x] = getTableElement(input, intervals, index);
    mergeTile[blockDim.x + threadIdx.x] = getTableElement(input, intervals, blockDim.x + index);
    //printf("%2d %2d %2d\n", blockIdx.x, mergeTile[threadIdx.x].key, mergeTile[blockDim.x + threadIdx.x].key);

    // Bitonic merge
    for (uint_t stride = blockDim.x; stride > 0; stride >>= 1) {
        __syncthreads();
        uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        compareExchange(&mergeTile[start], &mergeTile[start + stride], direction);
    }

    __syncthreads();
    output[index] = mergeTile[threadIdx.x];
    output[blockDim.x + index] = mergeTile[blockDim.x + threadIdx.x];
}
