#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes device memory.
*/
void memoryDataInit(el_t *h_table, el_t **d_table, el_t **d_buffer, interval_t **intervals,
    interval_t **intervalsBuffer, uint_t tableLen, uint_t intervalsLen) {
    hipError_t error;

    error = hipMalloc(d_table, tableLen * sizeof(**d_table));
    checkCudaError(error);
    error = hipMalloc(d_buffer, tableLen * sizeof(**d_buffer));
    checkCudaError(error);
    error = hipMalloc(intervals, intervalsLen * sizeof(**intervals));
    checkCudaError(error);
    error = hipMalloc(intervalsBuffer, intervalsLen * sizeof(**intervalsBuffer));
    checkCudaError(error);

    error = hipMemcpy(*d_table, h_table, tableLen * sizeof(**d_table), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(el_t *table, uint_t tableLen, uint_t phasesBitonicSort, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t subBlockSize = 1 << phasesBitonicSort;
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, subBlockSize * sizeof(*table)>>>(
        table, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

void runInitIntervalsKernel(el_t *table, interval_t *intervals, uint_t tableLen, uint_t phasesAll,
                            uint_t stepStart, uint_t stepEnd) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t intervalsLen = 1 << (phasesAll - stepEnd);
    uint_t threadBlockSize = min(intervalsLen / 2, THREADS_PER_INIT_INTERVALS);
    dim3 dimGrid(intervalsLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    initIntervalsKernel<<<dimGrid, dimBlock, 2 * threadBlockSize * sizeof(*intervals)>>>(
        table, intervals, tableLen, stepStart, stepEnd
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing kernel for initializing intervals");*/
}

void runGenerateIntervalsKernel(el_t *table, interval_t *input, interval_t *output, uint_t tableLen,
                                uint_t phasesAll, uint_t phase, uint_t stepStart, uint_t stepEnd) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t intervalsLen = 1 << (phasesAll - stepEnd);
    uint_t threadBlockSize = min(intervalsLen / 2, THREADS_PER_GEN_INTERVALS);
    dim3 dimGrid(intervalsLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    generateIntervalsKernel<<<dimGrid, dimBlock, 2 * threadBlockSize * sizeof(*input)>>>(
        table, input, output, tableLen, phase, stepStart, stepEnd
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing kernel for generating intervals");*/
}

void runBitoicMergeKernel(el_t *input, el_t *output, interval_t *intervals, uint_t tableLen,
                          uint_t phasesBitonicMerge, uint_t phase, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t subBlockSize = 1 << phasesBitonicMerge;
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    startStopwatch(&timer);
    bitonicMergeKernel<<<dimGrid, dimBlock, subBlockSize * sizeof(*input)>>>(
        input, output, intervals, phase, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge kernel");*/
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    el_t *d_table, *d_buffer;
    interval_t *d_intervals, *d_intervalsBuffer;
    // Every thread loads and sorts 2 elements in first bitonic sort kernel
    uint_t phasesAll = log2((double)tableLen);
    uint_t phasesBitonicSort = log2((double)min(tableLen, 2 * THREADS_PER_SORT));
    uint_t phasesBitonicMerge = log2((double)2 * THREADS_PER_MERGE);
    uint_t phasesInitIntervals = log2((double)2 * THREADS_PER_INIT_INTERVALS);
    uint_t phasesGenerateIntervals = log2((double)2 * THREADS_PER_GEN_INTERVALS);
    uint_t intervalsLen = 1 << (phasesAll - phasesBitonicMerge);

    LARGE_INTEGER timer;
    double time;
    hipError_t error;

    memoryDataInit(h_input, &d_table, &d_buffer, &d_intervals, &d_intervalsBuffer, tableLen, intervalsLen);

    startStopwatch(&timer);
    runBitoicSortKernel(d_table, tableLen, phasesBitonicSort, orderAsc);
    //runPrintTableKernel(d_table, tableLen);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++) {
        uint_t stepStart = phase;
        uint_t stepEnd = max((double)phasesBitonicMerge, (double)phase - phasesInitIntervals);
        runInitIntervalsKernel(d_table, d_intervals, tableLen, phasesAll, stepStart, stepEnd);

        while (stepEnd > phasesBitonicMerge) {
            interval_t *tempIntervals = d_intervals;
            d_intervals = d_intervalsBuffer;
            d_intervalsBuffer = tempIntervals;

            stepStart = stepEnd;
            stepEnd = max((double)phasesBitonicMerge, (double)stepStart - phasesGenerateIntervals);
            runGenerateIntervalsKernel(d_table, d_intervalsBuffer, d_intervals, tableLen, phasesAll, phase,
                                       stepStart, stepEnd);
        }

        runBitoicMergeKernel(d_table, d_buffer, d_intervals, tableLen, phasesBitonicMerge, phase, orderAsc);
        //runPrintTableKernel(d_table, tableLen);

        el_t *tempTable = d_table;
        d_table = d_buffer;
        d_buffer = tempTable;
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    time = endStopwatch(timer, "Executing parallel bitonic sort.");
    printf("Operations: %.2f M/s\n", tableLen / 1000.0 / time);

    error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_table);
}
