#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes device memory.
*/
void memoryDataInit(el_t *h_table, el_t **d_table, el_t **d_buffer, interval_t **intervals,
    interval_t **intervalsBuffer, uint_t tableLen, uint_t intervalsLen) {
    hipError_t error;

    error = hipMalloc(d_table, tableLen * sizeof(**d_table));
    checkCudaError(error);
    error = hipMalloc(d_buffer, tableLen * sizeof(**d_buffer));
    checkCudaError(error);
    error = hipMalloc(intervals, intervalsLen * sizeof(**intervals));
    checkCudaError(error);
    error = hipMalloc(intervalsBuffer, intervalsLen * sizeof(**intervalsBuffer));
    checkCudaError(error);

    error = hipMemcpy(*d_table, h_table, tableLen * sizeof(**d_table), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(el_t *table, uint_t tableLen, uint_t phasesBitonicSort, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t subBlockSize = 1 << phasesBitonicSort;
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, subBlockSize * sizeof(*table)>>>(
        table, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

void runInitIntervalsKernel(el_t *table, interval_t *intervals, uint_t tableLen, uint_t step,
                            uint_t phasesBitonicMerge) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t intervalsLen = 1 << (step - phasesBitonicMerge);
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(intervalsLen / 2, 1, 1);

    startStopwatch(&timer);
    initIntervalsKernel<<<dimGrid, dimBlock, intervalsLen * sizeof(*intervals)>>>(
        table, intervals, tableLen, step, phasesBitonicMerge
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing kernel for generating intervals");*/
}

void runGenerateIntervalsKernel(el_t *table, interval_t *input, interval_t *output, uint_t tableLen,
                                uint_t phase, uint_t step, uint_t phasesBitonicMerge) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t intervalsLen = 1 << (phase - phasesBitonicMerge);
    dim3 dimGrid(1, 1, 1);
    dim3 dimBlock(intervalsLen / 2, 1, 1);

    startStopwatch(&timer);
    generateIntervalsKernel<<<dimGrid, dimBlock, intervalsLen * sizeof(*input)>>>(
        table, input, output, tableLen, phase, step, phasesBitonicMerge
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing kernel for generating intervals");*/
}

void runBitoicMergeKernel(el_t *input, el_t *output, interval_t *intervals, uint_t tableLen,
                          uint_t phasesBitonicMerge, uint_t phase, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t subBlockSize = 1 << phasesBitonicMerge;
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    startStopwatch(&timer);
    bitonicMergeKernel<<<dimGrid, dimBlock, subBlockSize * sizeof(*input)>>>(
        input, output, intervals, phase, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    el_t *d_table, *d_buffer;
    interval_t *d_intervals, *d_intervalsBuffer;
    // Every thread loads and sorts 2 elements in first bitonic sort kernel
    uint_t phasesAll = log2((double)tableLen);
    uint_t phasesBitonicSort = 3;  // log2((double)min(tableLen / 2, THREADS_PER_SORT));
    uint_t phasesBitonicMerge = 1;  // log2((double)THREADS_PER_MERGE);
    uint_t intervalsLen = 1 << (phasesAll - phasesBitonicMerge);

    LARGE_INTEGER timer;
    double time;
    hipError_t error;

    memoryDataInit(h_input, &d_table, &d_buffer, &d_intervals, &d_intervalsBuffer, tableLen, intervalsLen);

    startStopwatch(&timer);
    runBitoicSortKernel(d_table, tableLen, phasesBitonicSort, orderAsc);
    runPrintTableKernel(d_table, tableLen);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++) {
        runInitIntervalsKernel(d_table, d_intervals, tableLen, phase, phasesBitonicMerge + 1);

        interval_t *tempIntervals = d_intervals;
        d_intervals = d_intervalsBuffer;
        d_intervalsBuffer = tempIntervals;

        runGenerateIntervalsKernel(d_table, d_intervalsBuffer, d_intervals, tableLen, phase, phase - 2,
                                   phasesBitonicMerge);
        runBitoicMergeKernel(d_table, d_buffer, d_intervals, tableLen, phasesBitonicMerge, phase, orderAsc);
        //runPrintTableKernel(d_table, tableLen);

        el_t *tempTable = d_table;
        d_table = d_buffer;
        d_buffer = tempTable;
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    time = endStopwatch(timer, "Executing parallel bitonic sort.");
    printf("Operations: %.2f M/s\n", tableLen / 1000.0 / time);

    error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_table);
}
