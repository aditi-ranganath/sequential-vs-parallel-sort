#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "data_types.h"
#include "kernels.h"


/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(data_t *dataTable, uint_t tableLen, order_t sortOrder) {
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(dataTable, tableLen);
    }
    else
    {
        bitonicSortKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(dataTable, tableLen);
    }
}

void runInitIntervalsKernel(
    data_t *table, interval_t *intervals, uint_t tableLen, uint_t phasesAll, uint_t stepStart, uint_t stepEnd
)
{
    uint_t intervalsLen = 1 << (phasesAll - stepEnd);
    uint_t threadBlockSize = min(intervalsLen / 2, THREADS_PER_INIT_INTERVALS);
    dim3 dimGrid(intervalsLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    initIntervalsKernel<<<dimGrid, dimBlock, 2 * threadBlockSize * sizeof(*intervals)>>>(
        table, intervals, tableLen, stepStart, stepEnd
    );
}

void runGenerateIntervalsKernel(
    data_t *table, interval_t *input, interval_t *output, uint_t tableLen, uint_t phasesAll, uint_t phase,
    uint_t stepStart, uint_t stepEnd
)
{
    uint_t intervalsLen = 1 << (phasesAll - stepEnd);
    uint_t threadBlockSize = min(intervalsLen / 2, THREADS_PER_GEN_INTERVALS);
    dim3 dimGrid(intervalsLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    generateIntervalsKernel<<<dimGrid, dimBlock, 2 * threadBlockSize * sizeof(*input)>>>(
        table, input, output, tableLen, phase, stepStart, stepEnd
    );
}

void runBitoicMergeKernel(
    data_t *input, data_t *output, interval_t *intervals, uint_t tableLen, uint_t phasesBitonicMerge,
    uint_t phase, order_t sortOrder
)
{
    // Every thread loads and sorts 2 elements
    uint_t phases = min(phasesBitonicMerge, phase);
    uint_t subBlockSize = 1 << phases;
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicMergeKernel<ORDER_ASC><<<dimGrid, dimBlock, subBlockSize * sizeof(*input)>>>(
            input, output, intervals, phase
        );
    }
    else
    {
        bitonicMergeKernel<ORDER_DESC><<<dimGrid, dimBlock, subBlockSize * sizeof(*input)>>>(
            input, output, intervals, phase
        );
    }
}

double sortParallel(
    data_t *h_output, data_t *d_dataTable, data_t *d_dataBuffer, interval_t *d_intervals,
    interval_t *d_intervalsBuffer, uint_t tableLen, order_t sortOrder
)
{
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;

    // Every thread loads and processes 2 elements
    uint_t phasesAll = log2((double)tableLenPower2);
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesBitonicMerge = log2((double)2 * THREADS_PER_MERGE);
    uint_t phasesInitIntervals = log2((double)2 * THREADS_PER_INIT_INTERVALS);
    uint_t phasesGenerateIntervals = log2((double)2 * THREADS_PER_GEN_INTERVALS);
    uint_t intervalsLen = 1 << (phasesAll - phasesBitonicMerge);

    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    runBitoicSortKernel(d_dataTable, tableLen, sortOrder);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++) {
        uint_t stepStart = phase;
        uint_t stepEnd = max((double)phasesBitonicMerge, (double)phase - phasesInitIntervals);
        runInitIntervalsKernel(d_dataTable, d_intervals, tableLen, phasesAll, stepStart, stepEnd);

        // After initial intervals were generated intervals have to be evolved to the end
        while (stepEnd > phasesBitonicMerge) {
            interval_t *tempIntervals = d_intervals;
            d_intervals = d_intervalsBuffer;
            d_intervalsBuffer = tempIntervals;

            stepStart = stepEnd;
            stepEnd = max((double)phasesBitonicMerge, (double)stepStart - phasesGenerateIntervals);
            runGenerateIntervalsKernel(
                d_dataTable, d_intervalsBuffer, d_intervals, tableLen, phasesAll, phase, stepStart, stepEnd
            );
        }

        // Global merge with intervals
        runBitoicMergeKernel(
            d_dataTable, d_dataBuffer, d_intervals, tableLen, phasesBitonicMerge, phase, sortOrder
        );

        data_t *tempTable = d_dataTable;
        d_dataTable = d_dataBuffer;
        d_dataBuffer = tempTable;
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_output, d_dataTable, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
