#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <cudpp.h>

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_table, el_t **d_table, el_t **d_bufffer, uint_t **d_bucketOffsetsGlobal,
                uint_t **bucketOffsetsLocal, uint_t **d_bucketSizes, uint_t tableLen, uint_t bucketsLen) {
    hipError_t error;

    error = hipMalloc(d_table, tableLen * sizeof(**d_table));
    checkCudaError(error);
    error = hipMalloc(d_bufffer, tableLen * sizeof(**d_bufffer));
    checkCudaError(error);
    error = hipMalloc(d_bucketOffsetsGlobal, bucketsLen * sizeof(**bucketOffsetsLocal));
    checkCudaError(error);
    error = hipMalloc(bucketOffsetsLocal, bucketsLen * sizeof(**bucketOffsetsLocal));
    checkCudaError(error);
    error = hipMalloc(d_bucketSizes, bucketsLen * sizeof(**d_bucketSizes));
    checkCudaError(error);

    error = hipMemcpy(*d_table, h_table, tableLen * sizeof(**d_table), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Initializes library CUDPP, which implements scan() function
*/
void cudppInitScan(CUDPPHandle *scanPlan, uint_t tableLen) {
    // Initializes the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    *scanPlan = 0;
    CUDPPResult result = cudppPlan(theCudpp, scanPlan, config, tableLen, 1, 0);

    if (result != CUDPP_SUCCESS) {
        printf("Error creating CUDPPPlan\n");
        getchar();
        exit(-1);
    }
}

/*
Runs kernel, which sorts data blocks in shared memory with radix sort.
*/
void runRadixSortLocalKernel(el_t *table, uint_t tableLen, uint_t bitOffset, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t threadBlockSize = min(tableLen / 2, THREADS_PER_LOCAL_SORT);
    dim3 dimGrid(tableLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    radixSortLocalKernel<<<dimGrid, dimBlock, 2 * threadBlockSize * sizeof(*table)>>>(
        table, bitOffset, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing local parallel radix sort.");*/
}

void runGenerateBucketsKernel(el_t *table, uint_t *blockOffsets, uint_t *blockSizes, uint_t tableLen,
                              uint_t bitOffset) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t threadBlockSize = min(tableLen / 2, THREADS_PER_LOCAL_SORT);
    uint_t sharedMemSize = 2 * threadBlockSize * sizeof(uint_t) + 2 * RADIX * sizeof(uint_t);
    dim3 dimGrid(tableLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    generateBucketsKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        table, blockOffsets, blockSizes, bitOffset
    );
}

void runRadixSortGlobalKernel(el_t *input,  el_t *output, uint_t *offsetsLocal, uint_t *offsetsGlobal,
                              uint_t tableLen, uint_t bitOffset, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t threadBlockSize = min(tableLen / 2, THREADS_PER_GLOBAL_SORT);
    dim3 dimGrid(tableLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    radixSortGlobalKernel<<<dimGrid, dimBlock, 2 * threadBlockSize * sizeof(*input)>>>(
        input, output, offsetsLocal, offsetsGlobal, bitOffset
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing global parallel radix sort.");*/
}

void runPrintTableKernel(uint_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    el_t *d_table, *d_bufffer;
    uint_t *d_bucketOffsetsLocal, *d_bucketOffsetsGlobal, *d_bucketSizes;
    uint_t threadsPerSort = min(tableLen / 2, THREADS_PER_LOCAL_SORT);
    uint_t bucketsLen = RADIX * (tableLen / (2 * threadsPerSort));
    CUDPPHandle scanPlan;

    LARGE_INTEGER timer;
    hipError_t error;

    // Init memory and library CUDPP
    memoryInit(h_input, &d_table, &d_bufffer, &d_bucketOffsetsLocal, &d_bucketOffsetsGlobal, &d_bucketSizes,
               tableLen, bucketsLen);
    cudppInitScan(&scanPlan, bucketsLen);

    startStopwatch(&timer);

    /*runRadixSortLocalKernel(d_table, tableLen, 0, orderAsc);*/

    for (uint_t bitOffset = 0; bitOffset < sizeof(uint_t) * 8; bitOffset += BIT_COUNT) {
        runRadixSortLocalKernel(d_table, tableLen, bitOffset, orderAsc);
        runGenerateBucketsKernel(d_table, d_bucketOffsetsLocal, d_bucketSizes, tableLen, bitOffset);

        CUDPPResult result = cudppScan(scanPlan, d_bucketOffsetsGlobal, d_bucketSizes, bucketsLen);
        if (result != CUDPP_SUCCESS) {
            printf("Error in cudppScan()\n");
            getchar();
            exit(-1);
        }

        runRadixSortGlobalKernel(
            d_table, d_bufffer, d_bucketOffsetsLocal, d_bucketOffsetsGlobal, tableLen, bitOffset, orderAsc
        );

        el_t *temp = d_table;
        d_table = d_bufffer;
        d_bufffer = temp;
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing parallel radix sort.");

    error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_table);
    hipFree(d_bufffer);
}
