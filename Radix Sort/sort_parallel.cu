#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


void memoryDataInit() {
    // TODO
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    LARGE_INTEGER timer;
    hipError_t error;

    // TODO
}
