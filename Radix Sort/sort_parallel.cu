#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


void memoryInit(el_t *h_table, el_t **d_table, uint_t tableLen) {
    hipError_t error;

    error = hipMalloc(d_table, tableLen * sizeof(**d_table));
    checkCudaError(error);

    error = hipMemcpy(*d_table, h_table, tableLen * sizeof(**d_table), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void runSortBlockKernel(el_t *table, uint_t tableLen, uint_t startBit, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t threadBlockSize = min(tableLen / 2, THREADS_PER_SORT);
    dim3 dimGrid(tableLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    sortBlockKernel<<<dimGrid, dimBlock, 2 * threadBlockSize * sizeof(*table)>>>(
        table, startBit, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing parallel radix sort of blocks.");*/
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    el_t *d_table;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_input, &d_table, tableLen);

    startStopwatch(&timer);

    for (uint_t startBit = 0; startBit < sizeof(uint_t) * 8; startBit += BIT_COUNT) {
        runSortBlockKernel(d_table, tableLen, startBit, orderAsc);
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing parallel radix sort.");

    error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);
}
