#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <cudpp.h>

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Initializes library CUDPP, which implements scan() function
*/
void cudppInitScan(CUDPPHandle *scanPlan, uint_t tableLen)
{
    // Initializes the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    *scanPlan = 0;
    CUDPPResult result = cudppPlan(theCudpp, scanPlan, config, tableLen, 1, 0);

    if (result != CUDPP_SUCCESS)
    {
        printf("Error creating CUDPPPlan\n");
        getchar();
        exit(-1);
    }
}

/*
Adds padding of MAX/MIN values to input table, deppending if sort order is ascending or descending. This is
needed, if table length is not divisable with number of elements sorted by local radix sort. In order for
parallel sort to work, table length has to be divisable with number of elements processed by one thread block
in local radix sort.
*/
void runAddPaddingKernel(data_t *dataTable, data_t *dataBuffer, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerLocalSort = THREADS_PER_LOCAL_SORT * ELEMS_PER_THREAD_LOCAL;
    uint_t tableLenRoundedUp = roundUp(tableLen, elemsPerLocalSort);

    // If table length is already power of 2, than no padding is needed
    if (tableLen == tableLenRoundedUp)
    {
        return;
    }

    uint_t paddingLength = tableLenRoundedUp - tableLen;

    uint_t elemsPerThreadBlock = THREADS_PER_PADDING * ELEMS_PER_THREAD_PADDING;;
    dim3 dimGrid((paddingLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_PADDING, 1, 1);

    // Depending on sort order different value is used for padding.
    if (sortOrder == ORDER_ASC)
    {
        addPaddingKernel<MAX_VAL><<<dimGrid, dimBlock>>>(dataTable, dataBuffer, tableLen, paddingLength);
    }
    else
    {
        addPaddingKernel<MIN_VAL><<<dimGrid, dimBlock>>>(dataTable, dataBuffer, tableLen, paddingLength);
    }
}

/*
Runs kernel, which sorts data blocks in shared memory with radix sort according to current radix diggit,
which is specified with "bitOffset".
*/
void runRadixSortLocalKernel(data_t *dataTable, uint_t tableLen, uint_t bitOffset, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_SORT * ELEMS_PER_THREAD_LOCAL;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        radixSortLocalKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(dataTable, bitOffset);
    }
    else
    {
        radixSortLocalKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(dataTable, bitOffset);
    }
}

void runGenerateBucketsKernel(
    data_t *dataTable, uint_t *blockOffsets, uint_t *blockSizes, uint_t tableLen, uint_t bitOffset
)
{
    uint_t elemsPerLocalSort = THREADS_PER_LOCAL_SORT * ELEMS_PER_THREAD_LOCAL;
    // Shared memory size:
    // - "elemsPerLocalSort"  -> container for elements read from global memory into shared memory
    // - "2 * RADIX_PARALLEL" -> bucket local sizes + bucket local offsets
    uint_t sharedMemSize = elemsPerLocalSort * sizeof(uint_t) + 2 * RADIX_PARALLEL * sizeof(uint_t);

    dim3 dimGrid((tableLen - 1) / elemsPerLocalSort + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GEN_BUCKETS, 1, 1);

    generateBucketsKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        dataTable, blockOffsets, blockSizes, bitOffset
    );
}

/*
Scatters elements to their corresponding buckets according to current radix diggit, which is specified
with "bitOffset".
*/
void runRadixSortGlobalKernel(
    data_t *dataTable,  data_t *dataBuffer, uint_t *offsetsLocal, uint_t *offsetsGlobal, uint_t tableLen,
    uint_t bitOffset, order_t sortOrder
)
{
    uint_t elemsPerLocalSort = THREADS_PER_LOCAL_SORT * ELEMS_PER_THREAD_LOCAL;
    uint_t sharedMemSIze = elemsPerLocalSort * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerLocalSort + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_SORT, 1, 1);

    radixSortGlobalKernel<<<dimGrid, dimBlock, sharedMemSIze>>>(
        dataTable, dataBuffer, offsetsLocal, offsetsGlobal, bitOffset
    );
}

/*
Sorts data with parallel radix sort.
*/
double sortParallel(
    data_t *h_output, data_t *d_dataTable, data_t *d_dataBuffer, uint_t *d_bucketOffsetsLocal,
    uint_t *d_bucketOffsetsGlobal, uint_t *d_bucketSizes, uint_t tableLen, order_t sortOrder
)
{
    uint_t elemsPerLocalSort = THREADS_PER_LOCAL_SORT * ELEMS_PER_THREAD_LOCAL;
    uint_t bucketsLen = RADIX_PARALLEL * ((tableLen - 1) / elemsPerLocalSort + 1);
    CUDPPHandle scanPlan;
    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    cudppInitScan(&scanPlan, bucketsLen);
    runAddPaddingKernel(d_dataTable, d_dataBuffer, tableLen, sortOrder);

    for (uint_t bitOffset = 0; bitOffset < sizeof(data_t) * 8; bitOffset += BIT_COUNT_PARALLEL)
    {
        runRadixSortLocalKernel(d_dataTable, tableLen, bitOffset, sortOrder);
        runGenerateBucketsKernel(d_dataTable, d_bucketOffsetsLocal, d_bucketSizes, tableLen, bitOffset);

        // Performs global scan in order to calculate global bucket offsets from local bucket sizes
        CUDPPResult result = cudppScan(scanPlan, d_bucketOffsetsGlobal, d_bucketSizes, bucketsLen);
        if (result != CUDPP_SUCCESS)
        {
            printf("Error in cudppScan()\n");
            getchar();
            exit(-1);
        }

        runRadixSortGlobalKernel(
            d_dataTable, d_dataBuffer, d_bucketOffsetsLocal, d_bucketOffsetsGlobal, tableLen, bitOffset, sortOrder
        );

        data_t *temp = d_dataTable;
        d_dataTable = d_dataBuffer;
        d_dataBuffer = temp;
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_output, d_dataTable, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
