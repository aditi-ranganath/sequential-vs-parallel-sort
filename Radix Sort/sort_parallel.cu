#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <cudpp.h>

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Initializes library CUDPP, which implements scan() function
*/
void cudppInitScan(CUDPPHandle *scanPlan, uint_t tableLen)
{
    // Initializes the CUDPP Library
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_UINT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    *scanPlan = 0;
    CUDPPResult result = cudppPlan(theCudpp, scanPlan, config, tableLen, 1, 0);

    if (result != CUDPP_SUCCESS)
    {
        printf("Error creating CUDPPPlan\n");
        getchar();
        exit(-1);
    }
}

/*
Adds padding of MAX/MIN values to input table, deppending if sort order is ascending or descending. This is
needed, if table length is not divisable with number of elements sorted by local radix sort. In order for
parallel sort to work, table length has to be divisable with number of elements processed by one thread block
in local radix sort.
*/
void runAddPaddingKernel(data_t *dataTable, data_t *dataBuffer, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerLocalSort = THREADS_PER_LOCAL_SORT * ELEMS_PER_THREAD_LOCAL;
    uint_t tableLenRoundedUp = roundUp(tableLen, elemsPerLocalSort);

    // If table length is already power of 2, than no padding is needed
    if (tableLen == tableLenRoundedUp)
    {
        return;
    }

    uint_t paddingLength = tableLenRoundedUp - tableLen;

    uint_t elemsPerThreadBlock = THREADS_PER_PADDING * ELEMS_PER_THREAD_PADDING;;
    dim3 dimGrid((paddingLength - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_PADDING, 1, 1);

    // Depending on sort order different value is used for padding.
    if (sortOrder == ORDER_ASC)
    {
        addPaddingKernel<MAX_VAL><<<dimGrid, dimBlock>>>(dataTable, dataBuffer, tableLen, paddingLength);
    }
    else
    {
        addPaddingKernel<MIN_VAL><<<dimGrid, dimBlock>>>(dataTable, dataBuffer, tableLen, paddingLength);
    }
}

/*
Runs kernel, which sorts data blocks in shared memory with radix sort according to current radix diggit,
which is specified with "bitOffset".
*/
void runRadixSortLocalKernel(data_t *dataTable, uint_t tableLen, uint_t bitOffset, order_t sortOrder)
{
    uint_t threadBlockSize = min((tableLen - 1) / ELEMS_PER_THREAD_LOCAL + 1, THREADS_PER_LOCAL_SORT);
    uint_t sharedMemSize = threadBlockSize * ELEMS_PER_THREAD_LOCAL * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / (ELEMS_PER_THREAD_LOCAL * threadBlockSize) + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        radixSortLocalKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, bitOffset
        );
    }
    else
    {
        radixSortLocalKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(
            dataTable, bitOffset
        );
    }
}

void runGenerateBucketsKernel(
    data_t *dataTable, uint_t *blockOffsets, uint_t *blockSizes, uint_t tableLen, uint_t bitOffset
)
{
    uint_t threadBlockSize = min((tableLen - 1) / ELEMS_PER_THREAD_LOCAL + 1, THREADS_PER_LOCAL_SORT);
    uint_t sharedMemSize = ELEMS_PER_THREAD_LOCAL * threadBlockSize * sizeof(uint_t) + 2 * RADIX_PARALLEL * sizeof(uint_t);

    dim3 dimGrid((tableLen - 1) / (ELEMS_PER_THREAD_LOCAL * threadBlockSize) + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    generateBucketsKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        dataTable, blockOffsets, blockSizes, bitOffset
    );
}

void runRadixSortGlobalKernel(
    data_t *dataTable,  data_t *dataBuffer, uint_t *offsetsLocal, uint_t *offsetsGlobal, uint_t tableLen,
    uint_t bitOffset, order_t sortOrder
)
{
    uint_t threadBlockSize = min((tableLen - 1) / ELEMS_PER_THREAD_LOCAL, THREADS_PER_GLOBAL_SORT);
    uint_t sharedMemSIze = ELEMS_PER_THREAD_LOCAL * threadBlockSize * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / (ELEMS_PER_THREAD_LOCAL * threadBlockSize) + 1, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    radixSortGlobalKernel<<<dimGrid, dimBlock, sharedMemSIze>>>(
        dataTable, dataBuffer, offsetsLocal, offsetsGlobal, bitOffset
    );
}

//void runPrintTableKernel(uint_t *table, uint_t tableLen) {
//    printTableKernel<<<1, 1>>>(table, tableLen);
//    hipError_t error = hipDeviceSynchronize();
//    checkCudaError(error);
//}

double sortParallel(
    data_t *h_output, data_t *d_dataTable, data_t *d_dataBuffer, uint_t *d_bucketOffsetsLocal,
    uint_t *d_bucketOffsetsGlobal, uint_t *d_bucketSizes, uint_t tableLen, order_t sortOrder
)
{
    uint_t threadsPerSortLocal = min((tableLen - 1) / ELEMS_PER_THREAD_LOCAL + 1, THREADS_PER_LOCAL_SORT);
    uint_t bucketsLen = RADIX_PARALLEL * (tableLen / (threadsPerSortLocal * ELEMS_PER_THREAD_LOCAL));
    CUDPPHandle scanPlan;
    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    cudppInitScan(&scanPlan, bucketsLen);
    runAddPaddingKernel(d_dataTable, d_dataBuffer, tableLen, sortOrder);

    for (uint_t bitOffset = 0; bitOffset < sizeof(data_t) * 8; bitOffset += BIT_COUNT_PARALLEL)
    {
        runRadixSortLocalKernel(d_dataTable, tableLen, bitOffset, sortOrder);
        runGenerateBucketsKernel(d_dataTable, d_bucketOffsetsLocal, d_bucketSizes, tableLen, bitOffset);

        CUDPPResult result = cudppScan(scanPlan, d_bucketOffsetsGlobal, d_bucketSizes, bucketsLen);
        if (result != CUDPP_SUCCESS)
        {
            printf("Error in cudppScan()\n");
            getchar();
            exit(-1);
        }

        runRadixSortGlobalKernel(
            d_dataTable, d_dataBuffer, d_bucketOffsetsLocal, d_bucketOffsetsGlobal, tableLen, bitOffset, sortOrder
        );

        data_t *temp = d_dataTable;
        d_dataTable = d_dataBuffer;
        d_dataBuffer = temp;
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_output, d_dataTable, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
