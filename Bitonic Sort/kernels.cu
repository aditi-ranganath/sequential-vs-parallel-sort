#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if ((elem1->key <= elem2->key) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

__global__ void printTableKernel(el_t *table, uint_t tableLen) {
    for (uint_t i = 0; i < tableLen; i++) {
        printf("%2d ", table[i]);
    }
    printf("\n\n");
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
__global__ void bitonicSortKernel(el_t *table, bool orderAsc) {
    extern __shared__ el_t sortTile[];
    bool blockDirection = orderAsc ^ (blockIdx.x & 1);

    // Every thread loads 2 elements
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    sortTile[threadIdx.x] = table[index];
    sortTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    for (uint_t subBlockSize = 1; subBlockSize <= blockDim.x; subBlockSize <<= 1) {
        bool direction = blockDirection ^ ((threadIdx.x & subBlockSize) != 0);

        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            __syncthreads();
            // In first step of every phase END index has to be reversed
            uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            compareExchange(&sortTile[start], &sortTile[start + stride], direction);
        }
    }

    __syncthreads();
    table[index] = sortTile[threadIdx.x];
    table[blockDim.x + index] = sortTile[blockDim.x + threadIdx.x];
}

__global__ void multiStepKernel(el_t *table, uint_t phase, uint_t step, uint_t degree, bool orderAsc) {
    el_t tile[4];
    uint_t tileHalfSize = 1 << (degree - 1);
    uint_t stridePhase = (1 << (phase - 1));
    uint_t strideStep = (1 << (step - 1));
    uint_t threadsPerSubBlock = strideStep / tileHalfSize;
    uint_t indexThread = blockIdx.x * blockDim.x + threadIdx.x;
    uint_t indexTable = (indexThread / threadsPerSubBlock * 2 * strideStep) + (indexThread % threadsPerSubBlock);
    uint_t direction = orderAsc ^ ((indexThread / (stridePhase / tileHalfSize)) % 2);

    for (uint_t i = 0; i < tileHalfSize; i++) {
        uint_t start = indexTable + i * (1 << (step - degree));
        uint_t end = start + strideStep;

        /*if (phase == 5 && step == 3) {
            printf("%2d %2d %2d %2d\n", threadIdx.x, start, end, direction);
        }*/

        tile[i] = table[start];
        tile[i + tileHalfSize] = table[end];
    }

    /*printf("%2d %2d %2d %2d\n", tile[0].key, tile[1].key, tile[2].key, tile[3].key);*/

    // Syncthreads is not needed, because every thread proceses an separated subsection of partition
    for (uint_t stride = tileHalfSize; stride > 0; stride >>= 1) {
        for (uint_t i = 0; i < tileHalfSize; i++) {
            // In first step of every phase END index has to be reversed
            uint_t start = 2 * i - (i & (stride - 1));
            //uint_t end = start + (phase != step ? stride : 2 * stride - 2 * (i % stride) - 1);
            uint_t end = start + stride;

            /*printf("%2d %2d %2d %2d\n", threadIdx.x, tile[start].key, tile[end].key, direction);*/
            compareExchange(&tile[start], &tile[end], direction);
            /*printf("%2d %2d %2d %2d\n", threadIdx.x, tile[start].key, tile[end].key, direction);*/
        }
    }

    /*__syncthreads();
    if (threadIdx.x == 0) {
        printf("\n\n");
    }
    printf("%2d %2d %2d %2d\n", tile[0].key, tile[1].key, tile[2].key, tile[3].key);*/

    /*if (threadIdx.x == 0) {
        printf("%2d %2d %2d %2d\n", tile[0].key, tile[1].key, tile[2].key, tile[3].key);
    }*/

    for (int i = 0; i < tileHalfSize; i++) {
        uint_t start = indexTable + i * (1 << (step - degree));
        uint_t end = start + strideStep;

        table[start] = tile[i];
        table[end] = tile[i + tileHalfSize];
    }
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
__global__ void bitonicMergeKernel(el_t *table, uint_t phase, bool orderAsc) {
    extern __shared__ el_t mergeTile[];
    uint_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint_t threadsPerSubBlock = 1 << (phase - 1);
    bool subBlockDirection = (threadIndex / threadsPerSubBlock) % 2;

    // Every thread loads 2 elements
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    mergeTile[threadIdx.x] = table[index];
    mergeTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    uint_t direction = orderAsc ^ ((threadIdx.x & blockDim.x) != 0) ^ subBlockDirection;
    for (uint_t stride = blockDim.x; stride > 0; stride >>= 1) {
        __syncthreads();
        // In first step of every phase END index has to be reversed
        uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        compareExchange(&mergeTile[start], &mergeTile[start + stride], direction);
    }

    __syncthreads();
    table[index] = mergeTile[threadIdx.x];
    table[blockDim.x + index] = mergeTile[blockDim.x + threadIdx.x];
}
