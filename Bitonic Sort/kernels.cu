#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if ((elem1->key <= elem2->key) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
__global__ void bitonicSortKernel(el_t *table, bool orderAsc) {
    extern __shared__ el_t sortTile[];
    // If shared memory size is lower than table length, than every block has to be ordered
    // in opposite direction -> bitonic sequence.
    bool blockDirection = orderAsc ^ (blockIdx.x & 1);

    // Every thread loads 2 elements
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    sortTile[threadIdx.x] = table[index];
    sortTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    // Bitonic sort
    for (uint_t subBlockSize = 1; subBlockSize <= blockDim.x; subBlockSize <<= 1) {
        bool direction = blockDirection ^ ((threadIdx.x & subBlockSize) != 0);

        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            __syncthreads();
            uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            compareExchange(&sortTile[start], &sortTile[start + stride], direction);
        }
    }

    __syncthreads();
    table[index] = sortTile[threadIdx.x];
    table[blockDim.x + index] = sortTile[blockDim.x + threadIdx.x];
}

/*
Global bitonic merge for sections, where stride IS GREATER than max shared memory.
*/
__global__ void bitonicMergeGlobalKernel(el_t *table, uint_t phase, uint_t step, bool orderAsc) {
    uint_t stride = 1 << (step - 1);
    uint_t indexThread = blockIdx.x * blockDim.x + threadIdx.x;
    uint_t indexTable = 2 * indexThread - (indexThread & (stride - 1));
    // Elements inside same sub-block have to be ordered in same direction
    bool direction = orderAsc ^ ((indexTable >> phase) & 1);

    el_t el1 = table[indexTable];
    el_t el2 = table[indexTable + stride];

    compareExchange(&el1, &el2, direction);

    table[indexTable] = el1;
    table[indexTable + stride] = el2;
}

/*
Global bitonic merge for sections, where stride IS LOWER OR EQUAL than max shared memory.
*/
__global__ void bitonicMergeLocalKernel(el_t *table, uint_t phase, bool orderAsc) {
    extern __shared__ el_t mergeTile[];
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    // Elements inside same sub-block have to be ordered in same direction
    bool direction = orderAsc ^ ((index >> phase) & 1);

    // Every thread loads 2 elements
    mergeTile[threadIdx.x] = table[index];
    mergeTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    // Bitonic merge
    for (uint_t stride = blockDim.x; stride > 0; stride >>= 1) {
        __syncthreads();
        uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        compareExchange(&mergeTile[start], &mergeTile[start + stride], direction);
    }

    __syncthreads();
    table[index] = mergeTile[threadIdx.x];
    table[blockDim.x + index] = mergeTile[blockDim.x + threadIdx.x];
}
