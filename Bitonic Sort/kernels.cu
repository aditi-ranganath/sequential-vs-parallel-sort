#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if ((elem1->key <= elem2->key) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

__global__ void printTableKernel(el_t *table, uint_t tableLen) {
    for (uint_t i = 0; i < tableLen; i++) {
        printf("%2d ", table[i]);
    }
    printf("\n\n");
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
__global__ void bitonicSortKernel(el_t *table, bool orderAsc) {
    extern __shared__ el_t sortTile[];
    bool blockDirection = orderAsc ^ (blockIdx.x & 1);

    // Every thread loads 2 elements
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    sortTile[threadIdx.x] = table[index];
    sortTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    for (uint_t subBlockSize = 1; subBlockSize <= blockDim.x; subBlockSize <<= 1) {
        bool direction = blockDirection ^ ((threadIdx.x & subBlockSize) != 0);

        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            __syncthreads();
            // In first step of every phase END index has to be reversed
            uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            compareExchange(&sortTile[start], &sortTile[start + stride], direction);
        }
    }

    __syncthreads();
    table[index] = sortTile[threadIdx.x];
    table[blockDim.x + index] = sortTile[blockDim.x + threadIdx.x];
}

__global__ void bitonicMergeGlobalKernel(el_t *table, uint_t phase, uint_t step, bool orderAsc) {
    uint_t strideGlobal = 1 << (step - 1);
    uint_t indexThread = blockIdx.x * blockDim.x + threadIdx.x;
    uint_t indexTable = 2 * indexThread - (indexThread & (strideGlobal - 1));
    bool direction = orderAsc ^ ((indexThread >> (phase - 1)) & 1);

    el_t el1 = table[indexTable];
    el_t el2 = table[indexTable + strideGlobal];

    compareExchange(&el1, &el2, direction);

    table[indexTable] = el1;
    table[indexTable + strideGlobal] = el2;
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
__global__ void bitonicMergeKernel(el_t *table, uint_t phase, bool orderAsc) {
    extern __shared__ el_t mergeTile[];
    uint_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    uint_t threadsPerSubBlock = 1 << (phase - 1);
    bool subBlockDirection = (threadIndex / threadsPerSubBlock) % 2;

    // Every thread loads 2 elements
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    mergeTile[threadIdx.x] = table[index];
    mergeTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    uint_t direction = orderAsc ^ ((threadIdx.x & blockDim.x) != 0) ^ subBlockDirection;
    for (uint_t stride = blockDim.x; stride > 0; stride >>= 1) {
        __syncthreads();
        // In first step of every phase END index has to be reversed
        uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        compareExchange(&mergeTile[start], &mergeTile[start + stride], direction);
    }

    __syncthreads();
    table[index] = mergeTile[threadIdx.x];
    table[blockDim.x + index] = mergeTile[blockDim.x + threadIdx.x];
}
