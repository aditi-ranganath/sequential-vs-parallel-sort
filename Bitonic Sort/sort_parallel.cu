#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"

/*
Initializes device memory.
*/
void memoryDataInit(el_t *h_table, el_t **d_table, uint_t tableLen) {
    hipError_t error;

    error = hipMalloc(d_table, tableLen * sizeof(**d_table));
    checkCudaError(error);
    error = hipMemcpy(*d_table, h_table, tableLen * sizeof(**d_table), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(el_t *dataTable, uint_t tableLen, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable)>>>(
        dataTable, tableLen, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

void runBitonicMergeGlobalKernel(el_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeGlobalKernel<<<dimGrid, dimBlock>>>(dataTable, tableLen, step, step == phase, sortOrder);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge global kernel");*/
}

void runBitoicMergeLocalKernel(el_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeLocalKernel<<<dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable)>>>(
        dataTable, tableLen, step, phase == step, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge local kernel");*/
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, order_t sortOrder) {
    el_t *d_table;

    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    LARGE_INTEGER timer;
    hipError_t error;

    // Global bitonic merge doesn't use shared memory -> preference can be set for L1
    hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(bitonicMergeGlobalKernel), hipFuncCachePreferL1);
    memoryDataInit(h_input, &d_table, tableLen);

    startStopwatch(&timer);
    runBitoicSortKernel(d_table, tableLen, sortOrder);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++) {
        uint_t step = phase;
        while (step > phasesMergeLocal) {
            runBitonicMergeGlobalKernel(d_table, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel(d_table, tableLen, phase, step, sortOrder);
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel bitonic sort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_table);
}
