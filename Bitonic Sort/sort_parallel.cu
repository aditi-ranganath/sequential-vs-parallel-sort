#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(data_t *dataTable, uint_t tableLen, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);

    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    if (sortOrder == ORDER_ASC)
    {
        bitonicSortKernel<ORDER_ASC><<<dimGrid, dimBlock, sharedMemSize>>>(dataTable, tableLen);
    }
    else
    {
        bitonicSortKernel<ORDER_DESC><<<dimGrid, dimBlock, sharedMemSize>>>(dataTable, tableLen);
    }
}

/*
Merges array, if data blocks are larger than shared memory size. It executes only of STEP on PHASE per
kernel launch.
*/
void runBitonicMergeGlobalKernel(data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder)
{
    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_ASC, true><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_ASC, false><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
    }
    else
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeGlobalKernel<ORDER_DESC, true><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
        else
        {
            bitonicMergeGlobalKernel<ORDER_DESC, false><<<dimGrid, dimBlock>>>(dataTable, tableLen, step);
        }
    }
}

/*
Merges array when stride is lower than shared memory size. It executes all remaining STEPS of current PHASE.
*/
void runBitoicMergeLocalKernel(data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder)
{
    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    uint_t sharedMemSize = elemsPerThreadBlock * sizeof(*dataTable);
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    bool isFirstStepOfPhase = phase == step;

    if (sortOrder == ORDER_ASC)
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeLocalKernel<ORDER_ASC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_ASC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
    }
    else
    {
        if (isFirstStepOfPhase)
        {
            bitonicMergeLocalKernel<ORDER_DESC, true><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
        else
        {
            bitonicMergeLocalKernel<ORDER_DESC, false><<<dimGrid, dimBlock, sharedMemSize>>>(
                dataTable, tableLen, step
            );
        }
    }
}

/*
Sorts data with NORMALIZED BITONIC SORT.
*/
double sortParallel(data_t *h_output, data_t *d_dataTable, uint_t tableLen, order_t sortOrder)
{
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);
    runBitoicSortKernel(d_dataTable, tableLen, sortOrder);

    // Bitonic merge
    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++)
    {
        uint_t step = phase;
        while (step > phasesMergeLocal)
        {
            runBitonicMergeGlobalKernel(d_dataTable, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel(d_dataTable, tableLen, phase, step, sortOrder);
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer);

    error = hipMemcpy(h_output, d_dataTable, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);

    return time;
}
