#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes device memory.
*/
void memoryDataInit(el_t *h_table, el_t **d_table, uint_t tableLen) {
    hipError_t error;

    error = hipMalloc(d_table, tableLen * sizeof(**d_table));
    checkCudaError(error);
    error = hipMemcpy(*d_table, h_table, tableLen * sizeof(**d_table), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(el_t *table, uint_t tableLen, uint_t subBlockSize, uint_t phase, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, subBlockSize * sizeof(*table)>>>(
        table, phase, orderAsc
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    //endStopwatch(timer, "Executing bitonic sort kernel");
}

void runMultiStepKernel(el_t *table, uint_t tableLen, uint_t phase, uint_t step, uint_t degree, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t partitionSize = tableLen / (1 << degree);
    uint_t maxThreadBlockSize = getMaxThreadsPerBlock();
    uint_t threadBlockSize = min(partitionSize, maxThreadBlockSize);
    dim3 dimGrid(partitionSize / threadBlockSize, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    multiStepKernel<<<dimGrid, dimBlock>>>(table, phase, step, degree, orderAsc);
    error = hipDeviceSynchronize();
    checkCudaError(error);
    /*endStopwatch(timer, "Executing multistep kernel");*/
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    el_t *d_table;
    // Every thread loads and sorts 2 elements in first bitonic sort kernel
    uint_t subBlockSize = 2;  // min(tableLen, 2 * getMaxThreadsPerBlock());
    int_t phasesAll = log2((double)tableLen);
    int_t phasesSharedMem = log2((double)subBlockSize);

    LARGE_INTEGER timer;
    hipError_t error;

    memoryDataInit(h_input, &d_table, tableLen);

    startStopwatch(&timer);
    runBitoicSortKernel(d_table, tableLen, subBlockSize, phasesSharedMem, orderAsc);

    /*printf("After bitonic sort\n");
    runPrintTableKernel(d_table, tableLen);*/

    for (uint_t phase = phasesSharedMem + 1; phase <= phasesAll; phase++) {
        int_t step = phase;

        for (; step >= phasesSharedMem + 2; step -= 2) {
            runMultiStepKernel(d_table, tableLen, phase, step, 2, orderAsc);
            /*printf("After 2-multistep\n");
            runPrintTableKernel(d_table, tableLen);*/
        }
        for (; step >= phasesSharedMem + 1; step -= 1) {
            runMultiStepKernel(d_table, tableLen, phase, step, 1, orderAsc);
            /*printf("After 1-multistep\n");
            runPrintTableKernel(d_table, tableLen);*/
        }

        // Here only last phase is needed
        runBitoicSortKernel(d_table, tableLen, subBlockSize, phase, orderAsc);

        /*printf("After bitonic merge\n");
        runPrintTableKernel(d_table, tableLen);*/
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing parallel bitonic sort.");

    error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);
}
