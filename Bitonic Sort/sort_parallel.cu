#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "../Utils/data_types_common.h"
#include "../Utils/hip/hip_runtime.h"
#include "../Utils/host.h"
#include "constants.h"
#include "kernels.h"


/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(data_t *dataTable, uint_t tableLen, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_BITONIC_SORT, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable)>>>(
        dataTable, tableLen, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic sort kernel");*/
}

void runBitonicMergeGlobalKernel(data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elemsPerThreadBlock = THREADS_PER_GLOBAL_MERGE * ELEMS_PER_THREAD_GLOBAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_GLOBAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeGlobalKernel<<<dimGrid, dimBlock>>>(dataTable, tableLen, step, step == phase, sortOrder);
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge global kernel");*/
}

void runBitoicMergeLocalKernel(data_t *dataTable, uint_t tableLen, uint_t phase, uint_t step, order_t sortOrder) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    uint_t elemsPerThreadBlock = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;
    dim3 dimGrid((tableLen - 1) / elemsPerThreadBlock + 1, 1, 1);
    dim3 dimBlock(THREADS_PER_LOCAL_MERGE, 1, 1);

    startStopwatch(&timer);
    bitonicMergeLocalKernel<<<dimGrid, dimBlock, elemsPerThreadBlock * sizeof(*dataTable)>>>(
        dataTable, tableLen, step, phase == step, sortOrder
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing bitonic merge local kernel");*/
}

void runPrintTableKernel(data_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sortParallel(data_t *h_input, data_t *h_output, data_t *d_dataTable, uint_t tableLen, order_t sortOrder) {
    uint_t tableLenPower2 = nextPowerOf2(tableLen);
    uint_t elemsPerBlockBitonicSort = THREADS_PER_BITONIC_SORT * ELEMS_PER_THREAD_BITONIC_SORT;
    uint_t elemsPerBlockMergeLocal = THREADS_PER_LOCAL_MERGE * ELEMS_PER_THREAD_LOCAL_MERGE;

    // Number of phases, which can be executed in shared memory (stride is lower than shared memory size)
    uint_t phasesBitonicSort = log2((double)min(tableLenPower2, elemsPerBlockBitonicSort));
    uint_t phasesMergeLocal = log2((double)min(tableLenPower2, elemsPerBlockMergeLocal));
    uint_t phasesAll = log2((double)tableLenPower2);

    LARGE_INTEGER timer;
    hipError_t error;

    // Global bitonic merge doesn't use shared memory -> preference can be set for L1
    // TODO test
    hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(bitonicMergeGlobalKernel), hipFuncCachePreferL1);

    startStopwatch(&timer);
    runBitoicSortKernel(d_dataTable, tableLen, sortOrder);

    for (uint_t phase = phasesBitonicSort + 1; phase <= phasesAll; phase++) {
        uint_t step = phase;
        while (step > phasesMergeLocal) {
            runBitonicMergeGlobalKernel(d_dataTable, tableLen, phase, step, sortOrder);
            step--;
        }

        runBitoicMergeLocalKernel(d_dataTable, tableLen, phase, step, sortOrder);
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);

    double time = endStopwatch(timer);
    printf("Parallel: %.5lf ms. Swaps/s: %.2f M/s\n", time, tableLen / 500.0 / time);

    error = hipMemcpy(h_output, d_dataTable, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);
}
