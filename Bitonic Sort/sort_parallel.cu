#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"


void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    printf("TODO parallel implementation\n");
}
