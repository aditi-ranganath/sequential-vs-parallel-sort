#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, uint_t tableLen) {
    hipError_t error;

    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void runQuickSortLocalKernel(el_t *input, el_t *output, uint_t tableLen, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t threadBlockSize = THREADS_PER_SORT_LOCAL;
    uint_t elementsPerBlock = ELEMENTS_PER_THREAD_LOCAL * threadBlockSize;
    dim3 dimGrid(tableLen / elementsPerBlock, 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    quickSortLocalKernel<<<dimGrid, dimBlock, elementsPerBlock * sizeof(*input)>>>(
        input, output, tableLen, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing local parallel quicksort.");*/
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, bool orderAsc) {
    el_t *d_dataInput, *d_dataBuffer;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_dataInput, &d_dataInput, &d_dataBuffer, tableLen);

    startStopwatch(&timer);

    // Quicksort
    runQuickSortLocalKernel(d_dataInput, d_dataBuffer, tableLen, orderAsc);

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel quicksort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataBuffer, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
    hipFree(d_dataBuffer);
}
