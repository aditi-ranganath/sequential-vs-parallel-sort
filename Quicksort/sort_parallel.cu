#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes memory needed for paralel sort implementation.
*/
void memoryInit() {
    hipError_t error;
    // TODO
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    LARGE_INTEGER timer;
    hipError_t error;

    startStopwatch(&timer);

    // TODO

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel radix sort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    /*error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);*/

    /*hipFree(d_table);
    hipFree(d_bufffer);*/
}
