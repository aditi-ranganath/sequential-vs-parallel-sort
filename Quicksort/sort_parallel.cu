#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, uint_t tableLen) {
    hipError_t error;

    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, bool orderAsc) {
    el_t *d_dataInput, *d_dataBuffer;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_dataInput, &d_dataInput, &d_dataBuffer, tableLen);

    startStopwatch(&timer);

    // TODO

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel radix sort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    /*error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);*/

    /*hipFree(d_table);
    hipFree(d_bufffer);*/
}
