#include <stdio.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes memory needed for paralel sort implementation.
*/
void memoryInit(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, lparam_t **d_localParams, uint_t tableLen) {
    hipError_t error;

    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);
    error = hipMalloc(d_localParams, MAX_SEQUENCES * sizeof(**d_localParams));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void runQuickSortLocalKernel(el_t *input, el_t *output, lparam_t *localParams, uint_t tableLen, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t elementsPerBlock = tableLen / MAX_SEQUENCES;
    dim3 dimGrid(MAX_SEQUENCES, 1, 1);
    dim3 dimBlock(THREADS_PER_SORT_LOCAL, 1, 1);

    startStopwatch(&timer);
    quickSortLocalKernel<<<dimGrid, dimBlock, elementsPerBlock * sizeof(*input)>>>(
        input, output, localParams, tableLen, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing local parallel quicksort.");*/
}

void quickSort(el_t *dataInput, el_t *dataBuffer, lparam_t *h_localParams, lparam_t *d_localParams,
               uint_t tableLen, bool orderAsc) {
    h_localParams[0].start = 3;
    h_localParams[0].length = 11;

    hipMemcpy(d_localParams, h_localParams, MAX_SEQUENCES * sizeof(*d_localParams), hipMemcpyHostToDevice);

    runQuickSortLocalKernel(dataInput, dataBuffer, d_localParams, tableLen, orderAsc);
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, bool orderAsc) {
    el_t *d_dataInput, *d_dataBuffer;
    lparam_t h_localParams[MAX_SEQUENCES], *d_localParams;

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInit(h_dataInput, &d_dataInput, &d_dataBuffer, &d_localParams, tableLen);

    startStopwatch(&timer);
    quickSort(d_dataInput, d_dataBuffer, h_localParams, d_localParams, tableLen, orderAsc);

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel quicksort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataBuffer, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
    hipFree(d_dataBuffer);
}
