#include "hip/hip_runtime.h"
#include <stdio.h>
#include <climits>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes HOST memory needed for paralel sort implementation.
*/
void memoryInitHost(h_glob_seq_t **h_globalSeqHost, h_glob_seq_t **h_globalSeqHostBuffer,
                    d_glob_seq_t **h_globalSeqDev, uint_t **h_globalSeqIndexes, loc_seq_t **h_localSeq,
                    uint_t maxNumSequences, uint_t maxNumThreadBlocks) {
    // TODO malloc in pinned memory
    *h_globalSeqHost = new h_glob_seq_t[maxNumSequences];
    *h_globalSeqHostBuffer = new h_glob_seq_t[maxNumSequences];
    *h_globalSeqDev = new d_glob_seq_t[maxNumSequences];
    *h_globalSeqIndexes = new uint_t[maxNumThreadBlocks];
    *h_localSeq = new loc_seq_t[maxNumSequences];
}

/*
Initializes DEVICE memory needed for paralel sort implementation.
*/
void memoryInitDevice(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, d_glob_seq_t **d_globalSeqDev,
                      uint_t **d_globalSeqIndexes, loc_seq_t **d_localSeq, uint_t tableLen,
                      uint_t maxNumSequences, uint_t maxNumThreadBlocks) {
    hipError_t error;

    // Data memory allocation
    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);
    // Sequence metadata memory allocation
    error = hipMalloc(d_globalSeqDev, maxNumSequences * sizeof(**d_globalSeqDev));
    checkCudaError(error);
    error = hipMalloc(d_globalSeqIndexes, maxNumThreadBlocks * sizeof(**d_globalSeqIndexes));
    checkCudaError(error);
    error = hipMalloc(d_localSeq, maxNumSequences * sizeof(**d_localSeq));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void runQuickSortGlobalKernel(el_t *dataInput, el_t* dataBuffer, d_glob_seq_t *h_globalSeqHost,
                              d_glob_seq_t *d_globalSeqHost, uint_t *h_globalSeqIndexes, uint_t *d_globalSeqIndexes,
                              uint_t numSeqGlobal, uint_t threadBlockCounter, uint_t tableLen) {
    hipError_t error;
    LARGE_INTEGER timer;

    // TODO comment shared memory size, 2 * size should be enough, because scan and min/max can be
    // performed in the same array
    // TODO comment: max(min/max, lower/greater)
    uint_t sharedMemSize = max(
        2 * THREADS_PER_SORT_GLOBAL * sizeof(data_t), 2 * THREADS_PER_SORT_GLOBAL * sizeof(uint_t)
    );
    dim3 dimGrid(threadBlockCounter, 1, 1);
    dim3 dimBlock(THREADS_PER_SORT_GLOBAL, 1, 1);

    startStopwatch(&timer);

    error = hipMemcpy(d_globalSeqHost, h_globalSeqHost, numSeqGlobal * sizeof(*d_globalSeqHost),
                       hipMemcpyHostToDevice);
    checkCudaError(error);
    error = hipMemcpy(d_globalSeqIndexes, h_globalSeqIndexes, threadBlockCounter * sizeof(*d_globalSeqIndexes),
                       hipMemcpyHostToDevice);
    checkCudaError(error);

    quickSortGlobalKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        dataInput, dataBuffer, d_globalSeqHost, d_globalSeqIndexes, tableLen
    );

    error = hipMemcpy(h_globalSeqHost, d_globalSeqHost, numSeqGlobal * sizeof(*h_globalSeqHost),
                       hipMemcpyDeviceToHost);
    checkCudaError(error);

    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing global parallel quicksort.");*/
}

void runQuickSortLocalKernel(el_t *dataInput, el_t *dataBuffer, loc_seq_t *h_localSeq, loc_seq_t *d_localSeq,
                             uint_t tableLen, uint_t numThreadBlocks, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // The same shared memory array is used for counting elements greater/lower than pivot and for bitonic sort.
    // max(intra-block scan array size, array size for bitonic sort)
    uint_t sharedMemSize = max(
        2 * THREADS_PER_SORT_LOCAL * sizeof(uint_t), BITONIC_SORT_SIZE_LOCAL * sizeof(*dataInput)
    );
    dim3 dimGrid(numThreadBlocks, 1, 1);
    dim3 dimBlock(THREADS_PER_SORT_LOCAL, 1, 1);

    startStopwatch(&timer);
    error = hipMemcpy(d_localSeq, h_localSeq, numThreadBlocks * sizeof(*d_localSeq), hipMemcpyHostToDevice);
    checkCudaError(error);

    quickSortLocalKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        dataInput, dataBuffer, d_localSeq, tableLen, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing local parallel quicksort.");*/
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void runMinMaxReductionKernel() {

}

void minMaxReduciton(el_t *d_dataInput, el_t *d_dataBuffer, data_t &minVal, data_t &maxVal, uint_t tableLen) {
    // Number of min/max values
    uint_t numValues = tableLen;

    while (numValues < THRESHOLD_REDUCTION) {
        // Half of the array for min values and the other half for max values
        uint_t sharedMemSize = 2 * THREADS_PER_REDUCTION * sizeof(data_t);

        dim3 dimGrid((numValues - 1) / (THREADS_PER_REDUCTION * ELEMENTS_PER_THREAD_REDUCTION) + 1, 1, 1);
        dim3 dimBlock(THREADS_PER_REDUCTION, 1, 1);


        numValues = dimGrid.x;
    }
}

// TODO handle empty sub-blocks
void quickSort(el_t *h_dataInput, el_t *d_dataInput, el_t *d_dataBuffer, h_glob_seq_t *h_globalSeqHost,
               h_glob_seq_t *h_globalSeqHostBuffer, d_glob_seq_t *h_globalSeqDev, d_glob_seq_t *d_globalSeqDev,
               uint_t *h_globalSeqIndexes, uint_t *d_globalSeqIndexes, loc_seq_t *h_localSeq,
               loc_seq_t *d_localSeq, uint_t tableLen, bool orderAsc) {
    // TODO parallel reduction for initial pivot
    // TODO in global quicksort there is no need to calculate min and max after it is calculated first time
    uint_t minVal = min(min(h_dataInput[0].key, h_dataInput[tableLen / 2].key), h_dataInput[tableLen - 1].key);
    uint_t maxVal = max(max(h_dataInput[0].key, h_dataInput[tableLen / 2].key), h_dataInput[tableLen - 1].key);
    h_globalSeqHost[0].setInitSeq(tableLen, (minVal + maxVal) / 2);

    uint_t numSeqGlobal = 1; // Number of sequences for GLOBAL quicksort
    uint_t numSeqLocal = 0;  // Number of sequences for LOCAL quicksort
    uint_t numSeqLimit = (tableLen - 1) / MIN_PARTITION_SIZE_GLOBAL + 1;
    uint_t elemsPerThreadBlock = THREADS_PER_SORT_GLOBAL * ELEMENTS_PER_THREAD_GLOBAL;
    hipError_t error;

    // TODO if statement for initial sequence length
    while (numSeqGlobal + numSeqLocal < numSeqLimit) {
        uint_t threadBlockCounter = 0;

        // Transfers host sequences to device sequences (device needs different data about sequence than host)
        for (uint_t seqIdx = 0; seqIdx < numSeqGlobal; seqIdx++) {
            uint_t threadBlocksPerSeq = (h_globalSeqHost[seqIdx].length - 1) / elemsPerThreadBlock + 1;
            h_globalSeqDev[seqIdx].setFromHostSeq(h_globalSeqHost[seqIdx], threadBlockCounter, threadBlocksPerSeq);

            // For all thread blocks in current iteration marks, they are assigned to current sequence.
            for (uint_t blockIdx = 0; blockIdx < threadBlocksPerSeq; blockIdx++) {
                h_globalSeqIndexes[threadBlockCounter++] = seqIdx;
            }
        }

        runQuickSortGlobalKernel(
            d_dataInput, d_dataBuffer, h_globalSeqDev, d_globalSeqDev, h_globalSeqIndexes,
            d_globalSeqIndexes, numSeqGlobal, threadBlockCounter, tableLen
        );

        /*printf("-----------------------------------\n");
        for (uint_t seqIdx = 0; seqIdx < numSeqGlobal; seqIdx++) {
            printf("%2d ", h_globalSeqHost[seqIdx].pivot);
        }
        printf("\n");
        runPrintTableKernel(d_dataInput, tableLen);
        runPrintTableKernel(d_dataBuffer, tableLen);
        printf("\n-----------------------------------\n");*/

        uint_t numSeqGlobalOld = numSeqGlobal;
        numSeqGlobal = 0;

        // Creates new sub-sequences
        // TODO if sequence length is > 0
        for (uint_t seqIdx = 0; seqIdx < numSeqGlobalOld; seqIdx++) {
            h_glob_seq_t seqHost = h_globalSeqHost[seqIdx];
            d_glob_seq_t seqDev = h_globalSeqDev[seqIdx];

            // New subsequece (lower)
            if (seqDev.offsetLower > MIN_PARTITION_SIZE_GLOBAL) {
                h_globalSeqHostBuffer[numSeqGlobal++].setLowerSeq(seqHost, seqDev);
            } else {
                h_localSeq[numSeqLocal++].setLowerSeq(seqHost, seqDev);
            }

            // New subsequece (greater)
            if (seqDev.offsetGreater > MIN_PARTITION_SIZE_GLOBAL) {
                h_globalSeqHostBuffer[numSeqGlobal++].setGreaterSeq(seqHost, seqDev);
            } else {
                h_localSeq[numSeqLocal++].setGreaterSeq(seqHost, seqDev);
            }
        }

        h_glob_seq_t *temp = h_globalSeqHost;
        h_globalSeqHost = h_globalSeqHostBuffer;
        h_globalSeqHostBuffer = temp;
    }

    // Adds sequences which were not partitioned by global quicksort to sequences for local quicksort
    for (uint_t seqIdx = 0; seqIdx < numSeqGlobal; seqIdx++) {
        h_localSeq[numSeqLocal++].setFromGlobalSeq(h_globalSeqHost[seqIdx]);
    }

    runQuickSortLocalKernel(
        d_dataInput, d_dataBuffer, h_localSeq, d_localSeq, tableLen, numSeqLocal, orderAsc
    );
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, bool orderAsc) {
    // Data memory
    el_t *d_dataInput, *d_dataBuffer;
    // Sequences metadata for GLOBAL quicksort on HOST
    h_glob_seq_t *h_globalSeqHost, *h_globalSeqHostBuffer;
    // Sequences metadata for GLOBAL quicksort on DEVICE
    d_glob_seq_t *h_globalSeqDev, *d_globalSeqDev;
    // Array of sequence indexes for thread blocks in GLOBAL quicksort. This way thread blocks know which
    // sequence they have to partition.
    uint_t *h_globalSeqIndexes, *d_globalSeqIndexes;
    // Sequences metadata for LOCAL quicksort
    loc_seq_t *h_localSeq, *d_localSeq;

    // Maximum number of sequneces which can get generated by global quicksort. In global quicksort sequences
    // are generated untill total number of sequences is lower than: tableLen / MIN_PARTITION_SIZE_GLOBAL.
    uint_t maxNumSequences = 2 * ((tableLen - 1) / MIN_PARTITION_SIZE_GLOBAL + 1);
    // Max number of all thread blocks in GLOBAL quicksort.
    uint_t elemsPerThreadBlock = (THREADS_PER_SORT_GLOBAL * ELEMENTS_PER_THREAD_GLOBAL);
    uint_t maxNumThreadBlocks = maxNumSequences * ((MIN_PARTITION_SIZE_GLOBAL - 1) / elemsPerThreadBlock + 1);

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInitHost(
        &h_globalSeqHost, &h_globalSeqHostBuffer, &h_globalSeqDev, &h_globalSeqIndexes, &h_localSeq,
        maxNumSequences, maxNumThreadBlocks
    );
    memoryInitDevice(
        h_dataInput, &d_dataInput, &d_dataBuffer, &d_globalSeqDev, &d_globalSeqIndexes, &d_localSeq,
        tableLen, maxNumSequences, maxNumThreadBlocks
    );

    startStopwatch(&timer);
    quickSort(
        h_dataInput, d_dataInput, d_dataBuffer, h_globalSeqHost, h_globalSeqHostBuffer, h_globalSeqDev,
        d_globalSeqDev, h_globalSeqIndexes, d_globalSeqIndexes, h_localSeq, d_localSeq, tableLen, orderAsc
    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel quicksort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataBuffer, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
    hipFree(d_dataBuffer);
}
