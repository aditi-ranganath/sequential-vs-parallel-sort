#include "hip/hip_runtime.h"
#include <stdio.h>
#include <climits>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes HOST memory needed for paralel sort implementation.
*/
void memoryInitHost(h_glob_seq_t **h_globalSeqHost, h_glob_seq_t **h_globalSeqHostBuffer,
                    d_glob_seq_t **h_globalSeqDev, uint_t **h_globalSeqIndexes, loc_seq_t **h_localSeq,
                    uint_t maxSequences, uint_t maxNumThreadBlocks) {
    *h_globalSeqHost = new h_glob_seq_t[maxSequences];
    *h_globalSeqHostBuffer = new h_glob_seq_t[maxSequences];
    *h_globalSeqDev = new d_glob_seq_t[maxSequences];
    *h_globalSeqIndexes = new uint_t[maxNumThreadBlocks];
    *h_localSeq = new loc_seq_t[maxSequences];
}

/*
Initializes DEVICE memory needed for paralel sort implementation.
*/
void memoryInitDevice(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, d_glob_seq_t **d_globalSeqDev,
                      uint_t **d_globalSeqIndexes, loc_seq_t **h_localSeq, uint_t tableLen,
                      uint_t maxSequences, uint_t maxNumThreadBlocks) {
    hipError_t error;

    // Data memory allocation
    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);
    // Sequence metadata memory allocation
    error = hipMalloc(d_globalSeqDev, maxSequences * sizeof(**d_globalSeqDev));
    checkCudaError(error);
    error = hipMalloc(d_globalSeqIndexes, maxNumThreadBlocks * sizeof(**d_globalSeqIndexes));
    checkCudaError(error);
    error = hipMalloc(h_localSeq, maxSequences * sizeof(**h_localSeq));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void runQuickSortGlobalKernel(el_t *dataInput, el_t* dataBuffer, d_glob_seq_t *h_globalSeqHost,
                              d_glob_seq_t *d_globalSeqHost, uint_t *h_globalSeqIndexes, uint_t *d_globalSeqIndexes,
                              uint_t hostWorkCounter, uint_t threadBlockCounter, uint_t tableLen) {
    hipError_t error;
    LARGE_INTEGER timer;

    startStopwatch(&timer);

    error = hipMemcpy(d_globalSeqHost, h_globalSeqHost, hostWorkCounter * sizeof(*d_globalSeqHost),
                       hipMemcpyHostToDevice);
    checkCudaError(error);
    error = hipMemcpy(d_globalSeqIndexes, h_globalSeqIndexes, threadBlockCounter * sizeof(*d_globalSeqIndexes),
                       hipMemcpyHostToDevice);
    checkCudaError(error);

    // TODO comment shared memory size, 2 * size should be enough, because scan and min/max can be
    // performed in the same array
    quickSortGlobalKernel<<<threadBlockCounter, THREADS_PER_SORT_GLOBAL, 2 * THREADS_PER_SORT_GLOBAL>>>(
        dataInput, dataBuffer, d_globalSeqHost, d_globalSeqIndexes, tableLen
    );

    error = hipMemcpy(h_globalSeqHost, d_globalSeqHost, hostWorkCounter * sizeof(*h_globalSeqHost),
                       hipMemcpyDeviceToHost);
    checkCudaError(error);

    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing global parallel quicksort.");*/
}

void runQuickSortLocalKernel(el_t *dataInput, el_t *dataBuffer, loc_seq_t *h_localSeq, loc_seq_t *d_localSeq,
                             uint_t tableLen, uint_t numThreadBlocks, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // The same shared memory array is used for counting elements greater/lower than pivot and for bitonic sort.
    // max(intra-block-scan array size, array size for bitonic sort)
    uint_t sharedMemSize = max(
        2 * THREADS_PER_SORT_LOCAL * sizeof(uint_t), BITONIC_SORT_SIZE_LOCAL * sizeof(*dataInput)
    );
    dim3 dimGrid(numThreadBlocks, 1, 1);
    dim3 dimBlock(THREADS_PER_SORT_LOCAL, 1, 1);

    startStopwatch(&timer);
    error = hipMemcpy(d_localSeq, h_localSeq, numThreadBlocks * sizeof(*d_localSeq), hipMemcpyHostToDevice);
    checkCudaError(error);

    quickSortLocalKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        dataInput, dataBuffer, d_localSeq, tableLen, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing local parallel quicksort.");*/
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

// TODO handle empty sub-blocks
void quickSort(el_t *h_dataInput, el_t *d_dataInput, el_t *d_dataBuffer, h_glob_seq_t *h_globalSeqHost,
               h_glob_seq_t *h_globalSeqHostBuffer, d_glob_seq_t *h_globalSeqDev, d_glob_seq_t *d_globalSeqDev,
               uint_t *h_globalSeqIndexes, uint_t *d_globalSeqIndexes, loc_seq_t *h_localSeq,
               loc_seq_t *d_localSeq, uint_t tableLen, uint_t maxSequences, bool orderAsc) {
    // Set starting work
    // TODO parallel reduction for initial pivot
    // TODO in global quicksort there is no need to calculate min and max after it is calculated first time
    uint_t minVal = min(min(h_dataInput[0].key, h_dataInput[tableLen / 2].key), h_dataInput[tableLen - 1].key);
    uint_t maxVal = max(max(h_dataInput[0].key, h_dataInput[tableLen / 2].key), h_dataInput[tableLen - 1].key);
    h_globalSeqHost[0].setInitSeq(tableLen, (minVal + maxVal) / 2);

    // TODO change variable names
    // Size of workstack
    uint_t workTotal = 1;
    uint_t hostWorkCounter = 1;
    uint_t localWorkCounter = 0;
    uint_t elemsPerThreadBlock = THREADS_PER_SORT_GLOBAL * ELEMENTS_PER_THREAD_GLOBAL;
    // Maximum number of sequences, which can be generated with global quicksort
    hipError_t error;

    // TODO if statement for initial sequence length
    while (workTotal < maxSequences) {
        uint_t threadBlockCounter = 0;

        // Store work to device
        for (uint_t workIdx = 0; workIdx < hostWorkCounter; workIdx++) {
            uint_t threadBlocksPerSequence = (h_globalSeqHost[workIdx].length - 1) / elemsPerThreadBlock + 1;

            // For every thread block marks, which sequence they have to partiton (which work they have to perform)
            for (uint_t blockIdx = 0; blockIdx < threadBlocksPerSequence; blockIdx++) {
                h_globalSeqIndexes[threadBlockCounter++] = workIdx;
            }

            // Store work, that thread blocks assigned to current sequence have to perform
            h_globalSeqDev[workIdx].setFromHostSeq(h_globalSeqHost[workIdx], threadBlocksPerSequence);
        }

        runQuickSortGlobalKernel(
            d_dataInput, d_dataBuffer, h_globalSeqDev, d_globalSeqDev, h_globalSeqIndexes,
            d_globalSeqIndexes, hostWorkCounter, threadBlockCounter, tableLen
        );

        /*runPrintTableKernel(d_dataBuffer, tableLen);*/

        uint_t oldHostWorkCounter = hostWorkCounter;
        hostWorkCounter = 0;

        // TODO if sequence length is > 0
        // TODO move to separate method
        // Create new sub-sequences
        for (uint_t workIdx = 0; workIdx < oldHostWorkCounter; workIdx++) {
            h_glob_seq_t hostParams = h_globalSeqHost[workIdx];
            d_glob_seq_t devParams = h_globalSeqDev[workIdx];

            // New subsequece (lower)
            if (devParams.offsetLower > MIN_PARTITION_SIZE_GLOBAL) {
                h_globalSeqHostBuffer[hostWorkCounter++].setLowerSeq(hostParams, devParams);
            } else {
                h_localSeq[localWorkCounter++].setLowerSeq(hostParams, devParams);
            }

            // New subsequece (greater)
            if (devParams.offsetLower > MIN_PARTITION_SIZE_GLOBAL) {
                h_globalSeqHostBuffer[hostWorkCounter++].setGreaterSeq(hostParams, devParams);
            } else {
                h_localSeq[localWorkCounter++].setGreaterSeq(hostParams, devParams);
            }

            workTotal++;
        }

        h_glob_seq_t *temp = h_globalSeqHost;
        h_globalSeqHost = h_globalSeqHostBuffer;
        h_globalSeqHostBuffer = temp;
    }

    // Add sequences which were not partitioned to min size
    for (uint_t workIdx = 0; workIdx < hostWorkCounter; workIdx++) {
        h_localSeq[localWorkCounter++].setFromGlobalSeq(h_globalSeqHost[workIdx]);
    }

    runQuickSortLocalKernel(
        d_dataInput, d_dataBuffer, h_localSeq, d_localSeq, tableLen, workTotal, orderAsc
    );
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, bool orderAsc) {
    // Data memory
    el_t *d_dataInput, *d_dataBuffer;
    // Sequences metadata for GLOBAL quicksort on HOST
    h_glob_seq_t *h_globalSeqHost, *h_globalSeqHostBuffer;
    // Sequences metadata for GLOBAL quicksort on DEVICE
    d_glob_seq_t *h_globalSeqDev, *d_globalSeqDev;
    // Array of sequence indexes for thread blocks in GLOBAL quicksort. This way thread blocks know which
    // sequence they have to partition.
    uint_t *h_globalSeqIndexes, *d_globalSeqIndexes;
    // Sequences metadata for LOCAL quicksort
    loc_seq_t *h_localSeq, *d_localSeq;

    // Maximum number of sequneces which can get generated by global quicksort. In global quicksort sequences
    // are generated untill total number of sequences is lower than tableLen / MIN_PARTITION_SIZE_GLOBAL.
    uint_t maxSequences = 2 * tableLen / MIN_PARTITION_SIZE_GLOBAL - 2;
    // Max number of all thread blocks in GLOBAL quicksort. TODO verify constant 2.
    uint_t maxNumThreadBlocks = 2 * tableLen / (THREADS_PER_SORT_GLOBAL * ELEMENTS_PER_THREAD_GLOBAL);

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInitHost(
        &h_globalSeqHost, &h_globalSeqHostBuffer, &h_globalSeqDev, &h_globalSeqIndexes, &h_localSeq,
        maxSequences, maxNumThreadBlocks
    );
    memoryInitDevice(
        h_dataInput, &d_dataInput, &d_dataBuffer, &d_globalSeqDev, &d_globalSeqIndexes, &d_localSeq,
        tableLen, maxSequences, maxNumThreadBlocks
    );

    startStopwatch(&timer);
    quickSort(
        h_dataInput, d_dataInput, d_dataBuffer, h_globalSeqHost, h_globalSeqHostBuffer, h_globalSeqDev,
        d_globalSeqDev, h_globalSeqIndexes, d_globalSeqIndexes, h_localSeq, d_localSeq, tableLen,
        maxSequences, orderAsc
    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel quicksort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataBuffer, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
    hipFree(d_dataBuffer);
}
