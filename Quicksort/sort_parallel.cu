#include "hip/hip_runtime.h"
#include <stdio.h>
#include <climits>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes HOST memory needed for paralel sort implementation.
*/
void memoryInitHost(h_glob_seq_t **h_globalSeqHost, h_glob_seq_t **h_globalSeqHostBuffer,
                    d_glob_seq_t **h_globalSeqDev, uint_t **h_globalSeqIndexes, loc_seq_t **h_localSeq,
                    uint_t maxSequences, uint_t maxNumThreadBlocks) {
    *h_globalSeqHost = new h_glob_seq_t[maxSequences];
    *h_globalSeqHostBuffer = new h_glob_seq_t[maxSequences];
    *h_globalSeqDev = new d_glob_seq_t[maxSequences];
    *h_globalSeqIndexes = new uint_t[maxNumThreadBlocks];
    *h_localSeq = new loc_seq_t[maxSequences];
}

/*
Initializes DEVICE memory needed for paralel sort implementation.
*/
void memoryInitDevice(el_t *h_input, el_t **d_dataInput, el_t **d_dataBuffer, d_glob_seq_t **d_globalSeqDev,
                      uint_t **d_globalSeqIndexes, loc_seq_t **h_localSeq, uint_t tableLen,
                      uint_t maxSequences, uint_t maxNumThreadBlocks) {
    hipError_t error;

    // Data memory allocation
    error = hipMalloc(d_dataInput, tableLen * sizeof(**d_dataInput));
    checkCudaError(error);
    error = hipMalloc(d_dataBuffer, tableLen * sizeof(**d_dataBuffer));
    checkCudaError(error);
    // Sequence metadata memory allocation
    error = hipMalloc(d_globalSeqDev, maxSequences * sizeof(**d_globalSeqDev));
    checkCudaError(error);
    error = hipMalloc(d_globalSeqIndexes, maxNumThreadBlocks * sizeof(**d_globalSeqIndexes));
    checkCudaError(error);
    error = hipMalloc(h_localSeq, maxSequences * sizeof(**h_localSeq));
    checkCudaError(error);

    error = hipMemcpy(*d_dataInput, h_input, tableLen * sizeof(**d_dataInput), hipMemcpyHostToDevice);
    checkCudaError(error);
}

void runQuickSortGlobalKernel(el_t *input, el_t* output, d_glob_seq_t *h_devGlobalParams,
                              d_glob_seq_t *d_devGlobalParams, uint_t *h_globalSeqIndexes, uint_t *d_globalSeqIndexes,
                              uint_t hostWorkCounter, uint_t threadBlockCounter, uint_t tableLen) {
    hipError_t error;
    LARGE_INTEGER timer;

    startStopwatch(&timer);

    error = hipMemcpy(d_devGlobalParams, h_devGlobalParams, hostWorkCounter * sizeof(*d_devGlobalParams),
                       hipMemcpyHostToDevice);
    checkCudaError(error);
    error = hipMemcpy(d_globalSeqIndexes, h_globalSeqIndexes, threadBlockCounter * sizeof(*d_globalSeqIndexes),
                       hipMemcpyHostToDevice);
    checkCudaError(error);

    // TODO comment shared memory size, 2 * size should be enough, because scan and min/max can be
    // performed in the same array
    quickSortGlobalKernel<<<threadBlockCounter, THREADS_PER_SORT_GLOBAL, 2 * THREADS_PER_SORT_GLOBAL>>>(
        input, output, d_devGlobalParams, d_globalSeqIndexes, tableLen
    );

    error = hipMemcpy(h_devGlobalParams, d_devGlobalParams, hostWorkCounter * sizeof(*h_devGlobalParams),
                       hipMemcpyDeviceToHost);
    checkCudaError(error);

    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing global parallel quicksort.");*/
}

void runQuickSortLocalKernel(el_t *input, el_t *output, loc_seq_t *h_localParams, loc_seq_t *d_localParams,
                             uint_t tableLen, uint_t numThreadBlocks, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // The same shared memory array is used for counting elements greater/lower than pivot and for bitonic sort.
    // max(intra-block-scan array size, array size for bitonic sort)
    uint_t sharedMemSize = max(
        2 * THREADS_PER_SORT_LOCAL * sizeof(uint_t), BITONIC_SORT_SIZE_LOCAL * sizeof(*input)
    );
    dim3 dimGrid(numThreadBlocks, 1, 1);
    dim3 dimBlock(THREADS_PER_SORT_LOCAL, 1, 1);

    startStopwatch(&timer);
    error = hipMemcpy(d_localParams, h_localParams, numThreadBlocks * sizeof(*d_localParams),
                       hipMemcpyHostToDevice);
    checkCudaError(error);

    quickSortLocalKernel<<<dimGrid, dimBlock, sharedMemSize>>>(
        input, output, d_localParams, tableLen, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing local parallel quicksort.");*/
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

// TODO handle empty sub-blocks
void quickSort(el_t *hostData, el_t *dataInput, el_t *dataBuffer, h_glob_seq_t *h_hostGlobalParams,
               h_glob_seq_t *h_hostGlobalBuffer, d_glob_seq_t *h_devGlobalParams, d_glob_seq_t *d_devGlobalParams,
               uint_t *h_globalSeqIndexes, uint_t *d_globalSeqIndexes, loc_seq_t *h_localParams,
               loc_seq_t *d_localParams, uint_t tableLen, bool orderAsc) {
    // Set starting work
    uint_t minVal = min(min(hostData[0].key, hostData[tableLen / 2].key), hostData[tableLen - 1].key);
    uint_t maxVal = max(max(hostData[0].key, hostData[tableLen / 2].key), hostData[tableLen - 1].key);
    // TODO pass pivot to constructor
    h_hostGlobalParams[0].setInitSeq(tableLen, (minVal + maxVal) / 2);

    // Size of workstack
    uint_t workTotal = 1;
    uint_t hostWorkCounter = 1;
    uint_t localWorkCounter = 0;
    uint_t elemsPerThreadBlock = THREADS_PER_SORT_GLOBAL * ELEMENTS_PER_THREAD_GLOBAL;
    // Maximum number of sequences, which can be generated with global quicksort
    uint_t maxSequences = (tableLen - 1) / (MIN_PARTITION_SIZE_GLOBAL * 1) + 1;  // TODO replace 1 with constant
    hipError_t error;

    // TODO if statement for initial sequence length
    while (workTotal < maxSequences) {
        uint_t threadBlockCounter = 0;

        // Store work to device
        for (uint_t workIdx = 0; workIdx < hostWorkCounter; workIdx++) {
            uint_t threadBlocksPerSequence = (h_hostGlobalParams[workIdx].length - 1) / elemsPerThreadBlock + 1;

            // For every thread block marks, which sequence they have to partiton (which work they have to perform)
            for (uint_t blockIdx = 0; blockIdx < threadBlocksPerSequence; blockIdx++) {
                h_globalSeqIndexes[threadBlockCounter++] = workIdx;
            }

            // Store work, that thread blocks assigned to current sequence have to perform
            h_devGlobalParams[workIdx].setFromHostSeq(h_hostGlobalParams[workIdx], threadBlocksPerSequence);
        }

        runQuickSortGlobalKernel(
            dataInput, dataBuffer, h_devGlobalParams, d_devGlobalParams, h_globalSeqIndexes,
            d_globalSeqIndexes, hostWorkCounter, threadBlockCounter, tableLen
        );

        runPrintTableKernel(dataBuffer, tableLen);

        uint_t oldHostWorkCounter = hostWorkCounter;
        hostWorkCounter = 0;

        // Create new sub-sequences
        for (uint_t workIdx = 0; workIdx < oldHostWorkCounter; workIdx++) {
            h_glob_seq_t hostParams = h_hostGlobalParams[workIdx];
            d_glob_seq_t devParams = h_devGlobalParams[workIdx];

            // New subsequece (lower)
            if (devParams.offsetLower > MIN_PARTITION_SIZE_GLOBAL) {
                h_hostGlobalBuffer[hostWorkCounter++].setLowerSeq(hostParams, devParams);
            } else {
                h_localParams[localWorkCounter++].setLowerSeq(hostParams, devParams);
            }

            // New subsequece (greater)
            if (devParams.offsetLower > MIN_PARTITION_SIZE_GLOBAL) {
                h_hostGlobalBuffer[hostWorkCounter++].setGreaterSeq(hostParams, devParams);
            } else {
                h_localParams[localWorkCounter++].setGreaterSeq(hostParams, devParams);
            }

            workTotal++;
        }

        h_glob_seq_t *temp = h_hostGlobalParams;
        h_hostGlobalParams = h_hostGlobalBuffer;
        h_hostGlobalBuffer = temp;
    }

    // Add sequences which were not partitioned to min size
    for (uint_t workIdx = 0; workIdx < hostWorkCounter; workIdx++) {
        h_localParams[localWorkCounter++].setFromGlobalSeq(h_hostGlobalParams[workIdx]);
    }

    runQuickSortLocalKernel(dataInput, dataBuffer, h_localParams, d_localParams, tableLen, workTotal, orderAsc);
}

void sortParallel(el_t *h_dataInput, el_t *h_dataOutput, uint_t tableLen, bool orderAsc) {
    // Data memory
    el_t *d_dataInput, *d_dataBuffer;
    // Sequences metadata for GLOBAL quicksort on HOST
    h_glob_seq_t *h_globalSeqHost, *h_globalSeqHostBuffer;
    // Sequences metadata for GLOBAL quicksort on DEVICE
    d_glob_seq_t *h_globalSeqDev, *d_globalSeqDev;
    // Array of sequence indexes for thread blocks in GLOBAL quicksort. This way thread blocks know which
    // sequence they have to partition.
    uint_t *h_globalSeqIndexes, *d_globalSeqIndexes;
    // Sequences metadata for LOCAL quicksort
    loc_seq_t *h_localSeq, *d_localSeq;

    // Maximum number of sequneces which can get generated by global quicksort. In global quicksort sequences
    // are generated untill total number of sequences is lower than tableLen / MIN_PARTITION_SIZE_GLOBAL.
    uint_t maxSequences = 2 * tableLen / MIN_PARTITION_SIZE_GLOBAL - 2;
    // Max number of all thread blocks in GLOBAL quicksort. TODO verify constant 2.
    uint_t maxNumThreadBlocks = 2 * tableLen / (THREADS_PER_SORT_GLOBAL * ELEMENTS_PER_THREAD_GLOBAL);

    LARGE_INTEGER timer;
    hipError_t error;

    memoryInitHost(
        &h_globalSeqHost, &h_globalSeqHostBuffer, &h_globalSeqDev, &h_globalSeqIndexes, &h_localSeq,
        maxSequences, maxNumThreadBlocks
    );
    memoryInitDevice(
        h_dataInput, &d_dataInput, &d_dataBuffer, &d_globalSeqDev, &d_globalSeqIndexes, &d_localSeq,
        tableLen, maxSequences, maxNumThreadBlocks
    );

    startStopwatch(&timer);
    quickSort(
        h_dataInput, d_dataInput, d_dataBuffer, h_globalSeqHost, h_globalSeqHostBuffer, h_globalSeqDev,
        d_globalSeqDev, h_globalSeqIndexes, d_globalSeqIndexes, h_localSeq, d_localSeq, tableLen, orderAsc
    );

    error = hipDeviceSynchronize();
    checkCudaError(error);
    double time = endStopwatch(timer, "Executing parallel quicksort.");
    printf("Operations (pair swaps): %.2f M/s\n", tableLen / 500.0 / time);

    error = hipMemcpy(h_dataOutput, d_dataBuffer, tableLen * sizeof(*h_dataOutput), hipMemcpyDeviceToHost);
    checkCudaError(error);

    hipFree(d_dataInput);
    hipFree(d_dataBuffer);
}
