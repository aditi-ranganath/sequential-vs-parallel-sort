#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "data_types.h"
#include "constants.h"

///////////////////////////////////////////////////////////////////
////////////////////////////// UTILS //////////////////////////////
///////////////////////////////////////////////////////////////////


////////////////////////// GENERAL UTILS //////////////////////////

/*
http://stackoverflow.com/questions/1582356/fastest-way-of-finding-the-middle-value-of-a-triple
*/
__device__ uint_t getMedian(uint_t a, uint_t b, uint_t c) {
    uint_t maxVal = max(max(a, b), c);
    uint_t minVal = min(min(a, b), c);

    return a ^ b ^ c ^ maxVal ^ minVal;
}


/////////////////////////// SCAN UTILS ////////////////////////////

/*
Performs scan and computes, how many elements have 'true' predicate before current element.
*/
__device__ uint_t intraWarpScan(volatile uint_t *scanTile, uint_t val, uint_t stride) {
    // The same kind of indexing as for bitonic sort
    uint_t index = 2 * threadIdx.x - (threadIdx.x & (stride - 1));

    scanTile[index] = 0;
    index += stride;
    scanTile[index] = val;

    if (stride > 1) {
        scanTile[index] += scanTile[index - 1];
    }
    if (stride > 2) {
        scanTile[index] += scanTile[index - 2];
    }
    if (stride > 4) {
        scanTile[index] += scanTile[index - 4];
    }
    if (stride > 8) {
        scanTile[index] += scanTile[index - 8];
    }
    if (stride > 16) {
        scanTile[index] += scanTile[index - 16];
    }

    // Converts inclusive scan to exclusive
    return scanTile[index] - val;
}

__device__ uint_t intraBlockScan(uint_t val) {
    extern __shared__ uint_t scanTile[];
    uint_t warpIdx = threadIdx.x / warpSize;
    uint_t laneIdx = threadIdx.x & (warpSize - 1);  // Thread index inside warp

    uint_t warpResult = intraWarpScan(scanTile, val, warpSize);
    __syncthreads();

    if (laneIdx == warpSize - 1) {
        scanTile[warpIdx] = warpResult;
    }
    __syncthreads();

    // Maximum number of elements for scan is warpSize ^ 2
    if (threadIdx.x < blockDim.x / warpSize) {
        scanTile[threadIdx.x] = intraWarpScan(scanTile, scanTile[threadIdx.x], blockDim.x / warpSize);
    }
    __syncthreads();

    return warpResult + scanTile[warpIdx];
}


/////////////////////// BITONIC SORT UTILS ////////////////////////

/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if (((int_t)(elem1->key - elem2->key) <= 0) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
Sorts input data with NORMALIZED bitonic sort (all comparisons are made in same direction,
easy to implement for input sequences of arbitrary size) and outputs them to output array.

- TODO use quick sort kernel instead of bitonic sort
*/
__device__ void normalizedBitonicSort(el_t *input, el_t *output, lparam_t localParams, uint_t tableLen, bool orderAsc) {
    extern __shared__ el_t sortTile[];

    // Read data from global to shared memory.
    for (uint_t tx = threadIdx.x; tx < localParams.length; tx += blockDim.x) {
        sortTile[tx] = input[localParams.start + tx];
    }
    __syncthreads();

    // Bitonic sort PHASES
    for (uint_t subBlockSize = 1; subBlockSize < localParams.length; subBlockSize <<= 1) {
        // Bitonic merge STEPS
        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            for (uint_t tx = threadIdx.x; tx < (tableLen / MAX_SEQUENCES) >> 1; tx += blockDim.x) {
                uint_t indexThread = tx;
                uint_t offset = stride;

                // In normalized bitonic sort, first STEP of every PHASE uses different offset than all other
                // STEPS. Also in first step of every phase, offsets sizes are generated in ASCENDING order
                // (normalized bitnic sort requires DESCENDING order). Because of that we can break the loop if
                // index + offset >= length (bellow). If we want to generate offset sizes in ASCENDING order,
                // than thread indexes inside every sub-block have to be reversed.
                if (stride == subBlockSize) {
                    indexThread = (tx / stride) * stride + ((stride - 1) - (tx % stride));
                    offset = ((tx & (stride - 1)) << 1) + 1;
                }

                uint_t index = (indexThread << 1) - (indexThread & (stride - 1));
                if (index + offset >= localParams.length) {
                    break;
                }

                compareExchange(&sortTile[index], &sortTile[index + offset], orderAsc);
            }
            __syncthreads();
        }
    }

    // Store data from shared to global memory
    for (uint_t tx = threadIdx.x; tx < localParams.length; tx += blockDim.x) {
        output[localParams.start + tx] = sortTile[tx];
    }
}


///////////////////////////////////////////////////////////////////
///////////////////////////// KERNELS /////////////////////////////
///////////////////////////////////////////////////////////////////

// TODO in general chech if __shared__ values work faster (pivot, array1, array2, ...)
// TODO try alignment with 32 because of bank conflicts.
__global__ void quickSortLocalKernel(el_t *input, el_t *output, lparam_t *localParams, uint_t tableLen,
                                     bool orderAsc) {
    __shared__ extern uint_t localSortTile[];

    // Array of counters for elements lower/greater than pivot. One element belongs to one thread.
    uint_t *lowerThanPivot = localSortTile;
    uint_t *greaterThanPivot = localSortTile + blockDim.x;

    // Explicit stack (instead of recursion) for work to be done
    // TODO allocate memory dynamically according to sub-block size
    __shared__ lparam_t workstack[32];
    __shared__ uint_t workstackCounter;

    workstack[0] = localParams[blockIdx.x];
    workstackCounter = 1;

    while (workstackCounter > 0) {
        // TODO try with explicit local values start, end, direction
        lparam_t params = workstack[workstackCounter - 1];
        uint_t end = params.start + params.length;

        if (params.length <= BITONIC_SORT_SIZE_LOCAL) {
            // Bitonic sort is executed in-place and sorted data has to be writter to output.
            el_t *inputTemp = params.direction ? output : input;

            normalizedBitonicSort(inputTemp, output, params, tableLen, orderAsc);
            // TODO verify if syncthreads() is needed
            __syncthreads();

            workstackCounter--;
            continue;
        }

        // In order not to spoil references *input and *output, additional 2 local references are used
        el_t *array1 = params.direction ? output : input;
        el_t *array2 = params.direction ? input : output;

        uint_t pivot = getMedian(
            array1[params.start].key, array1[(params.start + end) / 2].key, array1[end].key
        );

        // Counter of number of elements, which are lower/greater than pivot
        uint_t lowerCounter = 0;
        uint_t greaterCounter = 0;

        // Every thread counts the number of elements lower/greater than pivot
        for (uint_t index = params.start + threadIdx.x; index < end; index += blockDim.x) {
            el_t temp = array1[index];
            lowerCounter += temp.key < pivot;
            greaterCounter += temp.key > pivot;
        }
        __syncthreads();

        // Calculates global offsets for each thread
        lowerCounter = intraBlockScan(lowerCounter);
        greaterCounter = intraBlockScan(greaterCounter);
        __syncthreads();

        workstackCounter--;
    }
}
