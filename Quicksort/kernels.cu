#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <climits>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if (((int_t)(elem1->key - elem2->key) <= 0) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
Sorts input data with NORMALIZED bitonic sort (all comparisons are made in same direction,
easy to implement for input sequences of arbitrary size) and outputs them to output array.

- TODO use quick sort kernel instead of bitonic sort
*/
__device__ void bitonicSortKernel(el_t *input, el_t *output, lparam_t localParam, uint_t tableLen, bool orderAsc) {
    extern __shared__ el_t sortTile[];

    // Read data from global to shared memory.
    for (uint_t tx = threadIdx.x; tx < localParam.length; tx += blockDim.x) {
        sortTile[tx] = input[localParam.start + tx];
    }
    __syncthreads();

    // Bitonic sort PHASES
    for (uint_t subBlockSize = 1; subBlockSize < localParam.length; subBlockSize <<= 1) {
        // Bitonic merge STEPS
        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            for (uint_t tx = threadIdx.x; tx < (tableLen / MAX_SEQUENCES) >> 1; tx += blockDim.x) {
                uint_t indexThread = tx;
                uint_t offset = stride;

                // In normalized bitonic sort, first STEP of every PHASE uses different offset than all other
                // STEPS. Also in first step of every phase, offsets sizes are generated in ASCENDING order
                // (normalized bitnic sort requires DESCENDING order). Because of that we can break the loop if
                // index + offset >= length (bellow). If we want to generate offset sizes in ASCENDING order,
                // than thread indexes inside every sub-block have to be reversed.
                if (stride == subBlockSize) {
                    indexThread = (tx / stride) * stride + ((stride - 1) - (tx % stride));
                    offset = ((tx & (stride - 1)) << 1) + 1;
                }

                uint_t index = (indexThread << 1) - (indexThread & (stride - 1));
                if (index + offset >= localParam.length) {
                    break;
                }

                compareExchange(&sortTile[index], &sortTile[index + offset], orderAsc);
            }
            __syncthreads();
        }
    }

    // Store data from shared to global memory
    for (uint_t tx = threadIdx.x; tx < localParam.length; tx += blockDim.x) {
        output[localParam.start + tx] = sortTile[tx];
    }
}

__global__ void quickSortLocalKernel(el_t *input, el_t *output, lparam_t *localParams, uint_t tableLen,
                                     bool orderAsc) {
    bitonicSortKernel(input, output, localParams[blockIdx.x], tableLen, orderAsc);
}
