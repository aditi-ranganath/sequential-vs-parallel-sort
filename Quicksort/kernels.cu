#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if (((int_t)(elem1->key - elem2->key) <= 0) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
http://stackoverflow.com/questions/1582356/fastest-way-of-finding-the-middle-value-of-a-triple
*/
__device__ uint_t getMedian(uint_t a, uint_t b, uint_t c) {
    uint_t maxVal = max(max(a, b), c);
    uint_t minVal = min(min(a, b), c);

    return a ^ b ^ c ^ maxVal ^ minVal;
}

/*
Sorts input data with NORMALIZED bitonic sort (all comparisons are made in same direction,
easy to implement for input sequences of arbitrary size) and outputs them to output array.

- TODO use quick sort kernel instead of bitonic sort
*/
__device__ void normalizedBitonicSort(el_t *input, el_t *output, lparam_t localParams, uint_t tableLen, bool orderAsc) {
    extern __shared__ el_t sortTile[];

    // Read data from global to shared memory.
    for (uint_t tx = threadIdx.x; tx < localParams.length; tx += blockDim.x) {
        sortTile[tx] = input[localParams.start + tx];
    }
    __syncthreads();

    // Bitonic sort PHASES
    for (uint_t subBlockSize = 1; subBlockSize < localParams.length; subBlockSize <<= 1) {
        // Bitonic merge STEPS
        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            for (uint_t tx = threadIdx.x; tx < (tableLen / MAX_SEQUENCES) >> 1; tx += blockDim.x) {
                uint_t indexThread = tx;
                uint_t offset = stride;

                // In normalized bitonic sort, first STEP of every PHASE uses different offset than all other
                // STEPS. Also in first step of every phase, offsets sizes are generated in ASCENDING order
                // (normalized bitnic sort requires DESCENDING order). Because of that we can break the loop if
                // index + offset >= length (bellow). If we want to generate offset sizes in ASCENDING order,
                // than thread indexes inside every sub-block have to be reversed.
                if (stride == subBlockSize) {
                    indexThread = (tx / stride) * stride + ((stride - 1) - (tx % stride));
                    offset = ((tx & (stride - 1)) << 1) + 1;
                }

                uint_t index = (indexThread << 1) - (indexThread & (stride - 1));
                if (index + offset >= localParams.length) {
                    break;
                }

                compareExchange(&sortTile[index], &sortTile[index + offset], orderAsc);
            }
            __syncthreads();
        }
    }

    // Store data from shared to global memory
    for (uint_t tx = threadIdx.x; tx < localParams.length; tx += blockDim.x) {
        output[localParams.start + tx] = sortTile[tx];
    }
}

// TODO in general chech if __shared__ values work faster (pivot, array1, array2, ...)
// TODO try alignment with 32 because of bank conflicts.
__global__ void quickSortLocalKernel(el_t *input, el_t *output, lparam_t *localParams, uint_t tableLen,
                                     bool orderAsc) {
    __shared__ extern uint_t localSortTile[];

    // Array of counters for elements lower/greater than pivot. One element belongs to one thread.
    uint_t *lowerThanPivot = localSortTile;
    uint_t *greaterThanPivot = localSortTile + blockDim.x;

    // Explicit stack (instead of recursion) for work to be done
    // TODO allocate memory dynamically according to sub-block size
    __shared__ lparam_t workstack[32];
    __shared__ uint_t workstackCounter;

    workstack[0] = localParams[blockIdx.x];
    workstackCounter = 1;

    while (workstackCounter > 0) {
        // TODO try with explicit local values start, end, direction
        lparam_t params = workstack[workstackCounter - 1];

        if (params.length <= BITONIC_SORT_SIZE_LOCAL) {
            // Bitonic sort is executed in-place and sorted data has to be writter to output.
            el_t *inputTemp = params.direction ? output : input;
            normalizedBitonicSort(inputTemp, output, params, tableLen, orderAsc);

            workstackCounter--;
            continue;
        }

        // In order not to spoil references *input and *output, additional 2 local references are used
        el_t *array1 = params.direction ? output : input;
        el_t *array2 = params.direction ? input : output;

        uint_t pivot = getMedian(
            array1[params.start].key, array1[(params.start + params.length) / 2].key,
            array1[params.start + params.length].key
        );

        uint_t lowerCounter = 0;
        uint_t greaterCounter = 0;
    }
}
