#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <climits>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if (((int_t)(elem1->key - elem2->key) <= 0) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
Sorts input data with bitonic sort and outputs them to output array.
- TODO use quick sort kernel instead of bitonic sort
*/
__device__ void bitonicSortKernel(el_t *input, el_t *output, uint_t tableLen, bool orderAsc) {
    extern __shared__ el_t sortTile[];
    uint_t index = blockIdx.x * ELEMENTS_PER_THREAD_LOCAL * blockDim.x + threadIdx.x;

    // Read data from global to shared memory
    for (uint_t i = 0; i < ELEMENTS_PER_THREAD_LOCAL; i++) {
        sortTile[i * blockDim.x + threadIdx.x] = input[i * blockDim.x + index];
    }

    // By default every thread sorts 2 elements with bitonic sort
    for (uint_t offsetFactor = 0; offsetFactor < ELEMENTS_PER_THREAD_LOCAL / 2; offsetFactor++) {
        uint_t tx = offsetFactor * 2 * blockDim.x + threadIdx.x;

        // Bitonic sort
        for (uint_t subBlockSize = 1; subBlockSize <= blockDim.x; subBlockSize <<= 1) {
            bool direction = (tx & subBlockSize) != 0;

            for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
                __syncthreads();
                uint_t start = 2 * tx - (tx & (stride - 1));
                compareExchange(&sortTile[start], &sortTile[start + stride], direction);
            }
        }
    }

    // Store data from shared to global memory
    __syncthreads();
    for (uint_t i = 0; i < ELEMENTS_PER_THREAD_LOCAL; i++) {
        output[i * blockDim.x + index] = sortTile[i * blockDim.x + threadIdx.x];
    }
}

__global__ void quickSortLocalKernel(el_t *input, el_t *output, uint_t tableLen, bool orderAsc) {
    bitonicSortKernel(input, output, tableLen, orderAsc);
}
