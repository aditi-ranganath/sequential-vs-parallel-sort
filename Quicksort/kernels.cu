#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <math.h>
#include <climits>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if (((int_t)(elem1->key - elem2->key) <= 0) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

/*
Sorts input data with bitonic sort and outputs them to output array.
- TODO use quick sort kernel instead of bitonic sort
*/
__device__ void bitonicSortKernel(el_t *input, el_t *output, uint_t start, uint_t end, bool orderAsc) {
    extern __shared__ el_t sortTile[];
    uint_t elementsPerBlock = blockDim.x * ELEMENTS_PER_THREAD_LOCAL;

    // Read data from global to shared memory
    for (uint_t i = 0; i < ELEMENTS_PER_THREAD_LOCAL; i++) {
        uint_t index = start + i * blockDim.x + threadIdx.x;
        if (index >= end) {
            break;
        }

        sortTile[i * blockDim.x + threadIdx.x] = input[index];
    }
    __syncthreads();

    // Bitonic sort
    for (uint_t subBlockSize = 1; subBlockSize < elementsPerBlock; subBlockSize <<= 1) {
        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {

            // Every thread can sort/exchange 2 or more elements (at least 2 and only power of 2)
            for (uint_t offsetFactor = 0; offsetFactor < ELEMENTS_PER_THREAD_LOCAL / 2; offsetFactor++) {
                // TODO check if bottom 2 statements can be moved outside this for loop
                uint_t tx = offsetFactor * blockDim.x + threadIdx.x;
                bool direction = orderAsc ^ ((tx & subBlockSize) != 0);

                // Calculate the index, from which thread is going to do an exchange
                uint_t index = 2 * tx - (tx & (stride - 1));
                if (start + index + stride > end) {
                    break;
                }

                compareExchange(&sortTile[index], &sortTile[index + stride], direction);
            }
            __syncthreads();
        }
    }

    // Store data from shared to global memory
    for (uint_t i = 0; i < ELEMENTS_PER_THREAD_LOCAL; i++) {
        uint_t index = start + i * blockDim.x + threadIdx.x;
        if (index >= end) {
            break;
        }

        output[index] = sortTile[i * blockDim.x + threadIdx.x];
    }
}

__global__ void quickSortLocalKernel(el_t *input, el_t *output, uint_t tableLen, bool orderAsc) {
    uint_t start = blockIdx.x * (blockDim.x * ELEMENTS_PER_THREAD_LOCAL);
    uint_t end = (blockIdx.x + 1) * (blockDim.x * ELEMENTS_PER_THREAD_LOCAL);
    bitonicSortKernel(input, output, start, end, orderAsc);
}
