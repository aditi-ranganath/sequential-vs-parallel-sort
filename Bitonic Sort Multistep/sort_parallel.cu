#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <Windows.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"
#include "utils_cuda.h"
#include "utils_host.h"
#include "kernels.h"


/*
Initializes device memory.
*/
void memoryDataInit(el_t *h_table, el_t **d_table, uint_t tableLen) {
    hipError_t error;

    error = hipMalloc(d_table, tableLen * sizeof(**d_table));
    checkCudaError(error);
    error = hipMemcpy(*d_table, h_table, tableLen * sizeof(**d_table), hipMemcpyHostToDevice);
    checkCudaError(error);
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
void runBitoicSortKernel(el_t *table, uint_t tableLen, uint_t subBlockSize, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    startStopwatch(&timer);
    bitonicSortKernel<<<dimGrid, dimBlock, subBlockSize * sizeof(*table)>>>(
        table, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);*/
    //endStopwatch(timer, "Executing bitonic sort kernel");
}

void runMultiStepKernel(el_t *table, uint_t tableLen, uint_t phase, uint_t step, uint_t degree, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    uint_t partitionSize = tableLen / (1 << degree);
    uint_t maxThreadBlockSize = MAX_THREADS_PER_MULTISTEP;
    uint_t threadBlockSize = min(partitionSize, maxThreadBlockSize);
    dim3 dimGrid(tableLen / (2 * threadBlockSize), 1, 1);
    dim3 dimBlock(threadBlockSize, 1, 1);

    startStopwatch(&timer);
    multiStepKernel<<<dimGrid, dimBlock, 2 * MAX_THREADS_PER_MULTISTEP * sizeof(*table)>>>(
        table, phase, step, degree, orderAsc
    );
    error = hipDeviceSynchronize();
    checkCudaError(error);
    printf("\n");
    /*endStopwatch(timer, "Executing multistep kernel");*/
}

void runBitoicMergeKernel(el_t *table, uint_t tableLen, uint_t subBlockSize, uint_t phase, bool orderAsc) {
    hipError_t error;
    LARGE_INTEGER timer;

    // Every thread loads and sorts 2 elements
    dim3 dimGrid(tableLen / subBlockSize, 1, 1);
    dim3 dimBlock(subBlockSize / 2, 1, 1);

    startStopwatch(&timer);
    bitonicMergeKernel<<<dimGrid, dimBlock, subBlockSize * sizeof(*table)>>>(
        table, phase, orderAsc
    );
    /*error = hipDeviceSynchronize();
    checkCudaError(error);*/
    //endStopwatch(timer, "Executing bitonic sort kernel");
}

void runPrintTableKernel(el_t *table, uint_t tableLen) {
    printTableKernel<<<1, 1>>>(table, tableLen);
    hipError_t error = hipDeviceSynchronize();
    checkCudaError(error);
}

void sortParallel(el_t *h_input, el_t *h_output, uint_t tableLen, bool orderAsc) {
    el_t *d_table;
    // Every thread loads and sorts 2 elements in first bitonic sort kernel
    uint_t subBlockSize = 2;  //  min(tableLen, 2 * getMaxThreadsPerBlock());
    int_t phasesAll = log2((double)tableLen);
    int_t phasesSharedMem = log2((double)subBlockSize);

    LARGE_INTEGER timer;
    hipError_t error;

    // In multistep kernel no shared memory is used, that's why preference can be set for L1
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(multiStepKernel), hipFuncCachePreferL1);
    memoryDataInit(h_input, &d_table, tableLen);

    startStopwatch(&timer);
    runBitoicSortKernel(d_table, tableLen, subBlockSize, orderAsc);

    /*printf("After bitonic sort\n");
    runPrintTableKernel(d_table, tableLen);*/

    for (uint_t phase = phasesSharedMem + 1; phase <= phasesAll; phase++) {
        int_t step = phase;

        for (uint_t degree = MAX_MULTI_STEP; degree > 0; degree--) {
            for (; step >= phasesSharedMem + degree; step -= degree) {
                printf("Degree: %d\n", degree);
                runMultiStepKernel(d_table, tableLen, phase, step, degree, orderAsc);
                /*if (phase == 5) {
                printf("After 2-multistep\n");
                runPrintTableKernel(d_table, tableLen);
                }*/
            }
        }

        // Here only last phase is needed
        runBitoicMergeKernel(d_table, tableLen, subBlockSize, phase, orderAsc);

        /*if (phase == 2) {
            printf("After bitonic merge\n");
            runPrintTableKernel(d_table, tableLen);
        }*/
    }

    error = hipDeviceSynchronize();
    checkCudaError(error);
    endStopwatch(timer, "Executing parallel bitonic sort.");

    error = hipMemcpy(h_output, d_table, tableLen * sizeof(*h_output), hipMemcpyDeviceToHost);
    checkCudaError(error);
}
