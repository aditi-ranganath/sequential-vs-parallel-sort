#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "data_types.h"
#include "constants.h"


/*
Compares 2 elements and exchanges them according to orderAsc.
*/
__device__ void compareExchange(el_t *elem1, el_t *elem2, bool orderAsc) {
    if ((elem1->key <= elem2->key) ^ orderAsc) {
        el_t temp = *elem1;
        *elem1 = *elem2;
        *elem2 = temp;
    }
}

__global__ void printTableKernel(el_t *table, uint_t tableLen) {
    for (uint_t i = 0; i < tableLen; i++) {
        printf("%2d ", table[i]);
    }
    printf("\n\n");
}

/*
Sorts sub-blocks of input data with bitonic sort.
*/
__global__ void bitonicSortKernel(el_t *table, bool orderAsc) {
    extern __shared__ el_t sortTile[];
    // If shared memory size is lower than table length, than every block has to be ordered
    // in opposite direction -> bitonic sequence.
    bool blockDirection = orderAsc ^ (blockIdx.x & 1);

    // Every thread loads 2 elements
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    sortTile[threadIdx.x] = table[index];
    sortTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    // Bitonic sort
    for (uint_t subBlockSize = 1; subBlockSize <= blockDim.x; subBlockSize <<= 1) {
        bool direction = blockDirection ^ ((threadIdx.x & subBlockSize) != 0);

        for (uint_t stride = subBlockSize; stride > 0; stride >>= 1) {
            __syncthreads();
            uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
            compareExchange(&sortTile[start], &sortTile[start + stride], direction);
        }
    }

    __syncthreads();
    table[index] = sortTile[threadIdx.x];
    table[blockDim.x + index] = sortTile[blockDim.x + threadIdx.x];
}

/*
Multistep kernel using registers. Every thread loads and sorts all elements for it's corresponding
subsection of partition (one thread loads and sorts >= 2 elements).
*/
__global__ void multiStepRegistersKernel(el_t *table, uint_t phase, uint_t step, uint_t degree, bool orderAsc) {
    el_t tile[1 << MAX_MULTI_STEP];
    uint_t tileHalfSize = 1 << (degree - 1);
    uint_t strideGlobal = 1 << (step - 1);
    uint_t threadsPerSubBlock = 1 << (step - degree);
    uint_t indexThread = blockIdx.x * blockDim.x + threadIdx.x;
    uint_t indexTable = (indexThread >> (step - degree) << step) + indexThread % threadsPerSubBlock;
    bool direction = orderAsc ^ ((indexThread >> (phase - degree)) & 1);

    // Each thread loads elements for it's corresponding subsection of partition
    for (uint_t i = 0; i < tileHalfSize; i++) {
        uint_t start = indexTable + i * threadsPerSubBlock;
        tile[i] = table[start];
        tile[i + tileHalfSize] = table[start + strideGlobal];
    }

    // Syncthreads is not needed, because every thread proceses an separated subsection of partition
    for (uint_t strideLocal = tileHalfSize; strideLocal > 0; strideLocal >>= 1) {
        for (uint_t i = 0; i < tileHalfSize; i++) {
            uint_t start = 2 * i - (i & (strideLocal - 1));
            compareExchange(&tile[start], &tile[start + strideLocal], direction);
        }
    }

    for (int i = 0; i < tileHalfSize; i++) {
        uint_t start = indexTable + i * threadsPerSubBlock;
        table[start] = tile[i];
        table[start + strideGlobal] = tile[i + tileHalfSize];
    }
}

/*
Multistep kernel using registers. Every thread loads only 2 elements. Than all threads
execute bitonic merge.
*/
__global__ void multiStepSharedMemKernel(el_t *table, uint_t phase, uint_t step, uint_t degree, bool orderAsc) {
    extern __shared__ el_t tile[];
    uint_t strideGlobal = 1 << (step - 1);
    uint_t threadsPerSubBlock = 1 << (step - degree);
    uint_t indexThread = blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x + threadIdx.x;
    uint_t indexTable = ((indexThread * threadsPerSubBlock) % strideGlobal) + ((indexThread >> (degree - 1)) % threadsPerSubBlock);
    indexTable += indexThread >> (step - 1) << step;
    uint_t bla = (threadIdx.x >> (degree - 1) << (degree)) + (threadIdx.x % (1 << (degree - 1)));
    bool direction = orderAsc ^ ((indexThread >> (phase - 1)) & 1);

    tile[bla] = table[indexTable];
    tile[bla + (1 << (degree - 1))] = table[indexTable + strideGlobal];

    // All threads execute bitonic merge in shared memory (syncthreads needed).
    for (uint_t stride = 1 << (degree - 1); stride > 0; stride >>= 1) {
        __syncthreads();
        uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        compareExchange(&tile[start], &tile[start + stride], direction);
    }

    __syncthreads();
    table[indexTable] = tile[bla];
    table[indexTable + strideGlobal] = tile[bla + (1 << (degree - 1))];
}

/*
Global bitonic merge for sections, where stride IS LOWER OR EQUAL than max shared memory.
*/
__global__ void bitonicMergeKernel(el_t *table, uint_t phase, bool orderAsc) {
    extern __shared__ el_t mergeTile[];
    uint_t index = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    // Elements inside same sub-block have to be ordered in same direction
    bool direction = orderAsc ^ ((index >> phase) & 1);

    // Every thread loads 2 elements
    mergeTile[threadIdx.x] = table[index];
    mergeTile[blockDim.x + threadIdx.x] = table[blockDim.x + index];

    // Bitonic merge
    for (uint_t stride = blockDim.x; stride > 0; stride >>= 1) {
        __syncthreads();
        uint_t start = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
        compareExchange(&mergeTile[start], &mergeTile[start + stride], direction);
    }

    __syncthreads();
    table[index] = mergeTile[threadIdx.x];
    table[blockDim.x + index] = mergeTile[blockDim.x + threadIdx.x];
}
